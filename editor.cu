#include "hip/hip_runtime.h"
#include "../include/app.hpp"
#include "../include/backend.hpp"
#include "../include/layers/common.hpp"
#include "../include/layers/forward_renderer.hpp"
#include "../include/layers/image_renderer.hpp"
#include "../include/layers/objectifier.hpp"
#include "../include/layers/ui.hpp"
#include "../include/project.hpp"
#include "../include/scene.hpp"
#include "../include/shader_program.hpp"
#include "../include/ui/attachment.hpp"
#include "../include/engine/irradiance_computer.hpp"
#include "../include/amadeus/armada.cuh"
#include "../include/amadeus/path_tracer.cuh"
#include "../include/layers/framer.hpp"
#include "../include/cuda/color.cuh"
#include "../include/layers/denoiser.cuh"

// Forward declarations
struct ProgressBar;
struct InfoTab;
struct MaterialEditor;
struct RTXRenderer;
struct Viewport;

// TODO: add updated (emissive) materials as lights...

// TODO: logging attachment
// TODO: info tab that shows logging and framerate...
// TODO: viewport attachment

struct Editor : public kobra::BaseApp {
	kobra::Scene m_scene;
	kobra::Entity m_camera;

	kobra::layers::ForwardRenderer m_forward_renderer;
	kobra::layers::Objectifier m_objectifier;

	std::shared_ptr <kobra::layers::UI> m_ui;

	std::shared_ptr <ProgressBar> m_progress_bar;
	std::shared_ptr <InfoTab> m_info_tab;
	std::shared_ptr <MaterialEditor> m_material_editor;

	kobra::engine::IrradianceComputer m_irradiance_computer;
	bool m_saved_irradiance = false;

	// Renderers
	struct {
		std::shared_ptr <kobra::amadeus::System> system;
		std::shared_ptr <kobra::layers::MeshMemory> mesh_memory;
		std::shared_ptr <kobra::amadeus::ArmadaRTX> armada_rtx;

		kobra::layers::Denoiser denoiser;
		kobra::layers::Framer framer;

		std::mutex movement_mutex;
		std::queue <uint32_t> movement;

		int mode = 0;
		bool denoise = true;
	} m_renderers;

	// Viewport
	struct {
		kobra::ImageData image = nullptr;
		vk::raii::Sampler sampler = nullptr;
		ImVec2 min = {1/0.0f, 1/0.0f};
		ImVec2 max = {-1.0f, -1.0f};
	} m_viewport;

	// Buffers
	struct {
		hipDeviceptr_t traced;
		std::vector <uint8_t> traced_cpu;
	} m_buffers;

	struct Request {
		float x;
		float y;
	};

	std::queue <Request> request_queue;
	std::pair <int, int> m_selection = {-1, -1};

	// Input state
	// TODO: bring all other related fields here
	struct {
		bool viewport_hovered = false;
		bool viewport_focused = false;
	} m_input;

	Editor(const vk::raii::PhysicalDevice &, const std::vector <const char *> &);
	~Editor();

	void record(const vk::raii::CommandBuffer &, const vk::raii::Framebuffer&) override;
	void resize(const vk::Extent2D &) override;
	void after_present() override;

	static void mouse_callback(void *, const kobra::io::MouseEvent &);
	static void keyboard_callback(void *, const kobra::io::KeyboardEvent &);

	// TODO: frustrum culling structure to cull once per pass (store status
	// in a map) and then is passed to other layers for rendering
};

int main()
{
	// Load Vulkan physical device
	auto predicate = [](const vk::raii::PhysicalDevice &dev) {
		return kobra::physical_device_able(dev,  {
			VK_KHR_SWAPCHAIN_EXTENSION_NAME,
			VK_KHR_GET_MEMORY_REQUIREMENTS_2_EXTENSION_NAME,
			VK_KHR_EXTERNAL_MEMORY_EXTENSION_NAME,
			VK_KHR_EXTERNAL_SEMAPHORE_EXTENSION_NAME,
		});
	};

	vk::raii::PhysicalDevice phdev = kobra::pick_physical_device(predicate);

	Editor editor {
		phdev, {
			VK_KHR_SWAPCHAIN_EXTENSION_NAME,
			VK_KHR_EXTERNAL_MEMORY_FD_EXTENSION_NAME,
			VK_KHR_EXTERNAL_MEMORY_EXTENSION_NAME,
		},
	};

	editor.run();
}

// Progress bar UI Attachment
struct ProgressBar : public kobra::ui::ImGuiAttachment {
	std::string m_title;
	float m_progress = 0.0f;

	ProgressBar(const std::string &title)
		: m_title {title} {}

	void render() override {
		// Set font size
		ImGui::Begin(m_title.c_str());
		ImGui::SetWindowSize(ImVec2(500, 100), ImGuiCond_FirstUseEver);
		ImGui::ProgressBar(m_progress);
		ImGui::End();
	}
};

// Info UI Attachment
struct InfoTab : public kobra::ui::ImGuiAttachment {
	std::vector <std::string> m_lines;
	std::string m_message;

	InfoTab() {
		// Attach logger handler
		kobra::add_log_handler(this,
			[&](const char *str, std::streamsize n) {
				m_message += std::string(str, n);

				std::string message_remainder = m_message;
				for (size_t i = 0; i < m_message.size(); i++) {
					if (m_message[i] == '\n') {
						m_lines.push_back(m_message.substr(0, i));
						message_remainder = m_message.substr(i + 1);
					}
				}
			}
		);
	}

	~InfoTab() {
		kobra::remove_log_handler(this);
	}

	void render() override {
		// Output and performance tabs
		ImGui::Begin("Info");

		ImGui::SetWindowSize(ImVec2(500, 500), ImGuiCond_FirstUseEver);

		// TODO: dock for framerate and performance
		ImGui::Text("Output");
		ImGui::Separator();

		for (const auto &line : m_lines)
			ImGui::Text(line.c_str());

		ImGui::End();
	}
};

// Material editor UI attachment
class MaterialEditor : public kobra::ui::ImGuiAttachment {
	int m_prev_material_index = -1;

	vk::DescriptorSet m_diffuse_set;
	vk::DescriptorSet m_normal_set;

	glm::vec3 emission_base = glm::vec3(0.0f);
	float emission_strength = 0.0f;

	Editor *m_editor = nullptr;
	kobra::TextureLoader *m_texture_loader = nullptr;

	vk::DescriptorSet imgui_allocate_image(const std::string &path) {
		const kobra::ImageData &image = m_texture_loader->load_texture(path);
		const vk::raii::Sampler &sampler = m_texture_loader->load_sampler(path);

		return ImGui_ImplVulkan_AddTexture(
			static_cast <VkSampler> (*sampler),
			static_cast <VkImageView> (*image.view),
			static_cast <VkImageLayout> (image.layout)
		);
	}
public:
	int material_index = -1;

	MaterialEditor() = delete;
	MaterialEditor(Editor *editor, kobra::TextureLoader *texture_loader)
			: m_editor {editor}, m_texture_loader {texture_loader} {}

	void render() override {
		ImGui::Begin("Material Editor");
		if (material_index < 0) {
			ImGui::End();
			return;
		}

		// Check if it is a new material
		bool is_not_loaded = m_prev_material_index != material_index;
		m_prev_material_index = material_index;

		// For starters, print material data
		ImGui::Text("Material data:");
		ImGui::Separator();

		kobra::Material *material = &kobra::Material::all[material_index];

		glm::vec3 diffuse = material->diffuse;
		glm::vec3 specular = material->specular;
		glm::vec3 ambient = material->ambient;
		float roughness = material->roughness;

		// Decompose the emission if it is not loaded
		if (is_not_loaded) {
			emission_base = glm::vec3(0.0f);
			emission_strength = 0.0f;

			// If any component is greater than 1, normalize it
			glm::vec3 emission = material->emission;
			if (emission.r > 1.0f || emission.g > 1.0f || emission.b > 1.0f) {
				emission_strength = glm::length(emission);
				emission_base = emission / emission_strength;
			}
		}

		bool updated_material = false;

		if (ImGui::ColorEdit3("Diffuse", &diffuse.r)) {
			material->diffuse = diffuse;
			updated_material = true;
		}

		if (ImGui::ColorEdit3("Specular", &specular.r)) {
			material->specular = specular;
			updated_material = true;
		}

		// TODO: remove ambient from material

		// TODO: use an HSL color picker + intensity slider
		if (ImGui::ColorEdit3("Emission", &emission_base.r)) {
			material->emission = emission_strength * emission_base;
			updated_material = true;
		}

		if (ImGui::SliderFloat("Intensity", &emission_strength, 0.0f, 1000.0f)) {
			material->emission = emission_strength * emission_base;
			updated_material = true;
		}

		// TODO: emission intensity

		if (ImGui::SliderFloat("Roughness", &roughness, 0.0f, 1.0f)) {
			material->roughness = std::max(roughness, 0.001f);
			updated_material = true;
		}

		// Transmission index of refraction
		if (ImGui::SliderFloat("IOR", &material->refraction, 1.0f, 3.0f))
			updated_material = true;

		// TODO: option for transmission
		bool transmission = (material->type == eTransmission);
		if (ImGui::Checkbox("Transmission", &transmission)) {
			material->type = transmission ? eTransmission : eDiffuse;
			updated_material = true;
		}

		ImGui::Separator();

		if (material->has_albedo()) {
			ImGui::Text("Diffuse Texture:");

			std::string diffuse_path = material->albedo_texture;
			if (is_not_loaded)
				m_diffuse_set = imgui_allocate_image(diffuse_path);

			ImGui::Image(m_diffuse_set, ImVec2(256, 256));
			ImGui::Separator();
		}

		if (material->has_normal()) {
			ImGui::Text("Normal Texture:");

			std::string normal_path = material->normal_texture;
			if (is_not_loaded)
				m_normal_set = imgui_allocate_image(normal_path);

			ImGui::Image(m_normal_set, ImVec2(256, 256));
			ImGui::Separator();
		}

		// Notify the daemon that the material has been updated
		if (updated_material) {
			kobra::Material::daemon.update(material_index);
			std::lock_guard <std::mutex> lock_guard
				(m_editor->m_renderers.movement_mutex);
			m_editor->m_renderers.movement.push(0);
		}

		ImGui::End();
	}
};

// RTX Renderer UI attachment
class RTXRenderer : public kobra::ui::ImGuiAttachment {
	Editor *m_editor = nullptr;
	int m_path_depth = 0;
public:
	RTXRenderer() = delete;
	RTXRenderer(Editor *editor) : m_editor {editor}, m_path_depth {2} {
		m_editor->m_renderers.armada_rtx->set_depth(m_path_depth);
	}

	void render() override {
		ImGui::Begin("RTX Renderer");

		// Setting the path depth
		if (ImGui::SliderInt("Path Depth", &m_path_depth, 0, 10)) {
			m_editor->m_renderers.armada_rtx->set_depth(m_path_depth);
			std::lock_guard <std::mutex> lock_guard
				(m_editor->m_renderers.movement_mutex);
			m_editor->m_renderers.movement.push(0);
		}

		// Checkboxes for enabling/disabling denoising
		ImGui::Checkbox("Denoise", &m_editor->m_renderers.denoise);

		// TODO: roussian roulette, different integrators, and loading
		// RTX attachments

		ImGui::End();
	}
};

// Viewport UI attachment
class Viewport : public kobra::ui::ImGuiAttachment {
	Editor *m_editor = nullptr;
	vk::DescriptorSet m_dset;
	vk::Image m_old_image = nullptr;
	float m_old_aspect = 0.0f;
public:
	Viewport() = delete;
	Viewport(Editor *editor) : m_editor {editor} {
		m_old_aspect = m_editor->m_camera.get <kobra::Camera> ().aspect;
	}

	void render() override {
		ImGui::Begin("Viewport");

		vk::Image image = *m_editor->m_viewport.image.image;
		if (image == m_old_image) {
			// Get current window size
			ImVec2 window_size = ImGui::GetWindowSize();

			// Pad
			constexpr float padding = 20.0f;
			window_size.x -= padding;
			window_size.y -= padding;

			// TODO: set the window aspect ratio
			ImGui::Image(m_dset, window_size);

			// Get pixel range of the image
			ImVec2 image_min = ImGui::GetItemRectMin();
			ImVec2 image_max = ImGui::GetItemRectMax();

			m_editor->m_input.viewport_focused = ImGui::IsWindowFocused();
			m_editor->m_input.viewport_hovered = ImGui::IsItemHovered();

			m_editor->m_viewport.min = image_min;
			m_editor->m_viewport.max = image_max;

			// Fix aspect ratio if needed
			float aspect = (image_max.x - image_min.x) /
				(image_max.y - image_min.y);

			if (fabs(aspect - m_old_aspect) > 1e-6) {
				m_editor->m_camera.get <kobra::Camera> ().aspect = aspect;
				m_old_aspect = aspect;
			}
		} else {
			m_dset = ImGui_ImplVulkan_AddTexture(
				static_cast <VkSampler>
				(*m_editor->m_viewport.sampler),

				static_cast <VkImageView>
				(*m_editor->m_viewport.image.view),

				static_cast <VkImageLayout>
				(vk::ImageLayout::eShaderReadOnlyOptimal)
			);
		}

		m_old_image = image;
		ImGui::End();
	}
};

// Editor implementation
Editor::Editor(const vk::raii::PhysicalDevice &phdev,
		const std::vector <const char *> &extensions)
		: kobra::BaseApp {
			phdev, "Stress Test",
			vk::Extent2D {1500, 1000},
			extensions
		}
{
	// TODO: constructor should be loaded very fast, everything else should
	// be loaded as needed...
	int MIP_LEVELS = 5;

	// Load environment map
	// TODO: load HDR...
	kobra::ImageData &environment_map = m_texture_loader
		.load_texture(KOBRA_DIR "/resources/skies/background_1.jpg");

	m_irradiance_computer = kobra::engine::IrradianceComputer(
		get_context(), environment_map,
		MIP_LEVELS, 128,
		"irradiance_maps"
	);

	KOBRA_LOG_FUNC(kobra::Log::WARN) << "Starting irradiance computations...\n";

	// Load all the layers
	m_forward_renderer = kobra::layers::ForwardRenderer(get_context());
	m_objectifier = kobra::layers::Objectifier(get_context());

	// Configure ImGui
	ImGui::CreateContext();
	ImGui_ImplGlfw_InitForVulkan(window.handle, true);

	ImGuiIO &imgui_io = ImGui::GetIO();
	imgui_io.ConfigFlags |= ImGuiConfigFlags_DockingEnable;
	imgui_io.ConfigWindowsMoveFromTitleBarOnly = true;

	auto font = std::make_pair(KOBRA_DIR "/resources/fonts/NotoSans.ttf", 18);
	m_ui = std::make_shared <kobra::layers::UI> (
		get_context(), window,
		graphics_queue, font,
		vk::AttachmentLoadOp::eClear
	);

	// Load scene
	kobra::Project project = kobra::Project::load(".kobra/project");
	m_scene.load(get_context(), project.scene);

	// TODO: Create a camera somewhere outside...
	// plus icons for lights and cameras
	m_camera = m_scene.ecs.get_entity("Camera");
	m_camera.get <kobra::Camera> ().aspect = 1.5f;

	// IO callbacks
	io.mouse_events.subscribe(mouse_callback, this);
	io.keyboard_events.subscribe(keyboard_callback, this);

	/* Create the image viewer
	std::vector <const kobra::ImageData *> images;
	for (int i = 0; i < MIP_LEVELS; i++)
		images.emplace_back(m_irradiance_computer.irradiance_maps[i]); */

	// TODO: irradiance computer load from cache...

	// Configure the forward renderer
	m_forward_renderer.add_pipeline(
		"environment",
		KOBRA_DIR "/source/shaders/environment_lighter.frag",
		{
			kobra::DescriptorSetLayoutBinding {
				5, vk::DescriptorType::eCombinedImageSampler,
				5, vk::ShaderStageFlagBits::eFragment
			}
		},
		[&](const vk::raii::DescriptorSet &descriptor_set) {
			m_irradiance_computer.bind(device, descriptor_set, 5);
		}
	);

	// Load all the renderers
	m_renderers.system = std::make_shared <kobra::amadeus::System> ();
	m_renderers.mesh_memory = std::make_shared <kobra::layers::MeshMemory> (get_context());

	constexpr vk::Extent2D raytracing_extent = {1000, 1000};
	m_renderers.armada_rtx = std::make_shared <kobra::amadeus::ArmadaRTX> (
		get_context(), m_renderers.system,
		m_renderers.mesh_memory, raytracing_extent
	);

	m_renderers.armada_rtx->attach(
		"Path Tracer",
		std::make_shared <kobra::amadeus::PathTracer> ()
	);

	m_renderers.armada_rtx->set_envmap(KOBRA_DIR "/resources/skies/background_1.jpg");

	// Create the denoiser layer
	m_renderers.denoiser = kobra::layers::Denoiser::make(
		raytracing_extent,
		kobra::layers::Denoiser::eNone
		// kobra::layers::Denoiser::eNormal
		//	| kobra::layers::Denoiser::eAlbedo
	);

	m_renderers.framer = kobra::layers::Framer(get_context());

	// Allocate necessary buffers
	size_t size = m_renderers.armada_rtx->size();
	m_buffers.traced = kobra::cuda::alloc(size * sizeof(uint32_t));
	m_buffers.traced_cpu.resize(size);

	// Allocate the viewport resources
	m_viewport.image = kobra::ImageData(
		phdev, device,
		swapchain.format, window.extent,
		vk::ImageTiling::eOptimal,
		vk::ImageUsageFlagBits::eColorAttachment
			| vk::ImageUsageFlagBits::eSampled
			| vk::ImageUsageFlagBits::eTransferDst,
		vk::MemoryPropertyFlagBits::eDeviceLocal,
		vk::ImageAspectFlagBits::eColor
	);

	m_viewport.sampler = kobra::make_sampler(device, m_viewport.image);

	// Attach UI layers
	m_progress_bar = std::make_shared <ProgressBar> ("Irradiance Computation Progress");
	m_info_tab = std::make_shared <InfoTab> ();
	m_material_editor = std::make_shared <MaterialEditor> (this, &m_texture_loader);

	// m_ui->attach(m_image_viewer);
	m_ui->attach(m_progress_bar);
	m_ui->attach(m_info_tab);
	m_ui->attach(m_material_editor);
	m_ui->attach(std::make_shared <RTXRenderer> (this));
	m_ui->attach(std::make_shared <Viewport> (this));
}

Editor::~Editor()
{
	device.waitIdle();

	// TODO: method for total destruction
	ImGui_ImplVulkan_Shutdown();
	ImGui_ImplGlfw_Shutdown();
	ImGui::DestroyContext();
}

void Editor::record(const vk::raii::CommandBuffer &cmd,
		const vk::raii::Framebuffer &framebuffer)
{
	// Camera movement
	if (m_input.viewport_focused) {
		auto &transform = m_camera.get <kobra::Transform> ();

		float speed = 20.0f * frame_time;

		glm::vec3 forward = transform.forward();
		glm::vec3 right = transform.right();
		glm::vec3 up = transform.up();

		bool moved = false;
		if (io.input->is_key_down(GLFW_KEY_W)) {
			transform.move(forward * speed);
			moved = true;
		} else if (io.input->is_key_down(GLFW_KEY_S)) {
			transform.move(-forward * speed);
			moved = true;
		}

		if (io.input->is_key_down(GLFW_KEY_A)) {
			transform.move(-right * speed);
			moved = true;
		} else if (io.input->is_key_down(GLFW_KEY_D)) {
			transform.move(right * speed);
			moved = true;
		}

		if (io.input->is_key_down(GLFW_KEY_E)) {
			transform.move(up * speed);
			moved = true;
		} else if (io.input->is_key_down(GLFW_KEY_Q)) {
			transform.move(-up * speed);
			moved = true;
		}

		if (moved) {
			std::lock_guard <std::mutex> lock(m_renderers.movement_mutex);
			m_renderers.movement.push(0);
		}
	}

	std::vector <const kobra::Renderable *> renderables;
	std::vector <const kobra::Transform *> renderable_transforms;

	std::vector <const kobra::Light *> lights;
	std::vector <const kobra::Transform *> light_transforms;

	auto renderables_transforms = m_scene.ecs.tuples <kobra::Renderable, kobra::Transform> ();
	auto lights_transforms = m_scene.ecs.tuples <kobra::Light, kobra::Transform> ();

	auto ecs = m_scene.ecs;

	for (int i = 0; i < ecs.size(); i++) {
		if (ecs.exists <kobra::Renderable> (i)) {
			const auto *renderable = &ecs.get <kobra::Renderable> (i);
			const auto *transform = &ecs.get <kobra::Transform> (i);

			renderables.push_back(renderable);
			renderable_transforms.push_back(transform);
		}

		if (ecs.exists <kobra::Light> (i)) {
			const auto *light = &ecs.get <kobra::Light> (i);
			const auto *transform = &ecs.get <kobra::Transform> (i);

			lights.push_back(light);
			light_transforms.push_back(transform);
		}
	}

	kobra::layers::ForwardRenderer::Parameters params {
		.renderables = renderables_transforms,
		.lights = lights_transforms,
		.pipeline_package = "environment",
	};

	params.environment_map = KOBRA_DIR "/resources/skies/background_1.jpg";

	cmd.begin({});
		// TODO: also see the normal and albedo and depth buffers from
		// deferred renderer
		// TODO: drop down menu for selecting the renderer
		if (m_renderers.mode) {
			bool accumulate = m_renderers.movement.empty();

			{
				// Clear queue
				std::lock_guard <std::mutex> lock(m_renderers.movement_mutex);
				m_renderers.movement = std::queue <uint32_t> ();
			}

			m_renderers.armada_rtx->render(
				m_scene.ecs,
				m_camera.get <kobra::Camera> (),
				m_camera.get <kobra::Transform> (),
				accumulate
			);

			float4 *buffer = (float4 *) m_renderers.armada_rtx->color_buffer();
			if (m_renderers.denoise) {
				kobra::layers::denoise(m_renderers.denoiser, {
					.color = (hipDeviceptr_t) m_renderers.armada_rtx->color_buffer(),
					.normal = (hipDeviceptr_t) m_renderers.armada_rtx->normal_buffer(),
					.albedo = (hipDeviceptr_t) m_renderers.armada_rtx->albedo_buffer()
				});

				buffer = (float4 *) m_renderers.denoiser.result;
			}

			vk::Extent2D rtx_extent = m_renderers.armada_rtx->extent();

			kobra::cuda::hdr_to_ldr(
				buffer,
				(uint32_t *) m_buffers.traced,
				rtx_extent.width, rtx_extent.height,
				kobra::cuda::eTonemappingACES
			);

			kobra::cuda::copy(
				m_buffers.traced_cpu, m_buffers.traced,
				m_renderers.armada_rtx->size() * sizeof(uint32_t)
			);

			// TODO: import CUDA to Vulkan and render straight to the image
			m_renderers.framer.render(
				kobra::RawImage {
					.data = m_buffers.traced_cpu,
					.width = rtx_extent.width,
					.height = rtx_extent.height,
					.channels = 4
				},
				cmd, framebuffer, window.extent
			);
		} else {
			m_forward_renderer.render(
				params,
				m_camera.get <kobra::Camera> (),
				m_camera.get <kobra::Transform> (),
				cmd, framebuffer, window.extent
			);
		}

		m_irradiance_computer.sample(cmd);
		/* if (m_irradiance_computer.sample(cmd)
				&& !m_irradiance_computer.cached
				&& !m_saved_irradiance) {
			m_irradiance_computer.save_irradiance_maps(
				get_context(),
				"irradiance_maps"
			);

			m_saved_irradiance = true;
		} */

		// TODO: progress bar...
		// std::cout << "Sample count: " << m_irradiance_computer.samples << std::endl;
		m_progress_bar->m_progress = m_irradiance_computer.samples/128.0f;

		// Handle requests
		std::optional <Request> selection_request;
		while (!request_queue.empty()) {
			Request request = request_queue.front();
			request_queue.pop();

			selection_request = request;
		}

		if (selection_request) {
			m_objectifier.render(
				cmd,
				// TODO: pass extent...
				m_scene.ecs,
				m_camera.get <kobra::Camera> (),
				m_camera.get <kobra::Transform> ()
			);

			request_queue.push(*selection_request);
		}

		// If there is a selection, highlight it
		if (m_selection.first >= 0 && m_selection.second >= 0) {
			// TODO: only render the selected objetcs...
			// otherwise computtion becomes very wasted...
			m_objectifier.composite_highlight(
				cmd, framebuffer, window.extent,
				m_scene.ecs,
				m_camera.get <kobra::Camera> (),
				m_camera.get <kobra::Transform> (),
				m_selection
			);
		}

		// Copy framebuffer image to viewport image
		vk::ImageCopy copy_region {
			{vk::ImageAspectFlagBits::eColor, 0, 0, 1},
			{0, 0, 0},
			{vk::ImageAspectFlagBits::eColor, 0, 0, 1},
			{0, 0, 0},
			{window.extent.width, window.extent.height, 1}
		};

		// Transition layouts
		vk::ImageMemoryBarrier swapchain_barrier {
			{},
			vk::AccessFlagBits::eTransferWrite,
			vk::ImageLayout::ePresentSrcKHR,
			vk::ImageLayout::eTransferSrcOptimal,
			VK_QUEUE_FAMILY_IGNORED,
			VK_QUEUE_FAMILY_IGNORED,
			swapchain.images[frame_index],
			{vk::ImageAspectFlagBits::eColor, 0, 1, 0, 1}
		};

		vk::ImageMemoryBarrier viewport_barrier {
			{},
			vk::AccessFlagBits::eTransferWrite,
			m_viewport.image.layout,
			vk::ImageLayout::eTransferDstOptimal,
			VK_QUEUE_FAMILY_IGNORED,
			VK_QUEUE_FAMILY_IGNORED,
			*m_viewport.image.image,
			{vk::ImageAspectFlagBits::eColor, 0, 1, 0, 1}
		};

		cmd.pipelineBarrier(
			vk::PipelineStageFlagBits::eTopOfPipe,
			vk::PipelineStageFlagBits::eTransfer,
			{}, {}, {},
			{swapchain_barrier, viewport_barrier}
		);

		cmd.copyImage(
			swapchain.images[frame_index],
			vk::ImageLayout::eTransferSrcOptimal,
			*m_viewport.image.image,
			vk::ImageLayout::eTransferDstOptimal,
			copy_region
		);

		// Transition layouts back
		swapchain_barrier.srcAccessMask = vk::AccessFlagBits::eTransferWrite;
		swapchain_barrier.dstAccessMask = vk::AccessFlagBits::eMemoryRead;
		swapchain_barrier.oldLayout = vk::ImageLayout::eTransferSrcOptimal;
		swapchain_barrier.newLayout = vk::ImageLayout::ePresentSrcKHR;

		viewport_barrier.srcAccessMask = vk::AccessFlagBits::eTransferWrite;
		viewport_barrier.dstAccessMask = vk::AccessFlagBits::eShaderRead;
		viewport_barrier.oldLayout = vk::ImageLayout::eTransferDstOptimal;
		viewport_barrier.newLayout = vk::ImageLayout::eShaderReadOnlyOptimal;

		cmd.pipelineBarrier(
			vk::PipelineStageFlagBits::eTransfer,
			vk::PipelineStageFlagBits::eFragmentShader,
			{}, {}, {},
			{swapchain_barrier, viewport_barrier}
		);

		// Render the UI last
		m_ui->render(cmd,
			framebuffer, window.extent,
			kobra::RenderArea::full(), {true}
		);
	cmd.end();

	// TODO: after present actions...
}

void Editor::resize(const vk::Extent2D &extent)
{
	// m_camera.get <kobra::Camera> ().aspect = extent.width / (float) extent.height;
	// TODO: resize the objectifier...

	// Resize the viewport image
	// m_viewport.image.resize(extent); TODO: implement this

	m_viewport.image = kobra::ImageData(
		phdev, device,
		swapchain.format, extent,
		vk::ImageTiling::eOptimal,
		vk::ImageUsageFlagBits::eColorAttachment
			| vk::ImageUsageFlagBits::eSampled
			| vk::ImageUsageFlagBits::eTransferDst,
		vk::MemoryPropertyFlagBits::eDeviceLocal,
		vk::ImageAspectFlagBits::eColor
	);

	m_viewport.sampler = kobra::make_sampler(device, m_viewport.image);
}

void Editor::after_present()
{
	if (!request_queue.empty()) {
		// TODO: ideally should only be one type of request per after_present
		Request request = request_queue.front();
		request_queue.pop();

		ImVec2 min = m_viewport.min;
		ImVec2 max = m_viewport.max;

		ImVec2 fixed {
			(request.x - min.x) / (max.x - min.x),
			(request.y - min.y) / (max.y - min.y)
		};

		fixed.x *= window.extent.width;
		fixed.y *= window.extent.height;

		// TODO: get coordinates of the viewport image...
		auto ids = m_objectifier.query(fixed.x, fixed.y);
		m_selection = {int(ids.first) - 1, int(ids.second) - 1};

		// Update the material editor
		if (m_selection.first < 0 || m_selection.second < 0) {
			m_material_editor->material_index = -1;
		} else {
			kobra::Renderable &renderable = m_scene.ecs
				.get <kobra::Renderable> (m_selection.first);

			uint32_t material_index = renderable.material_indices[m_selection.second];
			m_material_editor->material_index = material_index;
		}
	}

	// Ping all systems using materials
	kobra::Material::daemon.ping_all();
}

void Editor::mouse_callback(void *us, const kobra::io::MouseEvent &event)
{
	static const int select_button = GLFW_MOUSE_BUTTON_LEFT;

	// Check if selecting
	if (event.action == GLFW_PRESS && event.button == select_button) {
		Editor *editor = static_cast <Editor *> (us);
		// TODO: this needs to query the position on the viewport
		// image...
		editor->request_queue.push({
			float(event.xpos),
			float(event.ypos)
		});
	}

	// Panning around
	static const int pan_button = GLFW_MOUSE_BUTTON_RIGHT;

	static const float sensitivity = 0.001f;

	static float px = 0.0f;
	static float py = 0.0f;

	static float yaw = 0.0f;
	static float pitch = 0.0f;

	// Deltas and directions
	float dx = event.xpos - px;
	float dy = event.ypos - py;

	// Check if panning
	static bool dragging = false;
	static bool alt_dragging = false;

	Editor *editor = static_cast <Editor *> (us);
	bool is_drag_button = (event.button == pan_button);
	if (event.action == GLFW_PRESS && is_drag_button && editor->m_input.viewport_hovered) {
		dragging = true;
		glfwSetInputMode(editor->window.handle, GLFW_CURSOR, GLFW_CURSOR_DISABLED);
	} else if (event.action == GLFW_RELEASE && is_drag_button && !editor->m_input.viewport_hovered) {
		dragging = false;
		glfwSetInputMode(editor->window.handle, GLFW_CURSOR, GLFW_CURSOR_NORMAL);
	}

	bool is_alt_down = editor->io.input->is_key_down(GLFW_KEY_LEFT_ALT);
	if (!alt_dragging && is_alt_down && editor->m_input.viewport_hovered) {
		alt_dragging = true;
		glfwSetInputMode(editor->window.handle, GLFW_CURSOR, GLFW_CURSOR_DISABLED);
	} else if (alt_dragging && !is_alt_down && !editor->m_input.viewport_hovered) {
		alt_dragging = false;
		glfwSetInputMode(editor->window.handle, GLFW_CURSOR, GLFW_CURSOR_NORMAL);
	}

	// Pan only when dragging
	if (dragging | alt_dragging) {
		yaw -= dx * sensitivity;
		pitch -= dy * sensitivity;

		if (pitch > 89.0f)
			pitch = 89.0f;
		if (pitch < -89.0f)
			pitch = -89.0f;

		kobra::Transform &transform = editor->m_camera.get <kobra::Transform> ();
		transform.rotation.x = pitch;
		transform.rotation.y = yaw;

		std::lock_guard <std::mutex> lock(editor->m_renderers.movement_mutex);
		editor->m_renderers.movement.push(0);
	}

	// Update previous position
	px = event.xpos;
	py = event.ypos;
}

void Editor::keyboard_callback(void *us, const kobra::io::KeyboardEvent &event)
{
	Editor *editor = static_cast <Editor *> (us);
	if (event.action == GLFW_PRESS) {
		if (event.key == GLFW_KEY_TAB)
			editor->m_renderers.mode = !editor->m_renderers.mode;
		if (event.key == GLFW_KEY_ESCAPE) {
			editor->m_selection = {-1, -1};
			editor->m_material_editor->material_index = -1;
		}
	}
}
