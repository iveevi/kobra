#include "hip/hip_runtime.h"
// OptiX headers
#include <optix_device.h>
#include <optix_host.h>
#include <optix_stack_size.h>

// ImGUI headers
#include <imgui.h>

// Engine headers
#include "../../include/camera.hpp"
#include "../../include/cuda/alloc.cuh"
#include "../../include/cuda/cast.cuh"
#include "../../include/cuda/color.cuh"
#include "../../include/cuda/interop.cuh"
#include "../../include/ecs.hpp"
#include "../../include/amadeus/armada.cuh"
#include "../../include/optix/core.cuh"
#include "../../include/transform.hpp"
#include "../../shaders/raster/bindings.h"
#include "../../include/profiler.hpp"

namespace kobra {

namespace amadeus {

// Create the layer
// TODO: all custom extent...
ArmadaRTX::ArmadaRTX(const Context &context,
		const std::shared_ptr <amadeus::System> &system,
		const std::shared_ptr <layers::MeshMemory> &mesh_memory,
		const vk::Extent2D &extent)
		: m_system(system), m_mesh_memory(mesh_memory),
		m_device(context.device), m_phdev(context.phdev),
		m_texture_loader(context.texture_loader),
		m_extent(extent), m_active_attachment()
{
	// Start the timer
	m_timer.start();

	// Initialize the host state
	m_host.last_updated = 0;

	// Initialize TLAS state
	m_tlas.null = true;
	m_tlas.last_updated = 0;

	// Configure launch parameters
	auto &params = m_launch_info;

	params.resolution = {
		extent.width,
		extent.height
	};

	params.max_depth = 10;
	params.samples = 0;
	params.accumulate = true;
	params.lights.quad_lights = nullptr;
	params.lights.quad_count = 0;
	params.lights.tri_lights = nullptr;
	params.lights.tri_count = 0;
	params.materials = nullptr;
	params.environment_map = 0;
	params.has_environment_map = false;

	// Allocate results
	int size = extent.width * extent.height;

	params.buffers.color = cuda::alloc <glm::vec4> (size);
	params.buffers.normal = cuda::alloc <glm::vec4> (size);
	params.buffers.albedo = cuda::alloc <glm::vec4> (size);
	params.buffers.position = cuda::alloc <glm::vec4> (size);

	// Add self to the material system ping list
	Material::daemon.ping_at(this,
		[](void *user, const std::set <uint32_t> &materials) {
			ArmadaRTX *armada = (ArmadaRTX *) user;
			armada->update_materials(materials);
		}
	);
}

// Set the environment map
void ArmadaRTX::set_envmap(const std::string &path)
{
	// First load the environment map
	const auto &map = m_texture_loader->load_texture(path);
	m_launch_info.environment_map = cuda::import_vulkan_texture(*m_device, map);
	m_launch_info.has_environment_map = true;
}

void ArmadaRTX::update_triangle_light_buffers
		(const std::set <_instance_ref> &emissive_submeshes_to_update)
{
	// TODO: share this setup with the renderables (another layer for
	// material buffer updates? or use the same daemon?)
	if (m_host.tri_lights.size() != m_host.emissive_count) {
		if (m_launch_info.lights.tri_lights) {
			// TODO: free this buffer only when rendering is
			// complete...
			cuda::free(m_launch_info.lights.tri_lights);
			m_launch_info.lights.tri_lights = nullptr;
			m_launch_info.lights.tri_count = 0;
		}

		m_host.tri_lights.clear();
		m_host.emissive_submesh_offsets.clear();

		if (m_host.emissive_count <= 0)
			return;

		for (const auto &pr : m_host.emissive_submeshes) {
			const Submesh *submesh = pr.submesh;
			const Transform *transform = pr.transform;

			const Material &material = Material::all[submesh->material_index];

			m_host.emissive_submesh_offsets[submesh] = m_host.tri_lights.size();
			for (int i = 0; i < submesh->triangles(); i++) {
				uint32_t i0 = submesh->indices[i * 3 + 0];
				uint32_t i1 = submesh->indices[i * 3 + 1];
				uint32_t i2 = submesh->indices[i * 3 + 2];

				glm::vec3 a = transform->apply(submesh->vertices[i0].position);
				glm::vec3 b = transform->apply(submesh->vertices[i1].position);
				glm::vec3 c = transform->apply(submesh->vertices[i2].position);

				// TODO: cache cuda textures
				m_host.tri_lights.push_back(
					optix::TriangleLight {
						cuda::to_f3(a),
						cuda::to_f3(b - a),
						cuda::to_f3(c - a),
						cuda::to_f3(material.emission)
						// TODO: what if material has
						// textured emission?
					}
				);
			}
		}

		std::cout << "# of triangle lights: " << m_host.tri_lights.size() << "\n";

		m_launch_info.lights.tri_lights = cuda::make_buffer(m_host.tri_lights);
		m_launch_info.lights.tri_count = m_host.tri_lights.size();

		// TODO: display logging in UI as well (add log routing)
		KOBRA_LOG_FUNC(Log::INFO) << "Uploaded " << m_host.tri_lights.size()
			<< " triangle lights to the GPU\n";
	} else if (emissive_submeshes_to_update.size() > 0) {
		size_t subrange_min = std::numeric_limits <size_t>::max();
		size_t subrange_max = 0;

		for (const auto &pr : emissive_submeshes_to_update) {
			const Submesh *submesh = pr.submesh;
			const Transform *transform = pr.transform;

			const Material &material = Material::all[submesh->material_index];

			size_t offset = m_host.emissive_submesh_offsets[submesh];
			for (int i = 0; i < submesh->triangles(); i++) {
				// TODO: check if transforms have changed; if so
				// then also update the triangle light position
				m_host.tri_lights[offset + i].intensity =
					cuda::to_f3(material.emission);
			}

			subrange_min = std::min(subrange_min, offset);
			subrange_max = std::max(subrange_max, offset + submesh->triangles());
		}

		hipMemcpy(
			&m_launch_info.lights.tri_lights[subrange_min],
			&m_host.tri_lights[subrange_min],
			(subrange_max - subrange_min) * sizeof(optix::TriangleLight),
			hipMemcpyHostToDevice
		);
	}
}

// Update the light buffers if needed
void ArmadaRTX::update_quad_light_buffers
		(const std::vector <const Light *> &lights,
		const std::vector <const Transform *> &light_transforms)
{
	// TODO: lighting system equivalent of System
	if (m_host.quad_lights.size() != lights.size()) {
		if (m_launch_info.lights.quad_lights)
			cuda::free(m_launch_info.lights.quad_lights);

		m_host.quad_lights.resize(lights.size());

		auto &quad_lights = m_host.quad_lights;
		for (int i = 0; i < lights.size(); i++) {
			const Light *light = lights[i];
			const Transform *transform = light_transforms[i];

			glm::vec3 a {-0.5f, 0, -0.5f};
			glm::vec3 b {0.5f, 0, -0.5f};
			glm::vec3 c {-0.5f, 0, 0.5f};

			a = transform->apply(a);
			b = transform->apply(b);
			c = transform->apply(c);

			quad_lights[i].a = cuda::to_f3(a);
			quad_lights[i].ab = cuda::to_f3(b - a);
			quad_lights[i].ac = cuda::to_f3(c - a);
			quad_lights[i].intensity = cuda::to_f3(light->power * light->color);
		}

		m_launch_info.lights.quad_lights = cuda::make_buffer(quad_lights);
		m_launch_info.lights.quad_count = quad_lights.size();

		KOBRA_LOG_FUNC(Log::INFO) << "Uploaded " << quad_lights.size()
			<< " quad lights to the GPU\n";
	}
}

// Update the SBT data
void ArmadaRTX::update_sbt_data
		(const std::vector <layers::MeshMemory::Cachelet> &cachelets,
		const std::vector <const Submesh *> &submeshes,
		const std::vector <const Transform *> &submesh_transforms)
{
	int submesh_count = submeshes.size();

	m_host.hit_records.clear();
	for (int i = 0; i < submesh_count; i++) {
		const Submesh *submesh = submeshes[i];
		const Material &mat = Material::all[submesh->material_index];

		HitRecord hit_record {};

                // TODO: use ecs indirection...
		hit_record.data.model = submesh_transforms[i]->matrix();
		hit_record.data.material_index = submesh->material_index;

		hit_record.data.triangles = cachelets[i].m_cuda_triangles;
		hit_record.data.vertices = cachelets[i].m_cuda_vertices;

		// If the material is emissive, then we need to
		//	give a valid light index
		hit_record.data.light_index = -1;
		if (glm::length(mat.emission) > 0.0f) {
			hit_record.data.light_index =
				m_host.emissive_submesh_offsets[submesh];
		}

		// Push back
		m_host.hit_records.push_back(hit_record);
	}
}

void ArmadaRTX::update_materials(const std::set <uint32_t> &material_indices)
{
	// If host buffer is empty, assume the armada is not initialized
	if (m_host.materials.size() == 0)
		return;

	std::set <_instance_ref> emissive_submeshes_to_update;
	for (uint32_t mat_index : material_indices) {
		const Material &material = Material::all[mat_index];
		cuda::_material &mat = m_host.materials[mat_index];

		bool was_emissive = (length(mat.emission) > 0.0f)
				|| mat.textures.has_emission;

		// Copy basic data
		mat.diffuse = cuda::to_f3(material.diffuse);
		mat.specular = cuda::to_f3(material.specular);
		mat.emission = cuda::to_f3(material.emission);
		mat.ambient = cuda::to_f3(material.ambient);
		mat.shininess = material.shininess;
		mat.roughness = material.roughness;
		mat.refraction = material.refraction;
		mat.type = material.type;

		bool is_emissive = (length(mat.emission) > 0.0f)
				|| mat.textures.has_emission;

		// TODO: textures

		const auto &refs = m_host.material_submeshes[mat_index];

		// TODO: check previous state (to see whether to remove from
		// emissive submeshes)
		if (is_emissive) {
			for (const auto &pr : refs) {
				emissive_submeshes_to_update.insert(pr);
				if (m_host.emissive_submeshes.find(pr) !=
						m_host.emissive_submeshes.end())
					continue;

				m_host.emissive_submeshes.insert(pr);
				m_host.emissive_count += pr.submesh->triangles();
			}
		} else if (was_emissive && !is_emissive) {
			// Remove from emissive submeshes
			for (const auto &pr : refs) {
				m_host.emissive_submeshes.erase(pr);
				m_host.emissive_count -= pr.submesh->triangles();
			}
		}

		// TODO: what if the net change is 0 (with multiple material
		// emission changes)?
	}

	// Copy to GPU
	// TODO: only copy subregions
	hipMemcpy(m_launch_info.materials,
		m_host.materials.data(),
		m_host.materials.size() * sizeof(cuda::_material),
		hipMemcpyHostToDevice
	);

	bool sbt_needs_update = (m_host.tri_lights.size() != m_host.emissive_count);

	// Also update the emissive submeshes if needed
	// TODO: use the reutrn from tihs instead to check for sbt udpate...
	update_triangle_light_buffers(emissive_submeshes_to_update);

	// Update the SBT if needed (e.g. when a new emissive submesh is added)
	if (sbt_needs_update) {
		update_sbt_data(
			m_host.cachelets,
			m_host.submeshes,
			m_host.submesh_transforms
		);

		m_host.last_updated = clock();
	}
}

// Preprocess scene data
// TODO: get rid of this method..
ArmadaRTX::preprocess_update ArmadaRTX::preprocess_scene
		(const ECS &ecs,
                const daemons::Transform &transform_daemon,
		const Camera &camera,
		const Transform &transform)
{
	// To return
	std::optional <OptixTraversableHandle> handle;
	std::vector <HitRecord> *hit_records = nullptr;

	// Set viewing position
	m_launch_info.camera.center = transform.position;

	auto uvw = kobra::uvw_frame(camera, transform);

	m_launch_info.camera.ax_u = uvw.u;
	m_launch_info.camera.ax_v = uvw.v;
	m_launch_info.camera.ax_w = uvw.w;

	m_launch_info.camera.projection = camera.perspective_matrix();
	m_launch_info.camera.view = camera.view_matrix(transform);

	// Get time
	m_launch_info.time = m_timer.elapsed_start();

	// Update the raytracing system
	bool updated = m_system->update(ecs);

	// Preprocess the entities
        // TODO: helper method for this... (tuples)
        std::vector <int> renderable_id;
	std::vector <const Renderable *> renderables;
	std::vector <const Transform *> renderable_transforms;

	std::vector <const Light *> lights;
	std::vector <const Transform *> light_transforms;

	for (int i = 0; i < ecs.size(); i++) {
		// TODO: one unifying renderer component, with options for
		// raytracing, etc
		if (ecs.exists <Renderable> (i)) {
			const auto *renderable = &ecs.get <Renderable> (i);
			const auto *transform = &ecs.get <Transform> (i);

                        renderable_id.push_back(i);
			renderables.push_back(renderable);
			renderable_transforms.push_back(transform);
		}

		if (ecs.exists <Light> (i)) {
			const auto *light = &ecs.get <Light> (i);
			const auto *transform = &ecs.get <Transform> (i);

			lights.push_back(light);
			light_transforms.push_back(transform);
		}
	}

	// Update data if necessary
	if (m_tlas.null) {
		/* Load the list of all submeshes
		std::vector <layers::MeshMemory::Cachelet> cachelets; // TODO: redo this method...
		std::vector <const Submesh *> submeshes;
		std::vector <const Transform *> submesh_transforms; */

		m_host.cachelets.clear();
		m_host.submesh_transforms.clear();
		m_host.submeshes.clear();

		// Reserve material-submesh reference structure
		m_host.material_submeshes.clear();
		m_host.material_submeshes.resize(Material::all.size());

		for (int i = 0; i < renderables.size(); i++) {
                        int id = renderable_id[i];
			const Renderable *renderable = renderables[i];
			const Transform *transform = renderable_transforms[i];

			// Cache the renderables
			// TODO: all update functions should go to a separate methods
			m_mesh_memory->cache_cuda(renderable);

			for (int j = 0; j < renderable->mesh->submeshes.size(); j++) {
				const Submesh *submesh = &renderable->mesh->submeshes[j];
				uint32_t material_index = submesh->material_index;
				m_host.material_submeshes[material_index].insert(
					{transform, submesh, id}
				);

				m_host.cachelets.push_back(m_mesh_memory->get(renderable, j));

                                // TODO: use instance ref vector instead...
                                m_host.entity_id.push_back(i);
				m_host.submeshes.push_back(submesh);
				m_host.submesh_transforms.push_back(transform);
			}
		}

		// Count number of emissive submeshes
		m_host.emissive_count = 0;

		// TODO: compute before hand
		for (int i = 0; i < m_host.submeshes.size(); i++) {
                        int id = m_host.entity_id[i];
			const Submesh *submesh = m_host.submeshes[i];
			const Transform *transform = m_host.submesh_transforms[i];

			const Material &material = Material::all[submesh->material_index];
			if (glm::length(material.emission) > 0
					|| material.has_emission()) {
				_instance_ref ref {transform, submesh, id};
				m_host.emissive_submeshes.insert(ref);
				m_host.emissive_count += submesh->triangles();
			}
		}

		// Update the data
		update_triangle_light_buffers({});
		update_quad_light_buffers(lights, light_transforms);
		update_sbt_data(
			m_host.cachelets,
			m_host.submeshes,
			m_host.submesh_transforms
		);

		// hit_records = &m_host.hit_records;
		m_host.last_updated = clock();

		// Reset the number of samples stored
		m_launch_info.samples = 0;

		/* Update TLAS state
		m_tlas.null = false;
		m_tlas.last_updated = clock(); */

		// Update the status
		updated |= true;
	}

        // If needed, build the TLAS
        if (updated) {
		m_tlas.null = false;
		m_tlas.last_updated = clock();
                m_tlas.handle = m_system->build_tlas(
			renderables,
			m_attachments[m_previous_attachment]->m_hit_group_count
		);
        }

	// Generate material buffer if needed
	if (!m_launch_info.materials) {
		std::cout << "Generating material buffer" << std::endl;

		m_host.materials.clear();
		for (const Material &material : Material::all) {
			cuda::_material mat;

			// Scalar/vector values
			mat.diffuse = cuda::to_f3(material.diffuse);
			mat.specular = cuda::to_f3(material.specular);
			mat.emission = cuda::to_f3(material.emission);
			mat.ambient = cuda::to_f3(material.ambient);
			mat.shininess = material.shininess;
			mat.roughness = material.roughness;
			mat.refraction = material.refraction;
			mat.type = material.type;

			// Textures
			if (material.has_albedo()) {
				const ImageData &diffuse = m_texture_loader
					->load_texture(material.albedo_texture);

				mat.textures.diffuse
					= cuda::import_vulkan_texture(*m_device, diffuse);
				mat.textures.has_diffuse = true;
			}

			if (material.has_normal()) {
				const ImageData &normal = m_texture_loader
					->load_texture(material.normal_texture);

				mat.textures.normal
					= cuda::import_vulkan_texture(*m_device, normal);
				mat.textures.has_normal = true;
			}

			if (material.has_specular()) {
				const ImageData &specular = m_texture_loader
					->load_texture(material.specular_texture);

				mat.textures.specular
					= cuda::import_vulkan_texture(*m_device, specular);
				mat.textures.has_specular = true;
			}

			if (material.has_emission()) {
				const ImageData &emission = m_texture_loader
					->load_texture(material.emission_texture);

				mat.textures.emission
					= cuda::import_vulkan_texture(*m_device, emission);
				mat.textures.has_emission = true;
			}

			if (material.has_roughness()) {
				const ImageData &roughness = m_texture_loader
					->load_texture(material.roughness_texture);

				mat.textures.roughness
					= cuda::import_vulkan_texture(*m_device, roughness);
				mat.textures.has_roughness = true;
			}

			m_host.materials.push_back(mat);
		}

		m_launch_info.materials = cuda::make_buffer(m_host.materials);
	}

        // Update triangle lights that have moved
        std::set <_instance_ref> emissive_to_update;
        for (auto ref : m_host.emissive_submeshes) {
                if (transform_daemon[ref.id])
                        emissive_to_update.insert(ref);
        }

        if (emissive_to_update.size() > 0)
                update_triangle_light_buffers(emissive_to_update);

        // Update host SBT data
        if (transform_daemon.size() > 0) {
                for (int i = 0; i < m_host.submeshes.size(); i++) {
                        int id = m_host.entity_id[i];
                        if (transform_daemon[id]) {
                                m_host.hit_records[i].data.model = m_host.submesh_transforms[i]->matrix();
                                m_host.last_updated = clock();
                        }
                }
        }

	// Send hit records to attachment if needed
	long long int attachment_time = m_host.times[m_previous_attachment];
	if (attachment_time < m_host.last_updated) {
		// Send the hit records
		hit_records = &m_host.hit_records;
		m_host.times[m_previous_attachment] = m_host.last_updated;
	}

	// Create acceleration structure for the attachment if needed
	// assuming that there is currently a valid attachment
	attachment_time = m_tlas.times[m_previous_attachment];
	if (attachment_time < m_tlas.last_updated) {
		// Create the acceleration structure
		m_tlas.times[m_previous_attachment] = m_tlas.last_updated;
                handle = m_tlas.handle;
	}

	return {handle, hit_records};
}

// Path tracing computation
void ArmadaRTX::render
                (const ECS &ecs,
                const daemons::Transform &transform_daemon,
		const Camera &camera,
		const Transform &transform,
		bool accumulate)
{
	// Skip and warn if no active attachment
	if (m_active_attachment.empty()) {
		KOBRA_LOG_FUNC(Log::WARN) << "No active attachment\n";
		return;
	}

	// Compare with previous attachment
	if (m_active_attachment != m_previous_attachment) {
		if (m_previous_attachment.size() > 0)
			m_attachments[m_previous_attachment]->unload();

		m_previous_attachment = m_active_attachment;
		m_attachments[m_previous_attachment]->load();
	}

	auto out = preprocess_scene(ecs, transform_daemon, camera, transform);

	// Reset the accumulation state if needed
	if (!accumulate || out.handle.has_value())
		m_launch_info.samples = 0;

	// Invoke render for current attachment
	auto &attachment = m_attachments[m_previous_attachment];
	attachment->render(this, m_launch_info, out.handle, out.hit_records, m_extent);

	// Increment number of samples
	m_launch_info.samples++;
}

}

}
