// Engine headers
#include "../../include/amadeus/armada.cuh"

#include <hip/hiprtc.h>
#include <filesystem>

// Launch parameters
struct NaivePathTracerParameters : kobra::amadeus::ArmadaLaunchInfo {
	OptixTraversableHandle traversable;

	float *halton_x;
	float *halton_y;

	bool russian_roulette;
};

#ifndef NAIVE_SHADER

using namespace kobra;
using namespace kobra::amadeus;

// Taken from Nvidia's book
static void generate_halton_sequence(int N, int b, std::vector <float> &dst)
{
	int n = 0;
	int d = 1;

	for (int i = 0; i < N; i++) {
		int x = d - n;
		if (x == 1) {
			n = 1;
			d *= b;
		} else {
			int y = d/b;
			while (x <= y)
				y /= b;
			n = (b + 1) * y - x;
		}

		dst[i] = (float) n / (float) d;
	}
}

static void generate_pixel_offsets(int N, std::vector <float> &x, std::vector <float> &y)
{
	static std::vector <int> bases {2, 3, 5, 7, 11, 13};

	srand(time(0));
	int r1 = rand() % bases.size();
	int r2 = rand() % bases.size();

	if (r1 == r2)
		r2 = (r2 + 1) % bases.size();

	int b1 = bases[r1];
	int b2 = bases[r2];

	generate_halton_sequence(N, b1, x);
	generate_halton_sequence(N, b2, y);

	for (int i = 0; i < N; i++) {
		x[i] -= 0.5f;
		y[i] -= 0.5f;
	}
}

// Classic Monte Carlo path tracer
class NaivePathTracer : public AttachmentRTX {
	// SBT record types
	using RaygenRecord = optix::Record <int>;
	using MissRecord = optix::Record <int>;

	// Extent
	vk::Extent2D m_extent;

	// Pipeline related information
	OptixModule m_module;

	OptixProgramGroup m_ray_generation;
	OptixProgramGroup m_closest_hit;
	OptixProgramGroup m_closest_hit_shadow;
	OptixProgramGroup m_miss;
	OptixProgramGroup m_miss_shadow;

	OptixPipeline m_pipeline;

	// Buffer for launch parameters
	NaivePathTracerParameters m_parameters;

	std::vector <float> m_pixel_offsets_x;
	std::vector <float> m_pixel_offsets_y;

	hipDeviceptr_t m_cuda_parameters;

	// TODO: stream

	// Create the program groups and pipeline
	void create_pipeline(const OptixDeviceContext &optix_context) {
		static constexpr const char OPTIX_PTX_FILE[] =
			"bin/ptx/naive_path_tracer.ptx";

		// Load module
		m_module = optix::load_optix_module(
			optix_context, OPTIX_PTX_FILE,
			ppl_compile_options, module_options
		);

		// Load programs
		OptixProgramGroupOptions program_options = {};

		// Descriptions of all the programs
		std::vector <OptixProgramGroupDesc> program_descs = {
			OPTIX_DESC_RAYGEN(m_module, "__raygen__"),
			OPTIX_DESC_HIT(m_module, "__closesthit__"),
			OPTIX_DESC_MISS(m_module, "__miss__"),
		};

		// Corresponding program groups
		std::vector <OptixProgramGroup *> program_groups = {
			&m_ray_generation,
			&m_closest_hit,
			&m_miss,
		};

		optix::load_program_groups(
			optix_context,
			program_descs,
			program_options,
			program_groups
		);

		m_pipeline = optix::link_optix_pipeline(
			optix_context,
			{
				m_ray_generation,
				m_closest_hit,
				m_miss,
			},
			ppl_compile_options,
			ppl_link_options
		);
	}

	OptixShaderBindingTable m_sbt;
public:
	// Constructor
	NaivePathTracer() : AttachmentRTX(1) {
		m_parameters.russian_roulette = false;
	}

	// Attaching and unloading
	// TODO: return bool to indicate success
	void attach(const ArmadaRTX &armada_rtx) override {
		// First load the pipeline
		create_pipeline(armada_rtx.system()->context());

		// Get the extent
		m_extent = armada_rtx.extent();

		// Allocate the SBT
		std::vector <RaygenRecord> ray_generation_records(1);
		std::vector <MissRecord> miss_records(1);

		// Fill the SBT
		optix::pack_header(m_ray_generation, ray_generation_records[0]);
		optix::pack_header(m_miss, miss_records[0]);

		// Create the SBT
		m_sbt = {};

		m_sbt.raygenRecord = cuda::make_buffer_ptr(ray_generation_records);

		m_sbt.missRecordBase = cuda::make_buffer_ptr(miss_records);
		m_sbt.missRecordStrideInBytes = sizeof(MissRecord);
		m_sbt.missRecordCount = miss_records.size();

		m_sbt.hitgroupRecordBase = 0;
		m_sbt.hitgroupRecordStrideInBytes = 0;
		m_sbt.hitgroupRecordCount = 0;

		// Initialize the parameters buffer
		m_cuda_parameters = (hipDeviceptr_t) cuda::alloc <NaivePathTracerParameters> (1);

		m_pixel_offsets_x.clear();
		m_pixel_offsets_y.clear();
	}

	void load() override {
		// Generate the pixel offsets
		if (m_pixel_offsets_x.empty() || m_pixel_offsets_y.empty()) {
			int width = m_extent.width;
			int height = m_extent.height;

			m_pixel_offsets_x.resize(width * height);
			m_pixel_offsets_y.resize(width * height);

			generate_pixel_offsets(
				width * height,
				m_pixel_offsets_x, m_pixel_offsets_y
			);

			// Copy the parameters to the GPU
			std::cout << "Copying parameters to GPU\n";
			std::cout << "\tResolution: " << width << 'x' << height << '\n';
			std::cout << "\tsize: " << m_pixel_offsets_x.size() << '\n';
			m_parameters.halton_x = cuda::make_buffer(m_pixel_offsets_x);
			m_parameters.halton_y = cuda::make_buffer(m_pixel_offsets_y);
		}
	}

	void unload() override {
	}

	// Options
	void set_option(const std::string &field, const OptionValue &value) override {
		if (field == "russian_roulette") {
			if (std::holds_alternative <bool> (value)) {
				m_parameters.russian_roulette = std::get <bool> (value);
				std::cout << "Russian roulette: " << std::get <bool> (value) << '\n';
			} else {
				KOBRA_LOG_FILE(Log::WARN) << "Invalid value for"
					" russian_roulette option, expected bool\n";
			}
		}
	}

	OptionValue get_option(const std::string &field) const override {
		if (field == "russian_roulette")
			return m_parameters.russian_roulette;

		return {};
	}

	// Rendering
	void render(const ArmadaRTX *armada_rtx,
			const ArmadaLaunchInfo &launch_info,
			const std::optional <OptixTraversableHandle> &handle,
			std::vector <HitRecord> *hit_records,
			const vk::Extent2D &extent) override {
		// Check if hit groups need to be updated, and update them if necessary
		if (hit_records) {
			// Free old buffer
			if (m_sbt.hitgroupRecordBase)
				cuda::free(m_sbt.hitgroupRecordBase);

			// Update the SBT
			std::vector <HitRecord> local_hit_records(hit_records->size());
			for (size_t i = 0; i < hit_records->size(); i++) {
				local_hit_records[i] = (*hit_records)[i];
				pack_header(m_closest_hit, local_hit_records[i]);
			}

			m_sbt.hitgroupRecordBase = cuda::make_buffer_ptr(local_hit_records);
			m_sbt.hitgroupRecordStrideInBytes = sizeof(HitRecord);
			m_sbt.hitgroupRecordCount = local_hit_records.size();
		}

		// Copy the parameters and launch
		std::memcpy(&m_parameters, &launch_info, sizeof(ArmadaLaunchInfo));

		if (handle)
			m_parameters.traversable = *handle;

		cuda::copy(m_cuda_parameters, &m_parameters, 1, hipMemcpyHostToDevice);

		// Execute the pipeline
		OPTIX_CHECK(
			optixLaunch(
				m_pipeline, 0,
				m_cuda_parameters,
				sizeof(NaivePathTracerParameters),
				&m_sbt, extent.width, extent.height, 1
			)
		);

		CUDA_SYNC_CHECK();
	}
};

extern "C" {

struct ret {
	const char *name;
	kobra::amadeus::AttachmentRTX *ptr;
};

ret load_attachment()
{
	std::vector <std::string> include_paths = {
		KOBRA_DIR,
		KOBRA_DIR "/thirdparty/glm",
		KOBRA_DIR "/thirdparty/optix",
		KOBRA_DIR "/thirdparty/termcolor/include",
	};

	// TODO: compile the necessary PTX files
	constexpr char CUDA_SHADER_FILE[] = KOBRA_DIR "/source/optix/naive_path_tracer.cu";

	// Write the PTX file
	const std::string ptx_file = "bin/ptx/naive_path_tracer.ptx";

	std::vector <const char *> options {
		"-std=c++17",
		"-arch=compute_75",
		"-lineinfo",
		"-g",
		"-DKOBRA_OPTIX_SHADER=0",
		"--expt-relaxed-constexpr",
	};

	for (auto &path : include_paths) {
		path = "-I" + path;
		options.push_back(path.c_str());
	}

	std::string flags;
	for (const auto &option : options) {
		std::cout << "Option: " << option << std::endl;
		flags += option;
		flags += " ";
	}

	// Compile the PTX file
	std::string cmd = "nvcc -ptx " + flags;
	cmd += CUDA_SHADER_FILE;
	cmd += " -o " + ptx_file;

	std::cout << "Compiling PTX file: " << cmd << std::endl;
	int ret = system(cmd.c_str());
	if (ret != 0) {
		KOBRA_LOG_FUNC(Log::ERROR) << "Failed to compile PTX file\n";
		return {nullptr, nullptr};
	}

	std::cout << "PTX file compiled successfully\n";

	// TODO: global wide cache for the PTX files...
	return {
		"Naive Path Tracer",
		new NaivePathTracer()
	};
}

// TODO: dellocate attachment

}

#endif
