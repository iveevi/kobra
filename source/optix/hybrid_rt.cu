#include "hip/hip_runtime.h"
// OptiX headers
#include <optix.h>

// Engine headers
#include "../../include/cuda/brdf.cuh"
#include "../../include/cuda/material.cuh"
#include "../../include/cuda/math.cuh"
#include "../../include/optix/core.cuh"
#include "../../include/optix/parameters.cuh"

using namespace kobra::cuda;
using namespace kobra::optix;

extern "C"
{
	__constant__ kobra::optix::HT_Parameters ht_params;
}

// Local constants
static const float eps = 1e-3f;

// Power heurestic
static const float p = 2.0f;

__device__ float power(float pdf_f, float pdf_g)
{
	float f = pow(pdf_f, p);
	float g = pow(pdf_g, p);

	return f/(f + g);
}

// Check shadow visibility
KCUDA_INLINE __device__
bool shadow_visibility(float3 origin, float3 dir, float R)
{
	bool vis = false;

	unsigned int j0, j1;
	pack_pointer <bool> (&vis, j0, j1);

	optixTrace(ht_params.traversable,
		origin, dir,
		0, R - 0.01f, 0,
		OptixVisibilityMask(0b1),
		OPTIX_RAY_FLAG_DISABLE_CLOSESTHIT
			| OPTIX_RAY_FLAG_DISABLE_ANYHIT
			| OPTIX_RAY_FLAG_TERMINATE_ON_FIRST_HIT,
		ht_params.instances, 0, 1,
		j0, j1
	);

	return vis;
}

// Direct lighting for specific types of lights
template <class Light>
__device__ float3 Ld_light(const Light &light, float3 x, float3 wo, float3 n,
		Material mat, bool entering, float3 &seed)
{
	float3 contr_nee {0.0f};
	float3 contr_brdf {0.0f};

	// NEE
	float3 lpos = sample_area_light(light, seed);
	float3 wi = normalize(lpos - x);
	float R = length(lpos - x);

	float3 f = brdf(mat, n, wi, wo, entering, mat.type) * abs(dot(n, wi));

	float ldot = abs(dot(light.normal(), wi));
	if (ldot > 1e-6) {
		float pdf_light = (R * R)/(light.area() * ldot);

		// TODO: how to decide ray type for this?
		float pdf_brdf = pdf(mat, n, wi, wo, entering, mat.type);

		bool vis = shadow_visibility(x + n * eps, wi, R);
		if (pdf_light > 1e-9 && vis) {
			float weight = power(pdf_light, pdf_brdf);
			float3 intensity = light.intensity;
			contr_nee += weight * f * intensity/pdf_light;
		}
	}

	// BRDF
	Shading out;
	float pdf_brdf;

	f = eval(mat, n, wo, entering, wi, pdf_brdf, out, seed) * abs(dot(n, wi));
	if (length(f) < 1e-6f)
		return contr_nee;

	float pdf_light = 0.0f;

	// TODO: need to check intersection for lights specifically (and
	// arbitrary ones too?)
	float ltime = light.intersects(x, wi);
	if (ltime <= 0.0f)
		return contr_nee;
	
	float weight = 1.0f;
	if (out & eTransmission) {
		return contr_nee;
		// pdf_light = (R * R)/(light.area() * ldot);
	} else {
		R = ltime;
		pdf_light = (R * R)/(light.area() * abs(dot(light.normal(), wi)));
		weight = power(pdf_brdf, pdf_light);
	};

	// TODO: shoot shadow ray up to R
	if (pdf_light > 1e-9 && pdf_brdf > 1e-9) {
		float3 intensity = light.intensity;
		contr_brdf += weight * f * intensity/pdf_brdf;
	}

	return contr_nee + contr_brdf;
}

// Trace ray into scene and get relevant information
__device__ float3 Ld(float3 x, float3 wo, float3 n,
		Material mat, bool entering, float3 &seed)
{
	int quad_count = ht_params.lights.quad_count;
	int tri_count = ht_params.lights.triangle_count;

	if (quad_count == 0 && tri_count == 0)
		return make_float3(0.0f);

	// TODO: multiply result by # of total lights

	// Random area light for NEE

// #define LIGHT_SAMPLES 5

#ifdef LIGHT_SAMPLES

	float3 contr {0.0f};

	for (int k = 0; k < LIGHT_SAMPLES; k++) {
		random3(seed);
		unsigned int i = seed.x * (hit_data->n_quad_lights + hit_data->n_tri_lights);
		i = min(i, hit_data->n_quad_lights + hit_data->n_tri_lights - 1);

		if (i < hit_data->n_quad_lights) {
			QuadLight light = hit_data->quad_lights[i];
			contr += Ld_light(light, hit_data, x, wo, n, mat, entering, seed);
		} else {
			TriangleLight light = hit_data->tri_lights[i - hit_data->n_quad_lights];
			contr += Ld_light(light, hit_data, x, wo, n, mat, entering, seed);
		}
	}

	return contr/LIGHT_SAMPLES;

#else 

	random3(seed);
	unsigned int i = seed.x * (quad_count + tri_count);
	i = min(i, quad_count + tri_count - 1);

	if (i < quad_count) {
		QuadLight light = ht_params.lights.quads[i];
		return Ld_light(light, x, wo, n, mat, entering, seed);
	}

	TriangleLight light = ht_params.lights.triangles[i - quad_count];
	return Ld_light(light, x, wo, n, mat, entering, seed);

#endif

}

// Ray packet data
struct RayPacket {
	float3	throughput;
	
	float3	value;
	float3	seed;
	float	ior;

	int	depth;
};

// Ray generation kernel
extern "C" __global__ void __raygen__rg()
{
	// TODO: perform the first direct lihgting in a CUDA kernel,
	// then pass the position, value and direction to the raygen kernel
	// to reduce the stack size

	// Get the launch index
	const uint3 idx = optixGetLaunchIndex();

	// Index to store and read the pixel
	const uint index = idx.x + idx.y * ht_params.resolution.x;

	int object_index = tex2D <int> (
		ht_params.ids, idx.x,
		ht_params.resolution.y - idx.y
	);

	// ht_params.color_buffer[index] = float4 {object_index/255.0f, 0.0f, 0.0f, 1.0f};
	// return;

	if (object_index <= 0) {
		const float3 U = ht_params.cam_u;
		const float3 V = ht_params.cam_v;
		const float3 W = ht_params.cam_w;

		// Compute ray origin and direction
		float2 d = 2.0f * make_float2(
			float(idx.x + 0.5f)/float(ht_params.resolution.x),
			float(idx.y + 0.5f)/float(ht_params.resolution.y)
		) - 1.0f;

		float3 dir = normalize(d.x * U + d.y * V + W);
	
		float u = atan2(dir.x, dir.z) / (2.0f * M_PI) + 0.5f;
		float v = asin(dir.y) / M_PI + 0.5f;

		float4 c = tex2D <float4> (ht_params.envmap, u, v);

		ht_params.color_buffer[index] = c;
		return;
	}

	// Calculate UV coordinates
	float2 uv = make_float2(
		(float) idx.x/(float) ht_params.resolution.x,
		(float) idx.y/(float) ht_params.resolution.y
	);

	uv.y = 1.0f - uv.y;

	// Extract the initial bounce information
	float3 x = make_float3(tex2D <float4> (ht_params.positions, uv.x, uv.y));
	float3 n = make_float3(tex2D <float4> (ht_params.normals, uv.x, uv.y));
	float3 wo = normalize(ht_params.camera - x);

	n = normalize(n);
	if (dot(n, wo) < 0.0f)
		n = -n;

	// TODO: why would the following condition be triggered
	if(isnan(n.x) || isnan(n.y) || isnan(n.z)) {
		ht_params.color_buffer[index] = float4 {0, 0, 0, 1};
		return;
	}

	Material mat {};
	mat.diffuse = make_float3(tex2D <float4> (ht_params.albedo, uv.x, uv.y));
	mat.specular = make_float3(tex2D <float4> (ht_params.specular, uv.x, uv.y));

	float4 extra = tex2D <float4> (ht_params.extra, uv.x, uv.y);

	mat.shininess = extra.x;
	mat.roughness = extra.y;
	mat.type = eDiffuse;

	// Store color
	float3 seed {float(idx.x), float(idx.y), ht_params.time};
	float3 direct = Ld(x, wo, n, mat, true, seed);
	
	// Generate new ray
	Shading out;
	float3 wi;
	float pdf;

	float3 f = eval(mat, n, wo, true, wi, pdf, out,seed);

	// Store the result
	RayPacket rp {
		.throughput = f * abs(dot(wi, n))/pdf,
		.value = direct,
		.seed = seed,
		.ior = 1, // TODO: get from textures
		.depth = 1,
	};

	// Pack the ray packet
	unsigned int i0, i1;
	pack_pointer(&rp, i0, i1);
	
	// Trace to get multibounce global illumination
	float3 offset = 1e-3f * n;
	if (out & Shading::eTransmission)
		offset = 1e-3f * wi;

	if (length(f) > 1e-6) {
		optixTrace(ht_params.traversable,
			x + offset, wi,
			0.0f, 1e16f, 0.0f,
			OptixVisibilityMask(0b1),
			OPTIX_RAY_FLAG_DISABLE_ANYHIT,
			0, 0, 0,
			i0, i1
		);
	}

	// Finally, store the result
	ht_params.color_buffer[index] = make_float4(rp.value);
}

struct mat3 {
	// Column major
	float m[9];

	__device__ __forceinline__ mat3() {}

	__device__ __forceinline__ mat3(float3 c1, float3 c2, float3 c3) {
		// Store in column major order
		m[0] = c1.x; m[3] = c2.x; m[6] = c3.x;
		m[1] = c1.y; m[4] = c2.y; m[7] = c3.y;
		m[2] = c1.z; m[5] = c2.z; m[8] = c3.z;
	}
};

__device__ __forceinline__ float3 operator*(mat3 m, float3 v)
{
	return make_float3(
		m.m[0] * v.x + m.m[3] * v.y + m.m[6] * v.z,
		m.m[1] * v.x + m.m[4] * v.y + m.m[7] * v.z,
		m.m[2] * v.x + m.m[5] * v.y + m.m[8] * v.z
	);
}

// Interpolate triangle values
template <class T>
__device__ T interpolate(T *arr, uint3 triagle, float2 bary)
{
	T a = arr[triagle.x];
	T b = arr[triagle.y];
	T c = arr[triagle.z];

	return (1.0f - bary.x - bary.y) * a + bary.x * b + bary.y * c;
}

// Sample from a texture
static __forceinline__ __device__ float4 sample_texture
		(Hit *hit_data, hipTextureObject_t tex, uint3 triangle, float2 bary)
{
	float2 uv = interpolate(hit_data->texcoords, triangle, bary);
	return tex2D <float4> (tex, uv.x, 1 - uv.y);
}

// Calculate hit normal
static __forceinline__ __device__ float3 calculate_normal
		(Hit *hit_data, uint3 triangle, float2 bary,
		 bool &entering)
{
	float3 e1 = hit_data->vertices[triangle.y] - hit_data->vertices[triangle.x];
	float3 e2 = hit_data->vertices[triangle.z] - hit_data->vertices[triangle.x];
	float3 ng = cross(e1, e2);

	if (dot(ng, optixGetWorldRayDirection()) > 0.0f) {
		ng = -ng;
		entering = false;
	} else {
		entering = true;
	}

	ng = normalize(ng);

	float3 normal = interpolate(hit_data->normals, triangle, bary);
	if (dot(normal, ng) < 0.0f)
		normal = -normal;

	normal = normalize(normal);

	if (hit_data->textures.has_normal) {
		float4 n4 = sample_texture(hit_data,
			hit_data->textures.normal,
			triangle, bary
		);

		float3 n = 2 * make_float3(n4.x, n4.y, n4.z) - 1;

		// Tangent and bitangent
		float3 tangent = interpolate(hit_data->tangents, triangle, bary);
		float3 bitangent = interpolate(hit_data->bitangents, triangle, bary);

		mat3 tbn = mat3(
			normalize(tangent),
			normalize(bitangent),
			normalize(normal)
		);

		normal = normalize(tbn * n);
	}

	return normal;
}

// Calculate relevant material data for a hit
__device__ void calculate_material
		(Hit *hit_data,
		Material &mat,
		uint3 triangle, float2 bary)
{
	if (hit_data->textures.has_diffuse) {
		mat.diffuse = make_float3(
			sample_texture(hit_data,
				hit_data->textures.diffuse,
				triangle, bary
			)
		);
	}

	if (hit_data->textures.has_roughness) {
		mat.roughness = sample_texture(hit_data,
			hit_data->textures.roughness,
			triangle, bary
		).x;
	}
}

#define MAX_DEPTH 3

// Closest hit kernel
extern "C" __global__ void __closesthit__ch()
{
	// Get payload
	RayPacket *rp;
	unsigned int i0 = optixGetPayload_0();
	unsigned int i1 = optixGetPayload_1();
	rp = unpack_point <RayPacket> (i0, i1);

	if (rp->depth > MAX_DEPTH)
		return;
	
	// Get data from the SBT
	Hit *hit = reinterpret_cast <Hit *> (optixGetSbtDataPointer());

	// Calculate relevant data for the hit
	float2 bary = optixGetTriangleBarycentrics();
	int primitive_index = optixGetPrimitiveIndex();
	uint3 triangle = hit->triangles[primitive_index];

	Material material = hit->material;

	// TODO: check for light, not just emissive material
	if (hit->material.type == Shading::eEmissive) {
		rp->value += rp->throughput * material.emission;
		rp->throughput = {0, 0, 0};
		return;
	}
	
	calculate_material(hit, material, triangle, bary);

	bool entering;
	float3 wo = -optixGetWorldRayDirection();
	float3 n = calculate_normal(hit, triangle, bary, entering);
	float3 x = interpolate(hit->vertices, triangle, bary);

	float3 direct = Ld(x, wo, n, material, entering, rp->seed);
	rp->value += rp->throughput * direct;

	// Generate new ray
	Shading out;
	float3 wi;
	float pdf;

	float3 f = eval(material, n, wo, entering, wi, pdf, out, rp->seed);
	if (length(f) < 1e-6f)
		return;
	
	float3 T = f * abs(dot(wi, n))/pdf;

	// Russian roulette
	float p = max(rp->throughput.x, max(rp->throughput.y, rp->throughput.z));
	float q = 1 - min(1.0f, p);

	if (fract(rp->seed.x) < q)
		return;

	rp->throughput *= T/(1 - q);

	// Recursive raytrace
	float3 offset = 1e-3f * n;
	if (out & Shading::eTransmission)
		offset = 1e-3f * wi;

	// Update ior
	rp->ior = material.refraction;
	rp->depth++;

	// Recurse
	optixTrace(ht_params.traversable,
		x + offset, wi,
		0.0f, 1e16f, 0.0f,
		OptixVisibilityMask(0b1),
		OPTIX_RAY_FLAG_DISABLE_ANYHIT,
		0, 0, 0,
		i0, i1
	);
}

extern "C" __global__ void __closesthit__shadow() {}

// Miss kernel
extern "C" __global__ void __miss__ms()
{
	// Get direction
	const float3 ray_direction = optixGetWorldRayDirection();

	float u = atan2(ray_direction.x, ray_direction.z)/(2.0f * M_PI) + 0.5f;
	float v = asin(ray_direction.y)/M_PI + 0.5f;

	float4 c = tex2D <float4> (ht_params.envmap, u, v);

	// Transfer to payload
	RayPacket *rp;
	unsigned int i0 = optixGetPayload_0();
	unsigned int i1 = optixGetPayload_1();
	rp = unpack_point <RayPacket> (i0, i1);

	rp->value += rp->throughput * make_float3(c);
}

extern "C" __global__ void __miss__shadow()
{
	unsigned int i0 = optixGetPayload_0();
	unsigned int i1 = optixGetPayload_1();
	bool *vis = unpack_point <bool> (i0, i1);
	*vis = true;
}
