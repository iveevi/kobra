#include "hip/hip_runtime.h"
// OptiX headers
#include <optix.h>

// Engine headers
#include "../../include/cuda/brdf.cuh"
#include "../../include/cuda/material.cuh"
#include "../../include/cuda/math.cuh"
#include "../../include/optix/core.cuh"
#include "../../include/optix/parameters.cuh"
#include "../../include/optix/lighting.cuh"
#include "../../include/cuda/matrix.cuh"

using namespace kobra::cuda;
using namespace kobra::optix;

extern "C"
{
	__constant__ kobra::optix::WadjetParameters parameters;
}

// Local constants
static const float eps = 1e-3f;

// Check shadow visibility
KCUDA_INLINE __device__
bool is_occluded(float3 origin, float3 dir, float R)
{
	bool vis = true;

	unsigned int j0, j1;
	pack_pointer <bool> (&vis, j0, j1);

	optixTrace(parameters.traversable,
		origin, dir,
		0, R - 0.01f, 0,
		OptixVisibilityMask(0b1),
		OPTIX_RAY_FLAG_DISABLE_CLOSESTHIT
			| OPTIX_RAY_FLAG_DISABLE_ANYHIT
			| OPTIX_RAY_FLAG_TERMINATE_ON_FIRST_HIT,
		parameters.instances, 0, 1,
		j0, j1
	);

	return vis;
}

// Trace ray into scene and get relevant information
__device__ float3 Ld(float3 x, float3 wo, float3 n,
		Material mat, bool entering, float3 &seed)
{
	int quad_count = parameters.lights.quad_count;
	int tri_count = parameters.lights.triangle_count;

	if (quad_count == 0 && tri_count == 0)
		return make_float3(0.0f);

#define LIGHT_SAMPLES 1

	float3 contr = make_float3(0.0f);
	for (int k = 0; k < LIGHT_SAMPLES; k++) {
		random3(seed);
		unsigned int i = seed.x * (quad_count + tri_count);
		i = min(i, quad_count + tri_count - 1);

		if (i < quad_count) {
			QuadLight light = parameters.lights.quads[i];
			contr += Ld_light(light, x, wo, n, mat, entering, seed);
		} else {
			TriangleLight light = parameters.lights.triangles[i - quad_count];
			contr += Ld_light(light, x, wo, n, mat, entering, seed);
		}
	}

	return contr/LIGHT_SAMPLES;
}

// Ray packet data
struct RayPacket {
	float3	value;
	float3	position;
	float3	normal;

	float	ior;
	
	int	depth;
	uint	index;
	
	float3	seed;
};

static KCUDA_INLINE KCUDA_HOST_DEVICE
void make_ray(uint3 idx,
		 float3 &origin,
		 float3 &direction,
		 float3 &seed)
{
	const float3 U = parameters.cam_u;
	const float3 V = parameters.cam_v;
	const float3 W = parameters.cam_w;
	
	/* Jittered halton
	int xoff = rand(parameters.image_width, seed);
	int yoff = rand(parameters.image_height, seed);

	// Compute ray origin and direction
	float xoffset = parameters.xoffset[xoff];
	float yoffset = parameters.yoffset[yoff];
	radius = sqrt(xoffset * xoffset + yoffset * yoffset)/sqrt(0.5f); */

	random3(seed);
	
	float xoffset = fract(seed.x) - 0.5f;
	float yoffset = fract(seed.y) - 0.5f;

	float2 d = 2.0f * make_float2(
		float(idx.x + xoffset)/parameters.resolution.x,
		float(idx.y + yoffset)/parameters.resolution.y
	) - 1.0f;

	origin = parameters.camera;
	direction = normalize(d.x * U + d.y * V + W);
}

// Ray generation kernel
extern "C" __global__ void __raygen__rg()
{
	// Get the launch index
	const uint3 idx = optixGetLaunchIndex();

	// Index to store and read the pixel
	const uint index = idx.x + idx.y * parameters.resolution.x;

	// Prepare the ray packet
	RayPacket rp {
		.value = make_float3(0.0f),
		.ior = 1.0f,
		.depth = 0,
		.index = index,
		.seed = make_float3(idx.x, idx.y, parameters.time)
	};
	
	// Trace ray and generate contribution
	unsigned int i0, i1;
	pack_pointer(&rp, i0, i1);

	float3 origin;
	float3 direction;

	make_ray(idx, origin, direction, rp.seed);

	optixTrace(parameters.traversable,
		origin, direction,
		0.0f, 1e16f, 0.0f,
		OptixVisibilityMask(0b11),
		OPTIX_RAY_FLAG_DISABLE_ANYHIT,
		0, 0, 0,
		i0, i1
	);
		
	// Finally, store the result
	float4 sample = make_float4(rp.value, 1.0f);
	if (parameters.accumulate) {
		float4 prev = parameters.color_buffer[index];
		parameters.color_buffer[index] = (prev * parameters.samples + sample)
			/(parameters.samples + 1);
	} else {
		parameters.color_buffer[index] = sample;
	}
}

// Interpolate triangle values
template <class T>
KCUDA_INLINE __device__
T interpolate(T *arr, uint3 triagle, float2 bary)
{
	T a = arr[triagle.x];
	T b = arr[triagle.y];
	T c = arr[triagle.z];

	return (1.0f - bary.x - bary.y) * a + bary.x * b + bary.y * c;
}

// Calculate hit normal
static __device__ float3 calculate_normal
		(Hit *hit_data, uint3 triangle,
		 float2 bary, float2 uv, bool &entering)
{
	float3 e1 = hit_data->vertices[triangle.y] - hit_data->vertices[triangle.x];
	float3 e2 = hit_data->vertices[triangle.z] - hit_data->vertices[triangle.x];
	float3 ng = cross(e1, e2);

	if (dot(ng, optixGetWorldRayDirection()) > 0.0f) {
		ng = -ng;
		entering = false;
	} else {
		entering = true;
	}

	ng = normalize(ng);

	float3 normal = interpolate(hit_data->normals, triangle, bary);
	if (dot(normal, ng) < 0.0f)
		normal = -normal;

	normal = normalize(normal);

	if (hit_data->textures.has_normal) {
		float4 n4 = tex2D <float4> (hit_data->textures.normal, uv.x, uv.y);
		float3 n = 2 * make_float3(n4.x, n4.y, n4.z) - 1;

		// Tangent and bitangent
		float3 tangent = interpolate(hit_data->tangents, triangle, bary);
		float3 bitangent = interpolate(hit_data->bitangents, triangle, bary);

		mat3 tbn = mat3(
			normalize(tangent),
			normalize(bitangent),
			normalize(normal)
		);

		normal = normalize(tbn * n);
	}

	return normal;
}

// Calculate relevant material data for a hit
KCUDA_INLINE __device__
void calculate_material(Hit *hit_data, Material &mat, uint3 triangle, float2 uv)
{
	if (hit_data->textures.has_diffuse) {
		float4 d4 = tex2D <float4> (hit_data->textures.diffuse, uv.x, uv.y);
		mat.diffuse = make_float3(d4);
	}

	if (hit_data->textures.has_roughness) {
		float4 r4 = tex2D <float4> (hit_data->textures.roughness, uv.x, uv.y);
		mat.roughness = r4.x;
	}
}

#define MAX_DEPTH 3

// Temporal resampling
KCUDA_INLINE __device__
float3 temporal_reuse(RayPacket *rp, const PathSample &sample, float weight)
{
	// Get reservoir
	auto &r_temporal = parameters.advanced.r_temporal[rp->index];

	// Proceed to add the current sample to the reservoir
	r_temporal.update(sample, weight);

	// Get resampled value
	return r_temporal.sample.value;
}

// Spatiotemporal resampling
KCUDA_INLINE __device__
float3 spatiotemporal_reuse(RayPacket *rp, float3 x, float3 n)
{
	// X and Y of the pixel
	int ix = rp->index % parameters.resolution.x;
	int iy = rp->index / parameters.resolution.x;

	// Then use spatial resampling
	auto &r_spatial = parameters.advanced.r_spatial[rp->index];
	
	const int SPATIAL_SAMPLES = (r_spatial.count < 250) ? 9 : 3;
	for (int i = 0; i < SPATIAL_SAMPLES; i++) {
		// Generate random neighboring pixel
		random3(rp->seed);

		float radius = 500.0f * fract(rp->seed.x);
		float angle = 2 * M_PI * fract(rp->seed.y);

		int ny = iy + radius * sin(angle);
		int nx = ix + radius * cos(angle);
		
		if ((nx < 0 || nx >= parameters.resolution.x)
				|| (ny < 0 || ny >= parameters.resolution.y))
			continue;

		int nindex = ny * parameters.resolution.x + nx;

		// Get the appropriate reservoir
		auto *reservoir = &parameters.advanced.r_spatial_prev[nindex];
		if (reservoir->count < 50)
			reservoir = &parameters.advanced.r_temporal_prev[nindex];

		// Get information relative to sample
		auto &sample = reservoir->sample;

		float3 direction = normalize(sample.p_pos - x);
		float distance = length(sample.p_pos - x);

		// Check if the sample is visible
		bool occluded;
		if (sample.missed)
			occluded = is_occluded(x, sample.dir, 1e6);
		else
			occluded = is_occluded(x, direction, distance);

		if (occluded)
			continue;

		// Check geometry similarity
		float depth_x = length(x - parameters.camera);
		float depth_s = length(sample.p_pos - parameters.camera);

		float theta = 180 * acos(dot(n, sample.s_normal))/M_PI;
		float ndepth = abs(depth_x - depth_s)/max(depth_x, depth_s);

		if (angle > 25 || ndepth > 0.1)
			continue;

		// Compute Jacobian
		float3 xq_1 = sample.p_pos;
		float3 xq_2 = sample.s_pos;
		float3 xr_1 = x;

		float3 v_r = xr_1 - xq_2;
		float3 v_q = xq_1 - xq_2;

		float d_r = length(v_r);
		float d_q = length(v_q);

		v_r /= d_r;
		v_q /= d_q;

		float phi_r = acos(dot(sample.s_normal, v_r));
		float phi_q = acos(dot(sample.s_normal, v_q));

		float J = abs(phi_r/phi_q) * (d_q * d_q)/(d_r * d_r);

		// If conditions are sufficient, merge reservoir
		if (!occluded) {
			r_spatial.merge(
				*reservoir,
				max(reservoir->sample.value)/J
			);
		}
	}

	// Get resampled value
	return r_spatial.sample.value;
}

// Closest hit kernel
extern "C" __global__ void __closesthit__ch()
{
	// Get payload
	RayPacket *rp;
	unsigned int i0 = optixGetPayload_0();
	unsigned int i1 = optixGetPayload_1();
	rp = unpack_pointer <RayPacket> (i0, i1);

	if (rp->depth > MAX_DEPTH)
		return;

	// Check if primary ray
	bool primary = (rp->depth == 0);
	
	// Get data from the SBT
	Hit *hit = reinterpret_cast <Hit *> (optixGetSbtDataPointer());

	// Calculate relevant data for the hit
	float2 bary = optixGetTriangleBarycentrics();
	int primitive_index = optixGetPrimitiveIndex();
	uint3 triangle = hit->triangles[primitive_index];

	// Get UV coordinates
	float2 uv = interpolate(hit->texcoords, triangle, bary);
	uv.y = 1 - uv.y;

	// Calculate the material
	Material material = hit->material;

	// TODO: check for light, not just emissive material
	if (hit->material.type == Shading::eEmissive) {
		rp->value = material.emission;
		return;
	}
	
	calculate_material(hit, material, triangle, uv);

	bool entering;
	float3 wo = -optixGetWorldRayDirection();
	float3 n = calculate_normal(hit, triangle, bary, uv, entering);
	float3 x = interpolate(hit->vertices, triangle, bary);

	float3 direct = Ld(x, wo, n, material, entering, rp->seed);

	// Generate new ray
	Shading out;
	float3 wi;
	float pdf;

	float3 f = eval(material, n, wo, entering, wi, pdf, out, rp->seed);
	if (length(f) < 1e-6f)
		return;

	// Get threshold value for current ray
	float3 T = f * abs(dot(wi, n))/pdf;

	// Recursive raytrace
	float3 offset = 1e-3f * n;
	if (out & Shading::eTransmission)
		offset = 1e-3f * wi;

	// Update ior
	rp->ior = material.refraction;
	rp->depth++;

	// Recurse
	optixTrace(parameters.traversable,
		x + offset, wi,
		0.0f, 1e16f, 0.0f,
		OptixVisibilityMask(0b1),
		OPTIX_RAY_FLAG_DISABLE_ANYHIT,
		0, 0, 0,
		i0, i1
	);

	// Post: advanced sampling techniques if any
	float3 indirect = rp->value;

	// ReSTIR GI
	if (parameters.samples == 0) {
		auto &r_temporal = parameters.advanced.r_temporal[rp->index];
		auto &r_spatial = parameters.advanced.r_spatial[rp->index];

		// Reset for motion
		r_temporal.reset();
		r_spatial.reset();
	}

	if (primary && parameters.samples > 0) {
		// TODO: The ray misses if its depth is 1
		//	but not if it hits a light (check value)
		//	this fixes lights being black with ReSTIR
		bool missed = (rp->depth == 1);

		// Generate sample and weight
		PathSample sample {
			.value = rp->value,
			.dir = wi,
			.p_pos = x,
			.p_normal = n,
			.s_pos = rp->position,
			.s_normal = rp->normal,
			.missed = missed
		};

		float weight = max(rp->value)/pdf;

		// First actually update the temporal reservoir
		temporal_reuse(rp, sample, weight);

		if (parameters.samples > 0) {
			// Then use spatiotemporal resampling
			indirect = spatiotemporal_reuse(rp, x, n);
		}
	}

	rp->value = direct + T * indirect;
	rp->position = x;
	rp->normal = n;
}

extern "C" __global__ void __closesthit__shadow() {}

// Miss kernel
extern "C" __global__ void __miss__ms()
{
	// Get direction
	const float3 ray_direction = optixGetWorldRayDirection();

	float u = atan2(ray_direction.x, ray_direction.z)/(2.0f * M_PI) + 0.5f;
	float v = asin(ray_direction.y)/M_PI + 0.5f;

	float4 c = tex2D <float4> (parameters.envmap, u, v);

	// Transfer to payload
	RayPacket *rp;
	unsigned int i0 = optixGetPayload_0();
	unsigned int i1 = optixGetPayload_1();
	rp = unpack_pointer <RayPacket> (i0, i1);

	rp->value = make_float3(c);
}

extern "C" __global__ void __miss__shadow()
{
	unsigned int i0 = optixGetPayload_0();
	unsigned int i1 = optixGetPayload_1();
	bool *vis = unpack_pointer <bool> (i0, i1);
	*vis = false;
}
