#include "hip/hip_runtime.h"
#include "basilisk_common.cuh"

// #define VOXEL_SPATIAL_REUSE
// #define VOXEL_NAIVE_RESERVOIRS
#define TEXTURE_MAPPED_RESERVOIRS

#if defined(VOXEL_SPATIAL_REUSE)

// Closest hit program for Voxel Reservoirs
extern "C" __global__ void __closesthit__voxel()
{
	LOAD_RAYPACKET();
	LOAD_INTERSECTION_DATA();

	// Check if primary ray
	bool primary = (rp->depth == 0);
	
	if (hit->material.type == Shading::eEmissive) {
		rp->value = material.emission;
		return;
	}
	
	// Offset by normal
	// TODO: use more complex shadow bias functions
	// TODO: an easier check for transmissive objects
	x += (material.type == Shading::eTransmission ? -1 : 1) * n * eps;

	float3 direct = Ld(x, wo, n, material, entering, rp->seed);

	// Generate new ray
	Shading out;
	float3 wi;
	float pdf;

	float3 f = eval(material, n, wo, entering, wi, pdf, out, rp->seed);
	if (length(f) < 1e-6f)
		return;

	// Get threshold value for current ray
	float3 T = f * abs(dot(wi, n))/pdf;

	// Update ior
	rp->ior = material.refraction;
	rp->depth++;
	
	// Get voxel coordinates
	float3 v_min = parameters.voxel.min;
	float3 v_max = parameters.voxel.max;
	int res = parameters.voxel.resolution;

	int3 c = make_int3(
		(x.x - v_min.x)/(v_max.x - v_min.x) * res,
		(x.y - v_min.y)/(v_max.y - v_min.y) * res,
		(x.z - v_min.z)/(v_max.z - v_min.z) * res
	);

	c = min(c, make_int3(res - 1));


	// Get reservoir at the voxel
	uint index = c.x + c.y * res + c.z * res * res;

	auto &r_voxel = parameters.voxel.reservoirs[index];
	int *lock = parameters.voxel.locks[index];

	float e = fract(random3(rp->seed).x);
	int count = r_voxel.count;

	
	// TODO: use a different threshold than 0.5 for spatial reuse
	// TODO: threshold should decrease over time
	float threshold = (1.0f - tanh(count/10))/2.0f;

	// TODO: analyze speedup when recursively updating voxels
	if (primary && count > 10 && e > 0.25) {
		float3 total_indirect = make_float3(0.0f);

		// TODO: how to determine number of samples to take?
		//	probably shouldnt be too low
		const int samples = 3; // 25, 100, etc
		const float max_radius = float(res);

		int success = 0;
		int n_occluded = 0;
		int n_out = 0;
		int n_void = 0;

		for (int i = 0; i < samples; i++) {
			// Generate random 3D offset index
			// TODO: use spherical mapping instead of rectangular
			
			float3 r = fract(random3(rp->seed));

			// NOTE: sqrt of the random variable results in larger
			// radii
			float radius = fract(sqrt(random3(r).x)) * max_radius;

			// TODO: select between these filters by sampling ~5
			// from each in the inital stage
#if 0

			// Cubic sampling
			r = r * 2.0f - 1.0f;

			int3 offset = make_int3(
				r.x * radius,
				r.y * radius,
				r.z * radius
			);

#elif 0

			// Spherical sampling
			float theta = r.x * 2.0f * M_PI;
			float phi = r.y * M_PI;

			float3 pre_offset = make_int3(
				radius * sin(phi) * cos(theta),
				radius * sin(phi) * sin(theta),
				radius * cos(phi)
			);

			// pre_offset += n * 

#else

			// Normal disk sampling
			float theta = r.x * 2.0f * M_PI;

			// Get vectors orthogonal to n
			const float3 up = make_float3(0.0f, 1.0f, 0.0f);
			const float3 right = make_float3(1.0f, 0.0f, 0.0f);

			float3 u = normalize(cross(n, up));
			if (length(u) < 1e-6f)
				u = normalize(cross(n, right));

			float3 v = normalize(cross(n, u));

			float3 pre_offset = make_float3(
				radius * cos(theta) * u.x + radius * sin(theta) * v.x,
				radius * cos(theta) * u.y + radius * sin(theta) * v.y,
				radius * cos(theta) * u.z + radius * sin(theta) * v.z
			);

			pre_offset += 0.5 * n * (2 * r.y - 1);
			int3 offset = make_int3(pre_offset);

#endif

			int3 nindex = c + offset;

			// Check if the offset is in bounds
			if (nindex.x < 0 || nindex.x >= res ||
				nindex.y < 0 || nindex.y >= res ||
				nindex.z < 0 || nindex.z >= res) {
				n_out++;
				continue;
			}

			// Get the reservoir at the offset
			int nindex_1d = nindex.x + nindex.y * res + nindex.z * res * res;

			// Get voxel and lock
			auto &r_voxel = parameters.voxel.reservoirs[nindex_1d];
			int *lock = parameters.voxel.locks[nindex_1d];

			// Lock and extract the sample
			// TODO: is the lock necessary?
			// while (atomicCAS(lock, 0, 1) == 0);
			float3 sample = r_voxel.samples[0].value;
			float3 position = r_voxel.samples[0].position;
			float3 direction = r_voxel.samples[0].direction;
			float W = r_voxel.W;
			int count = r_voxel.count;
			// atomicExch(lock, 0);

			// Skip if the reservoir is empty
			if (count == 0) {
				n_void++;
				continue;
			}

			// Check for occulsion
			float3 L = position - x;
			float3 L_n = normalize(L);

			bool occluded = is_occluded(x, L_n, length(L));
			if (occluded) {
				n_occluded++;
				continue;
			}

			// Add the contribution
			float3 brdf = kobra::cuda::brdf(material, n,
				direction, wo,
				entering, material.type
			);

			float pdf = kobra::cuda::pdf(material, n,
				direction, wo,
				entering, material.type
			);

			// total_indirect += sample * brdf * abs(dot(direction, n))/pdf;
			total_indirect += sample * brdf * abs(dot(direction, n)) * W;
			success++;
		}
		
		// TODO: spatial reservoirs as well...

// #define VISUALIZE

		if (success == 0) {
#ifdef VISUALIZE
			if (n_void > n_occluded)
				rp->value = make_float3(0, 0, n_void)/float(samples);
			else
				rp->value = make_float3(n_occluded, 0, 0)/float(samples);
#else
			// TODO: want to avoid this:
			trace <eVoxel> (x, wi, i0, i1);
			rp->value = direct + T * rp->value;
#endif
		} else {
#ifdef VISUALIZE
			rp->value = make_float3(0, 1, 0);
#else
			rp->value = direct + total_indirect/success;
#endif
		}
	} else if (primary) {
		// Recurse
		trace <eVoxel> (x, wi, i0, i1);
		float weight = length(rp->value)/pdf;

		// Update reservoir, locking
		VoxelSample vs {
			.value = rp->value,
			.position = rp->position,
			.direction = wi,
		};

		while (atomicCAS(lock, 0, 1) == 0);
		bool selected = r_voxel.update(vs, weight);
		float3 value = r_voxel.samples[0].value;
		float3 position = r_voxel.samples[0].position;
		float3 direction = r_voxel.samples[0].direction;
		float W = r_voxel.W  = r_voxel.weight/(r_voxel.count * length(value) + 1e-6);
		atomicExch(lock, 0);

#ifdef VISUALIZE
		rp->value = make_float3(1, 0, 1);
#else
		rp->value = direct + T * rp->value;
#endif
	} else {
		// Regular rays
		trace <eVoxel> (x, wi, i0, i1);
		rp->value = direct + T * rp->value;
	}

	rp->position = x;
	rp->normal = n;
}

#elif defined(VOXEL_NAIVE_RESERVOIRS)

extern "C" __global__ void __closesthit__voxel()
{
	// Get payload
	RayPacket *rp;
	unsigned int i0 = optixGetPayload_0();
	unsigned int i1 = optixGetPayload_1();
	rp = unpack_pointer <RayPacket> (i0, i1);
	
	if (rp->depth > MAX_DEPTH)
		return;

	// Check if primary ray
	bool primary = (rp->depth == 0);
	
	// Get data from the SBT
	Hit *hit = reinterpret_cast <Hit *> (optixGetSbtDataPointer());

	// Calculate relevant data for the hit
	float2 bary = optixGetTriangleBarycentrics();
	int primitive_index = optixGetPrimitiveIndex();
	uint3 triangle = hit->triangles[primitive_index];

	// Get UV coordinates
	float2 uv = interpolate(hit->texcoords, triangle, bary);
	uv.y = 1 - uv.y;

	// Calculate the material
	Material material = hit->material;

	// TODO: check for light, not just emissive material
	if (hit->material.type == Shading::eEmissive) {
		rp->value = material.emission;
		return;
	}
	
	calculate_material(hit, material, triangle, uv);

	bool entering;
	float3 wo = -optixGetWorldRayDirection();
	float3 n = calculate_normal(hit, triangle, bary, uv, entering);
	float3 x = interpolate(hit->vertices, triangle, bary);

	// Offset by normal
	// TODO: use more complex shadow bias functions

	// TODO: an easier check for transmissive objects
	x += (material.type == Shading::eTransmission ? -1 : 1) * n * eps;

	float3 direct = Ld(x, wo, n, material, entering, rp->seed);

	// Generate new ray
	Shading out;
	float3 wi;
	float pdf;

	float3 f = eval(material, n, wo, entering, wi, pdf, out, rp->seed);
	if (length(f) < 1e-6f)
		return;

	// Get threshold value for current ray
	float3 T = f * abs(dot(wi, n))/pdf;

	// Update ior
	rp->ior = material.refraction;

	// Get voxel coordinates
	float3 v_min = parameters.voxel.min;
	float3 v_max = parameters.voxel.max;
	int res = parameters.voxel.resolution;

	int3 c = make_int3(
		(x.x - v_min.x)/(v_max.x - v_min.x) * res,
		(x.y - v_min.y)/(v_max.y - v_min.y) * res,
		(x.z - v_min.z)/(v_max.z - v_min.z) * res
	);

	c = min(c, make_int3(res - 1));

	// Issue with this approach using the same sample in a voxel creates
	// extreme aliasing (you can distinguish the voxels by color...)

	// TODO: screen shot of the naive approach (no spatial reuse, only
	// temporal) for reference in a writeup

	// Get reservoir at the voxel
	uint index = c.x + c.y * res + c.z * res * res;

	auto &r_voxel = parameters.voxel.reservoirs[index];
	int *lock = parameters.voxel.locks[index];

	int success = 0;

	float3 total_indirect = make_float3(0);

	// primary = ((MAX_DEPTH - rp->depth) >= MAX_DEPTH);
	primary = (rp->depth < 1);
	for (int i = 0; i < Voxel_Reservoir::size; i++) {
		// while (atomicCAS(lock, 0, 1) == 0);

		auto sample = r_voxel.samples[i];

		float3 value = sample.value;
		float3 position = sample.position;
		float3 direction = sample.direction;

		// atomicExch(lock, 0);

		// Check if the sample is occluded
		float3 L = position - x;
		float3 L_n = normalize(L);

		bool occluded = is_occluded(x + n * 0.01, L_n, length(L));
		if (occluded)
			continue;

		// Add to indirect lighting
		float pdf = kobra::cuda::pdf(material, n,
			direction, wo,
			entering, material.type
		);

		if (isnan(pdf) || pdf < 0.01)
			continue;

		/* if (isnan(pdf) || isnan(1.0/pdf)) {
			printf("pdf: %f\t1/pdf: %f\n", pdf, 1.0/pdf);
			assert(false);
		} */
		
		float3 brdf = kobra::cuda::brdf(material, n,
			direction, wo,
			entering, material.type
		);

		float3 f = brdf * abs(dot(direction, n))/pdf;
		total_indirect += value * f;

		success++;
	}

	// Reuse only if primary
	float r = fract(random3(rp->seed).x);
	if (primary && success > 0) {
		// rp->value = make_float3(success/float(Voxel_Reservoir::size));
		rp->value = direct + total_indirect/float(success);
		return;
	}

	// Regular rays and add to reservoir
	rp->depth++;
	trace <eVoxel> (x, wi, i0, i1);

	// Construct sample if primary ray
	if (primary) {
		VoxelSample sample {
			.value = rp->value,
			.position = x,
			.direction = wi
		};

		float weight = length(sample.value)/pdf;

		// Add to reservoir
		while (atomicCAS(lock, 0, 1) == 0);
		r_voxel.update(sample, weight);
		atomicExch(lock, 0);
		
		// rp->value = make_float3(1, 0, 0);
		// return;
	}

	// rp->value = make_float3(1, 0, 0);
	rp->value = direct + T * rp->value;

	/* bool occluded = false;
	float3 cached_sample = make_float3(0.0f);
	float3 cached_position = make_float3(0.0f);
	float3 cached_direction = make_float3(0.0f);
	float cached_W = 0;

	if (r_voxel.count > 0) {
		while (atomicCAS(lock, 0, 1) == 0);
		cached_position = r_voxel.samples[0].position;
		cached_sample = r_voxel.samples[0].value;
		cached_direction = r_voxel.samples[0].direction;
		cached_W = r_voxel.weight/(r_voxel.count * length(cached_sample) + 1e-6);
		atomicExch(lock, 0);

		// Check if the sample is occluded
		float3 L = cached_position - x;
		float3 L_n = normalize(L);
		occluded = is_occluded(x + n * 0.01, L_n, length(L));
	}

	// TODO: analyze speedup when recursively updating voxels
	int count = r_voxel.count;
	
	if (primary && count > 0 && !occluded) {
		float3 brdf = kobra::cuda::brdf(material, n,
			cached_direction, wo,
			entering, material.type
		);

		float pdf = kobra::cuda::pdf(material, n,
			cached_direction, wo,
			entering, material.type
		);

		if (pdf > 0) {
			rp->value = direct + brdf * cached_sample *
				cached_W * abs(dot(cached_direction, n));
		} else {
			rp->value = direct;
		}
	} else if (primary) {
		// Recurse
		trace <eVoxel> (x, wi, i0, i1);
		float weight = length(rp->value)/pdf;

		// Update reservoir, locking
		VoxelSample vs {
			.value = rp->value,
			.position = rp->position,
			.direction = wi,
		};

		while (atomicCAS(lock, 0, 1) == 0);
		bool selected = r_voxel.update(vs, weight);
		float3 value = r_voxel.samples[0].value;
		float3 position = r_voxel.samples[0].position;
		float3 direction = r_voxel.samples[0].direction;
		float W = r_voxel.W  = r_voxel.weight/(r_voxel.count * length(value) + 1e-6);
		atomicExch(lock, 0);

		rp->value = direct + T * rp->value;
	} else {
		trace <eVoxel> (x, wi, i0, i1);
		rp->value = direct + T * rp->value;
	} */

	rp->position = x;
	rp->normal = n;
}

#elif defined(TEXTURE_MAPPED_RESERVOIRS)

template <class T>
__forceinline__ __device__
bool update_reservoir(WeightedReservoir <T> *res, const T &sample,
		float pdf_hat, float pdf, float3 &seed)
{
	res->mis += pdf + 1e-4;

	float mis = pdf/res->mis;
	float mis_confidence = 1.0f/float(res->count + 1);
	// float mis_confidence = res->count/float(res->count + 1);
	float weight = mis * mis_confidence * pdf_hat/pdf;
	
	res->weight += weight;
	res->count = min(res->count + 1, 20);
	// res->count++;

	float q = weight/res->weight;
	float e = fract(random3(seed)).x;

	bool selected = e < q;
	if (selected)
		res->sample = sample;

	return selected;
}

template <class T>
__forceinline__ __device__
void merge_reservoir(WeightedReservoir <T> *a, WeightedReservoir <T> *b,
		float pdf_hat, float3 &seed)
{
	a->mis += b->mis;

	float mis = b->mis/a->mis;
	float mis_confidence = b->count/float(a->count + b->count);
	float weight = mis * mis_confidence * pdf_hat/b->mis;

	a->weight += weight;
	a->count = min(a->count + b->count, 20);

	float q = weight/a->weight;
	float e = fract(random3(seed)).x;
	if (e < q)
		a->sample = b->sample;
}

const float isqrt2 = 0.70710676908493042;

__device__
float2 cubify(float3 s)
{
	float xx2 = s.x * s.x * 2.0;
	float yy2 = s.y * s.y * 2.0;

	float2 v {xx2 - yy2, yy2 - xx2};

	float ii = v.y - 3.0;
	ii *= ii;

	float isqrt = -sqrt(ii - 12.0 * xx2) + 3.0;

	v += {isqrt, isqrt};
	v = make_float2(sqrt(v.x), sqrt(v.y));
	v *= isqrt2;

	return v;

	// return sign(s) * make_float3(v, 1.0);
}

__device__
float2 sphere2cube(float3 sphere, int &face_index)
{
	float3 f = abs(sphere);

	bool a = f.y >= f.x && f.y >= f.z;
	bool b = f.x >= f.z;

	/* float3 s = a ? make_float3(sphere.y, sphere.z, sphere.x) :
		(b ? make_float3(sphere.x, sphere.z, sphere.y) :
		make_float3(sphere.x, sphere.y, sphere.z)); */

	// return a ? cubify(sphere.xzy).xzy : b ? cubify(sphere.yzx).zxy : cubify(sphere);
	float3 s;
	if (a) {
		s = make_float3(sphere.x, sphere.z, sphere.y);
		face_index = 0;
	} else {
		if (b) {
			s = make_float3(sphere.y, sphere.z, sphere.x);
			face_index = 1;
		} else{ 
			s = sphere;
			face_index = 2;
		}
	}

	if (s.z < 0.0) {
		s = -s;
		face_index += 3;
	}

	return cubify(s);
}

// TMRIS
// TODO: move to separate file and kernel
extern "C" __global__ void __closesthit__voxel()
{
	LOAD_RAYPACKET();
	LOAD_INTERSECTION_DATA();

	// Compute projection onto optimal plane
	float3 xvec = (x - hit->centroid);
	float3 nvec = hit->opt_normal;
	bool forward = dot(nvec, n) > 0;

// #define USE_CUBE_SPHERE_MAPPING
#define USE_PCA_MAPPING

#if defined(USE_SPHERICAL_MAPPING)

	float3 d = normalize(xvec + 0.5 * n);

	float u = atan2(d.x, d.z)/(2.0f * M_PI) + 0.5f;
	float v = asin(d.y)/M_PI + 0.5f;
	
	// TODO: dual textures?

	u = 0.5 * (u + forward);

#elif defined(USE_PCA_MAPPING)

	float3 xproj = xvec - dot(xvec, nvec) * nvec;
	
	float u = dot(xproj, hit->opt_tangent);
	float v = dot(xproj, hit->opt_bitangent);
	
	// Normalize
	float2 u_extent = hit->extent_tangent;
	float2 v_extent = hit->extent_bitangent;

	u = (u - u_extent.x)/(u_extent.y - u_extent.x);
	v = (v - v_extent.x)/(v_extent.y - v_extent.x);

	u = 0.5 * (u + forward);

#elif defined(USE_CUBE_SPHERE_MAPPING)

	// Project to sphere, then to cube
	float3 d = normalize(xvec + 2 * n);

	int face_index;
	float2 t_uv = sphere2cube(d, face_index);
	float u = t_uv.x;
	float v = t_uv.y;

	// Split into 6 faces
	u = 0.5 * (u + int(face_index/3));
	v = (v + face_index % 3)/3.0f;

#else

	float u = uv.x;
	float v = uv.y;

#endif
	
	// TODO: reolution based on mesh size/complexity (mostly size)
	constexpr int res = Hit::TMRIS_RESOLUTION;

	int ix = u * res;
	int iy = v * res;

	int index = ix + iy * res;
	index = clamp(index, 0, res * res - 1);

#if 0

	rp->value = make_float3(
		ix/float(res),
		iy/float(res),
		0
	);

	// int mod = (ix + iy) % 2;
	// rp->value = entering * make_float3(mod, mod, face_index/6.0f);
	// rp->value = make_float3(mod);
	// rp->value = make_float3(u,v,0);
	return;

#endif

	// Check for emissive objects
	if (hit->material.type == Shading::eEmissive) {
		rp->value = material.emission;
		rp->normal = n;
		rp->albedo = material.diffuse;
		return;
	}
	
	// Offset by normal
	// TODO: use more complex shadow bias functions
	// TODO: an easier check for transmissive objects
	x += (material.type == Shading::eTransmission ? -1 : 1) * n * eps;

	float3 direct = Ld(x, wo, n, material, entering, rp->seed);

	// Generate new ray
	Shading out;
	float3 wi;
	float pdf;

	float3 f = eval(material, n, wo, entering, wi, pdf, out, rp->seed);
	if (length(wi) + 0.01 < 1) {// TODO: caveat must "return" pos and n
		// rp->value = direct;
		rp->value = make_float3(1, 0, 1);
		return;
	}

	// Get threshold value for current ray
	float3 T = f * abs(dot(wi, n))/pdf;

	// Update ior
	rp->ior = material.refraction;
	rp->depth++;

	// TMRIS

	// Get reservoir and lock
	TMRIS_Reservoir *reservoir = &hit->tmris.f_res[index];

	int *lock = hit->tmris.f_locks[index];

	// NOTE: this dual buffering apparently does a lot... use it more effectively
	// TODO: capture the effects of this...
	if (!forward) {
		reservoir = &hit->tmris.b_res[index];
		lock = hit->tmris.b_locks[index];
	}

	// TODO: check if complete reuse is possible...

	trace <eRegular> (x, wi, i0, i1);

	// TODO: skip reservoir reuse for specular objects for now...
	float3 value = rp->value;
	float target = length(value);
	float weight = target/pdf;

	while (atomicCAS(lock, 0, 1) == 0);

	reservoir->count++;
	reservoir->weight += weight;

	float p = weight/reservoir->weight;
	float3 eta = fract(random3(rp->seed));

	if (eta.x < p || reservoir->count == 1) {
		TMRIS_Sample sample {
			.value = value,
			.position = rp->position,
			.source = x,
			.normal = n,
			.direction = wi,
			.missed = (rp->miss_depth == 1)
		};

		reservoir->sample = sample;
	}

	atomicExch(lock, 0);

	// TODO: additional reservoir for this step (in here...)

	// Get a random reservoir on the texture
	const float SAMPLING_RADIUS = 20.0f;

	eta = fract(random3(rp->seed));
	float radius = SAMPLING_RADIUS * sqrt(eta.x);
	float angle = 2.0f * M_PI * eta.y;

	int offx = radius * cos(angle);
	int offy = radius * sin(angle);

	int nix = ix + offx;
	int niy = iy + offy;

	// TODO: how efficient/valid is wraparound sampling?
	nix = (nix + res) % res;
	niy = (niy + res) % res;

	// Get reservoir and lock
	reservoir = &hit->tmris.f_res[nix + niy * res];
	lock = hit->tmris.f_locks[nix + niy * res];

	if (!forward) {
		reservoir = &hit->tmris.b_res[nix + niy * res];
		lock = hit->tmris.b_locks[nix + niy * res];
	}

	while (atomicCAS(lock, 0, 1) == 0);
	TMRIS_Sample sample = reservoir->sample;
	float W = reservoir->weight/length(sample.value);
	W /= reservoir->count;
	atomicExch(lock, 0);

	// TODO: compute jacobian of shift mapping and mis weights...

	// TODO: account for complete misses...
	bool occluded = true;

	float3 L = sample.position - x;
	float dist = length(L);
	L /= dist;

	occluded &= is_occluded(x, L, dist);

	// Compute Jacobian of reconnection shift mapping
	float cos_theta_x = abs(dot(sample.direction, sample.normal));
	float cos_theta_y = abs(dot(L, sample.normal));

	float dist_x = length(sample.position - sample.source);
	float dist_y = dist;

	float jacobian = (cos_theta_y/cos_theta_x);
	jacobian *= (dist_x * dist_x)/(dist_y * dist_y);

	W *= jacobian;

	// Compute final indirect
	f = brdf(material, n, sample.direction, wi, entering, out);
	float3 indirect = (1 - occluded) * sample.value; // TODO: incorporate
							 // proper W

	// Compute full lighting of reused ray
	rp->value = direct + f * indirect * abs(dot(wi, n));

	// NOTE: implement spatial sampling with current traced sample
	// + spatial sample like above

	// Pass through features
	rp->normal = n;
	rp->albedo = material.diffuse;
}

#endif
