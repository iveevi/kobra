#include "hip/hip_runtime.h"
#include "../../include/optix/parameters.cuh"
#include "common.cuh"

extern "C"
{
	__constant__ kobra::optix::BasiliskParameters parameters;
}

__forceinline__ __device__
float get(float3 a, int axis)
{
	if (axis == 0) return a.x;
	if (axis == 1) return a.y;
	if (axis == 2) return a.z;
}

extern "C" __global__ void __closesthit__voxel()
{
	LOAD_RAYPACKET();
	LOAD_INTERSECTION_DATA();

	// Check if primary ray
	bool primary = (rp->depth == 0);

	// TODO: first pass of rays is proxy for initialization?
	// TODO: extra buffer for direct lighting only, so that we can continue
	// with full lighting and show actual results?

	// Offset by normal
	x += (material.type == Shading::eTransmission ? -1 : 1) * n * eps;
	
	// Construct SurfaceHit instance for lighting calculations
	SurfaceHit surface_hit {
		.mat = material,
		.entering = entering,
		.n = n,
		.wo = wo,
		.x = x,
	};
	
	LightingContext lc {
		.quads = parameters.lights.quads,
		.triangles = parameters.lights.triangles,
		.quad_count = parameters.lights.quad_count,
		.triangle_count = parameters.lights.triangle_count,
		.has_envmap = parameters.has_envmap,
		.envmap = parameters.envmap,
	};

	// Reservoir for spatial sampling
	LightReservoir spatial {
		.sample = LightSample {},
		.count = 0,
		.weight = 0.0f,
	};

	// TODO: combine with vanilla ReSTIR

	// Obtain direct lighting sample
	// NOTE: decorrelating samples places into local and world space
	// reservoirs by using different samples for each
	// TODO: observe whether this is actually beneficial
	FullLightSample fls = sample_direct(lc, surface_hit, rp->seed);

	// Compute target function (unocculted lighting)
	float3 D = fls.point - surface_hit.x;
	float d = length(D);
	D /= d;

	float3 Li = direct_occluded(parameters.traversable, surface_hit, fls.Le, fls.normal, fls.type, D, d);
		
	// Contribution and weight
	float target = Li.x + Li.y + Li.z; // Luminance
	float pdf = fls.pdf;
	
	float w = (pdf > 0.0f) ? target/pdf : 0.0f;
		
	// Update reservoir
	// TODO: initialize sample to use
	reservoir_update(&spatial, LightSample {
		.value = Li,
		.target = target,
		.type = fls.type,
		.index = fls.index
	}, w, rp->seed);

	// World space resampling
	float3 direct = make_float3(0);

	if (parameters.kd_tree) {
		FullLightSample fls = sample_direct(lc, surface_hit, rp->seed);

		// Compute target function (unocculted lighting)
		float3 D = fls.point - surface_hit.x;
		float d = length(D);
		D /= d;

		float3 Li = direct_unoccluded(surface_hit, fls.Le, fls.normal, fls.type, D, d);
			
		// Contribution and weight
		float target = Li.x + Li.y + Li.z; // Luminance
		float pdf = fls.pdf;
		
		float w = (pdf > 0.0f) ? target/pdf : 0.0f;

		// TODO: skip traversal if w is zero?

		// Traverse the kd-tree
		WorldNode *kd_node = nullptr;

		int root = 0;
		int depth = 0;

		int lefts = 0;
		int rights = 0;

		float3 pos = surface_hit.x;
		
		while (root >= 0) {
			depth++;
			kd_node = &parameters.kd_tree[root];
			
			// If no valid branches, exit
			int left = kd_node->left;
			int right = kd_node->right;

			if (left == -1 && right == -1)
				break;

			// If only one valid branch, traverse it
			if (left == -1) {
				root = right;
				rights++;
				continue;
			}

			if (right == -1) {
				root = left;
				lefts++;
				continue;
			}

			// Otherwise, choose the branch according to the split
			float split = kd_node->split;
			int axis = kd_node->axis;

			if (get(pos, axis) < split) {
				root = left;
				lefts++;
			} else {
				root = right;
				rights++;
			}
		}

		// Lock and update the reservoir
		// TODO: similar scoped lock as std::lock_guard, in cuda/sync.h

#ifdef WSRIS_HASH_RESOLUION

		int base_idx = kd_node->data * WSRIS_HASH_RESOLUION;
		int res_idx = base_idx;

		// Hash position to get index in cell
		uint3 hash = pcg3d(*((uint3 *) &x));
		int cell_idx = (hash.x + hash.y + hash.z) % WSRIS_HASH_RESOLUION;

		res_idx += cell_idx;

		/* rp->value = make_float3(
			lefts/(float) depth,
			rights/(float) depth,
			cell_idx/(float) WSRIS_HASH_RESOLUION
		);

		return; */

#else

		int res_idx = kd_node->data;

#endif
		
		int *lock = parameters.kd_locks[res_idx];

		// while (atomicCAS(lock, 0, 1) == 0); // Lock

		auto *reservoir = &parameters.kd_reservoirs[res_idx];
		auto *sample = &reservoir->sample;

		reservoir_update(reservoir, LightSample {
			.value = fls.Le,
			.point = fls.point,
			.normal = fls.normal,
			.target = target,
			.type = fls.type,
			.index = fls.index
		}, w, rp->seed);

		LightSample ls = *sample;
		float w_sum = reservoir->weight;
		int count = reservoir->count;

		// atomicExch(lock, 0);			// Unlock

		// TODO: two strategies
		//	hierarchical: go up a few levels and then traverse down
		//	pick a random node and traverse down
		const int SPATIAL_SAMPLES = 1;

		// Choose a root node a few level up and randomly
		// traverse the tree to obtain a sample
		const int LEVELS = 10;

		// TODO: try selecting random indices in the tree instead?
		int levels = min(depth, LEVELS);
		while (levels--) {
			kd_node = &parameters.kd_tree[root];

			if (kd_node->parent == -1)
				break;

			root = kd_node->parent;
		}

		int successes = 0;
		for (int i = 0; i < SPATIAL_SAMPLES; i++) {
			int node = root;

			while (true) {
				kd_node = &parameters.kd_tree[node];

				float split = kd_node->split;
				int axis = kd_node->axis;

				// If no valid branches, exit
				int left = kd_node->left;
				int right = kd_node->right;

				if (left == -1 && right == -1)
					break;

				// If only one valid branch, go there
				if (left == -1) {
					node = right;
					continue;
				}

				if (right == -1) {
					node = left;
					continue;
				}

				// Otherwise, choose a random branch
				float eta = rand_uniform(rp->seed);

				if (eta < 0.5f)
					node = left;
				else
					node = right;
			}

			// Get necessary data
			// TODO: maybe lock?

#ifdef WSRIS_HASH_RESOLUION

			int base_idx = kd_node->data * WSRIS_HASH_RESOLUION;
			res_idx = base_idx;

			// Randomly select a cell
			int cell_idx = rand_uniform(WSRIS_HASH_RESOLUION, rp->seed);
			res_idx += cell_idx;

#else

			res_idx = kd_node->data;

#endif

			// TODO: syncronized pipeline, this one is copied
			// because no lock is used
			LightReservoir rsampled = parameters.kd_reservoirs_prev[res_idx];
			LightSample sample = rsampled.sample;

			// Compute value and target
			D = sample.point - surface_hit.x;
			d = length(D);
			D /= d;

			Li = direct_occluded(parameters.traversable,
				surface_hit, sample.value,
				sample.normal, sample.type, D, d
			);

			float denom = rsampled.count * sample.target;
			float W = (denom > 0.0f) ? rsampled.weight/denom : 0.0f;

			// Insert into spatial reservoir
			target = Li.x + Li.y + Li.z; // Luminance
			w = target * W * rsampled.count; // TODO: compute without doing repeated work

			int pcount = spatial.count;
			reservoir_update(&spatial, LightSample {
				.value = Li,
				.target = target,
				.type = sample.type,
				.index = sample.index
			}, w, rp->seed);

			// spatial.count = pcount + (target > 0.0f ? reservoir->count : 0);
			spatial.count = pcount + rsampled.count;
			successes += (target > 0.0f);

			/* Also insert into temporal reservoir
			Li = direct_unoccluded(surface_hit, sample.value, sample.normal, sample.type, D, d);

			target = Li.x + Li.y + Li.z; // Luminance
			denom = rsampled.count * target;
			W = (denom > 0.0f) ? rsampled.weight/denom : 0.0f;
			// assert(!isnan(W));

			w = target * W * rsampled.count;
			// assert(!isnan(w));

			reservoir->weight += w;
			// assert(!isnan(reservoir->weight));

			float p = w/reservoir->weight;
			float eta = rand_uniform(rp->seed);

			if (eta < p) {
				reservoir->sample = LightSample {
					.value = Li,
					.point = sample.point,
					.normal = sample.normal,
					.target = target,
					.type = sample.type,
					.index = sample.index
				};
			}

			reservoir->count += rsampled.count; */
		}
	}

	// Final direct lighting result
	float denom = spatial.count * spatial.sample.target;
	float W = (denom > 0) ? spatial.weight/denom : 0.0f;
	// assert(!isnan(W));

	direct = spatial.sample.value * W;
	
	// Add emission as well
	if (material.type == Shading::eEmissive)
		direct += material.emission;

	// Also compute indirect lighting
	Shading out;
	float3 wi;

	float3 f = eval(surface_hit, wi, pdf, out, rp->seed);

	// Get threshold value for current ray
	float3 T = f * abs(dot(wi, n))/pdf;

	// Update for next ray
	rp->ior = material.refraction;
	rp->pdf *= pdf;
	rp->depth++;
	
	// Trace the next ray
	float3 indirect = make_float3(0.0f);
	if (pdf > 0) {
		// trace <eRegular> (x, wi, i0, i1);
		trace <eVoxel> (
			parameters.traversable,
			x, wi, i0, i1
		);

		indirect = rp->value;
	}

	// Update the value
	bool skip_direct = (primary && parameters.options.indirect_only);
	if (!skip_direct)
		rp->value = direct;

	if (pdf > 0)
		rp->value += T * indirect;

	rp->position = make_float4(x, 1);
}
