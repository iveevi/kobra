#include "hip/hip_runtime.h"
#include "wadjet_common.cuh"

#define VOXEL_SPATIAL_REUSE
// #define VOXEL_NAIVE_RESERVOIRS

#if defined(VOXEL_SPATIAL_REUSE)

// Closest hit program for Voxel Reservoirs
extern "C" __global__ void __closesthit__voxel()
{
	LOAD_RAYPACKET();
	LOAD_INTERSECTION_DATA();

	// Check if primary ray
	bool primary = (rp->depth == 0);
	
	if (hit->material.type == Shading::eEmissive) {
		rp->value = material.emission;
		return;
	}
	
	// Offset by normal
	// TODO: use more complex shadow bias functions
	// TODO: an easier check for transmissive objects
	x += (material.type == Shading::eTransmission ? -1 : 1) * n * eps;

	float3 direct = Ld(x, wo, n, material, entering, rp->seed);

	// Generate new ray
	Shading out;
	float3 wi;
	float pdf;

	float3 f = eval(material, n, wo, entering, wi, pdf, out, rp->seed);
	if (length(f) < 1e-6f)
		return;

	// Get threshold value for current ray
	float3 T = f * abs(dot(wi, n))/pdf;

	// Update ior
	rp->ior = material.refraction;
	rp->depth++;
	
	// Get voxel coordinates
	float3 v_min = parameters.voxel.min;
	float3 v_max = parameters.voxel.max;
	int res = parameters.voxel.resolution;

	int3 c = make_int3(
		(x.x - v_min.x)/(v_max.x - v_min.x) * res,
		(x.y - v_min.y)/(v_max.y - v_min.y) * res,
		(x.z - v_min.z)/(v_max.z - v_min.z) * res
	);

	c = min(c, make_int3(res - 1));


	// Get reservoir at the voxel
	uint index = c.x + c.y * res + c.z * res * res;

	auto &r_voxel = parameters.voxel.reservoirs[index];
	int *lock = parameters.voxel.locks[index];

	float e = fract(random3(rp->seed).x);
	int count = r_voxel.count;

	
	// TODO: use a different threshold than 0.5 for spatial reuse
	// TODO: threshold should decrease over time
	float threshold = (1.0f - tanh(count/10))/2.0f;

	// TODO: analyze speedup when recursively updating voxels
	if (primary && count > 100) {
		float3 total_indirect = make_float3(0.0f);

		// TODO: how to determine number of samples to take?
		//	probably shouldnt be too low
		const int samples = 9; // 25, 100, etc
		const float radius = float(res)/5.0f;

		int success = 0;
		int n_occluded = 0;
		int n_out = 0;
		int n_void = 0;

		for (int i = 0; i < samples; i++) {
			// Generate random 3D offset index
			// TODO: use spherical mapping instead of rectangular

			float3 r = fract(random3(rp->seed));
			r = r * 2.0f - 1.0f;

			int3 offset = make_int3(
				r.x * radius,
				r.y * radius,
				r.z * radius
			);

			int3 nindex = c + offset;

			// Check if the offset is in bounds
			if (nindex.x < 0 || nindex.x >= res ||
				nindex.y < 0 || nindex.y >= res ||
				nindex.z < 0 || nindex.z >= res) {
				n_out++;
				continue;
			}

			// Get the reservoir at the offset
			int nindex_1d = nindex.x + nindex.y * res + nindex.z * res * res;

			// Get voxel and lock
			auto &r_voxel = parameters.voxel.reservoirs[nindex_1d];
			int *lock = parameters.voxel.locks[nindex_1d];

			// Lock and extract the sample
			// TODO: is the lock necessary?
			// while (atomicCAS(lock, 0, 1) == 0);
			float3 sample = r_voxel.sample.value;
			float3 position = r_voxel.sample.position;
			float3 direction = r_voxel.sample.direction;
			float W = r_voxel.W;
			int count = r_voxel.count;
			// atomicExch(lock, 0);

			// Skip if the reservoir is empty
			if (count == 0) {
				n_void++;
				continue;
			}

			// Check for occulsion
			float3 L = position - x;
			float3 L_n = normalize(L);

			bool occluded = is_occluded(x, L_n, length(L));
			if (occluded) {
				n_occluded++;
				continue;
			}

			// Add the contribution
			float3 brdf = kobra::cuda::brdf(material, n,
				direction, wo,
				entering, material.type
			);

			float pdf = kobra::cuda::pdf(material, n,
				direction, wo,
				entering, material.type
			);

			// total_indirect += sample * brdf * abs(dot(direction, n))/pdf;
			total_indirect += sample * brdf * abs(dot(direction, n)) * W;
			success++;
		}

		/* if (success == 0) {
			// NOTE: keep this viualization of occlusions density
			// TODO: also show sample density (i.e. the threshold value)
			// rp->value = make_float3(float(n_empty)/float(samples));

			// TODO: want to avoid this:
			trace <eVoxel> (x, wi, i0, i1);
			rp->value = direct + T * rp->value;
		} else {
			rp->value = direct + total_indirect/success;
		} */

		rp->value = make_float3(n_void)/float(samples);
	} else if (primary) {
		// Recurse
		trace <eVoxel> (x, wi, i0, i1);
		float weight = length(rp->value)/pdf;

		// Update reservoir, locking
		VoxelSample vs {
			.value = rp->value,
			.position = rp->position,
			.direction = wi,
		};

		while (atomicCAS(lock, 0, 1) == 0);
		bool selected = r_voxel.update(vs, weight);
		float3 value = r_voxel.sample.value;
		float3 position = r_voxel.sample.position;
		float3 direction = r_voxel.sample.direction;
		float W = r_voxel.W  = r_voxel.weight/(r_voxel.count * length(value) + 1e-6);
		atomicExch(lock, 0);

		// rp->value = direct + T * rp->value;
		rp->value = make_float3(1, 0, 1);
	} else {
		trace <eVoxel> (x, wi, i0, i1);
		rp->value = direct + T * rp->value;
	}

	rp->position = x;
	rp->normal = n;
}

#elif defined(VOXEL_NAIVE_RESERVOIRS)

extern "C" __global__ void __closesthit__voxel()
{
	// Get payload
	RayPacket *rp;
	unsigned int i0 = optixGetPayload_0();
	unsigned int i1 = optixGetPayload_1();
	rp = unpack_pointer <RayPacket> (i0, i1);
	
	if (rp->depth > MAX_DEPTH)
		return;

	// Check if primary ray
	bool primary = (rp->depth == 0);
	
	// Get data from the SBT
	Hit *hit = reinterpret_cast <Hit *> (optixGetSbtDataPointer());

	// Calculate relevant data for the hit
	float2 bary = optixGetTriangleBarycentrics();
	int primitive_index = optixGetPrimitiveIndex();
	uint3 triangle = hit->triangles[primitive_index];

	// Get UV coordinates
	float2 uv = interpolate(hit->texcoords, triangle, bary);
	uv.y = 1 - uv.y;

	// Calculate the material
	Material material = hit->material;

	// TODO: check for light, not just emissive material
	if (hit->material.type == Shading::eEmissive) {
		rp->value = material.emission;
		return;
	}
	
	calculate_material(hit, material, triangle, uv);

	bool entering;
	float3 wo = -optixGetWorldRayDirection();
	float3 n = calculate_normal(hit, triangle, bary, uv, entering);
	float3 x = interpolate(hit->vertices, triangle, bary);

	// Offset by normal
	// TODO: use more complex shadow bias functions

	// TODO: an easier check for transmissive objects
	x += (material.type == Shading::eTransmission ? -1 : 1) * n * eps;

	float3 direct = Ld(x, wo, n, material, entering, rp->seed);

	// Generate new ray
	Shading out;
	float3 wi;
	float pdf;

	float3 f = eval(material, n, wo, entering, wi, pdf, out, rp->seed);
	if (length(f) < 1e-6f)
		return;

	// Get threshold value for current ray
	float3 T = f * abs(dot(wi, n))/pdf;

	// Update ior
	rp->ior = material.refraction;
	rp->depth++;

	// Get voxel coordinates
	float3 v_min = parameters.voxel.min;
	float3 v_max = parameters.voxel.max;
	int res = parameters.voxel.resolution;

	int3 c = make_int3(
		(x.x - v_min.x)/(v_max.x - v_min.x) * res,
		(x.y - v_min.y)/(v_max.y - v_min.y) * res,
		(x.z - v_min.z)/(v_max.z - v_min.z) * res
	);

	c = min(c, make_int3(res - 1));

	// Issue with this approach using the same sample in a voxel creates
	// extreme aliasing (you can distinguish the voxels by color...)

	// TODO: screen shot of the naive approach (no spatial reuse, only
	// temporal) for reference in a writeup

	// Get reservoir at the voxel
	uint index = c.x + c.y * res + c.z * res * res;

	auto &r_voxel = parameters.voxel.reservoirs[index];
	int *lock = parameters.voxel.locks[index];

	bool occluded = false;
	float3 cached_sample = make_float3(0.0f);
	float3 cached_position = make_float3(0.0f);
	float3 cached_direction = make_float3(0.0f);
	float cached_W = 0;

	if (r_voxel.count > 0) {
		while (atomicCAS(lock, 0, 1) == 0);
		cached_position = r_voxel.sample.position;
		cached_sample = r_voxel.sample.value;
		cached_direction = r_voxel.sample.direction;
		cached_W = r_voxel.weight/(r_voxel.count * length(cached_sample) + 1e-6);
		atomicExch(lock, 0);

		// Check if the sample is occluded
		float3 L = cached_position - x;
		float3 L_n = normalize(L);
		occluded = is_occluded(x + n * 0.01, L_n, length(L));
	}

	// TODO: analyze speedup when recursively updating voxels
	int count = r_voxel.count;
	
	if (primary && count > 0 && !occluded) {
		float3 brdf = kobra::cuda::brdf(material, n,
			cached_direction, wo,
			entering, material.type
		);

		float pdf = kobra::cuda::pdf(material, n,
			cached_direction, wo,
			entering, material.type
		);

		if (pdf > 0) {
			rp->value = direct + brdf * cached_sample *
				abs(dot(cached_direction, n))/pdf;
		} else {
			rp->value = direct;
		}
	} else if (primary) {
		// Recurse
		trace <eVoxel> (x, wi, i0, i1);
		float weight = length(rp->value)/pdf;

		// Update reservoir, locking
		VoxelSample vs {
			.value = rp->value,
			.position = rp->position,
			.direction = wi,
		};

		while (atomicCAS(lock, 0, 1) == 0);
		bool selected = r_voxel.update(vs, weight);
		float3 value = r_voxel.sample.value;
		float3 position = r_voxel.sample.position;
		float3 direction = r_voxel.sample.direction;
		float W = r_voxel.W  = r_voxel.weight/(r_voxel.count * length(value) + 1e-6);
		atomicExch(lock, 0);

		rp->value = direct + T * rp->value;
	} else {
		trace <eVoxel> (x, wi, i0, i1);
		rp->value = direct + T * rp->value;
	}

	rp->position = x;
	rp->normal = n;
}

#endif
