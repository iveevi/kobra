#include "hip/hip_runtime.h"
#include "wadjet_common.cuh"

// Weighting function
KCUDA_INLINE static __device__
float weight_kernel(const PathSample &sample)
{
	return length(sample.value);
}

/* Temporal resampling
KCUDA_INLINE __device__
float3 temporal_reuse(RayPacket *rp, const PathSample &sample, float weight)
{
	// Get reservoir
	auto &r_temporal = parameters.advanced.r_temporal[rp->index];

	// Proceed to add the current sample to the reservoir
	r_temporal.update(sample, weight);
	r_temporal.W = r_temporal.weight/(
		r_temporal.count * weight_kernel(r_temporal.sample)
		+ 1e-6f
	);

	// Get resampled value
	return r_temporal.sample.value;
}

// Spatiotemporal resampling
KCUDA_INLINE __device__
float3 spatiotemporal_reuse(RayPacket *rp, float3 x, float3 n)
{
	// X and Y of the pixel
	int ix = rp->index % parameters.resolution.x;
	int iy = rp->index / parameters.resolution.x;

	// Then use spatial resampling
	auto &r_spatial = parameters.advanced.r_spatial[rp->index];
	auto &r_temporal = parameters.advanced.r_temporal[rp->index];
	auto &s_radius = parameters.advanced.sampling_radii[rp->index];
	// s_radius = 10.0f;

	int Z = 0;
	int success = 0;

	r_spatial.merge(r_temporal, weight_kernel(r_temporal.sample));
	Z += r_temporal.count;
	
	const int SPATIAL_SAMPLES = (r_spatial.count < 250) ? 9 : 3;
	for (int i = 0; i < SPATIAL_SAMPLES; i++) {
		// Generate random neighboring pixel
		random3(rp->seed);

		float radius = s_radius * fract(rp->seed.x);
		float angle = 2 * M_PI * fract(rp->seed.y);

		int ny = iy + radius * sin(angle);
		int nx = ix + radius * cos(angle);
		
		if ((nx < 0 || nx >= parameters.resolution.x)
				|| (ny < 0 || ny >= parameters.resolution.y))
			continue;

		int nindex = ny * parameters.resolution.x + nx;

		// Get the appropriate reservoir
		auto *reservoir = &parameters.advanced.r_spatial_prev[nindex];
		if (reservoir->count > 50)
			reservoir = &parameters.advanced.r_temporal_prev[nindex];

		if (reservoir->count == 0)
			continue;

		// Get information relative to sample
		auto &sample = reservoir->sample;

		// Check geometry similarity
		float depth_x = length(x - parameters.camera);
		float depth_s = length(sample.p_pos - parameters.camera);

		float theta = 180 * acos(dot(n, sample.s_normal))/M_PI;
		float ndepth = abs(depth_x - depth_s)/max(depth_x, depth_s);

		if (angle > 25 || ndepth > 0.1)
			continue;

		// Check if the sample is visible
		float3 direction = normalize(sample.p_pos - x);
		float distance = length(sample.p_pos - x);

		bool occluded;
		if (sample.missed)
			occluded = is_occluded(x + sample.dir * eps, sample.dir, 1e6);
		else
			occluded = is_occluded(x + direction * eps, direction, distance);

		if (occluded)
			continue;

		// Compute Jacobian
		float3 xq_1 = sample.p_pos;
		float3 xq_2 = sample.s_pos;
		float3 xr_1 = x;

		float3 v_r = xr_1 - xq_2;
		float3 v_q = xq_1 - xq_2;

		float d_r = length(v_r);
		float d_q = length(v_q);

		v_r /= d_r;
		v_q /= d_q;

		float phi_r = acos(dot(sample.s_normal, v_r));
		float phi_q = acos(dot(sample.s_normal, v_q));

		float J = abs(phi_r/phi_q) * (d_q * d_q)/(d_r * d_r);

		// If conditions are sufficient, merge reservoir
		r_spatial.merge(
			*reservoir,
			weight_kernel(reservoir->sample)/J
		);

		Z += reservoir->count;
		success++;
	}

	// Compute final weight
	r_spatial.W = r_spatial.weight/(
		Z * weight_kernel(r_spatial.sample)
		+ 1e-6f
	);

	// Reduce radius if no samples were found
	if (success == 0)
		s_radius = max(s_radius * 0.5f, 3.0f);

	// Get resampled value
	return r_spatial.sample.value;
} */

// Closest hit program for ReSTIR
extern "C" __global__ void __closesthit__restir()
{
	/* Get payload
	RayPacket *rp;
	unsigned int i0 = optixGetPayload_0();
	unsigned int i1 = optixGetPayload_1();
	rp = unpack_pointer <RayPacket> (i0, i1);
	
	if (rp->depth > MAX_DEPTH)
		return; */

	LOAD_RAYPACKET();
	LOAD_INTERSECTION_DATA();

	// Check if primary ray
	bool primary = (rp->depth == 0);
	
	// TODO: check for light, not just emissive material
	if (hit->material.type == Shading::eEmissive) {
		rp->value = material.emission;
		return;
	}
	
	// Offset by normal
	// TODO: use more complex shadow bias functions
	// TODO: an easier check for transmissive objects
	x += (material.type == Shading::eTransmission ? -1 : 1) * n * eps;

	float3 direct = Ld(x, wo, n, material, entering, rp->seed);

	// Generate new ray
	Shading out;
	float3 wi;
	float pdf;

	float3 f = eval(material, n, wo, entering, wi, pdf, out, rp->seed);
	if (length(f) < 1e-6f)
		return;

	// Get threshold value for current ray
	float3 T = f * abs(dot(wi, n))/pdf;

	// Update ior
	rp->ior = material.refraction;
	rp->depth++;

	// Recurse
	trace <eRegular> (x, wi, i0, i1);

	// Post: advanced sampling techniques if any
	float3 indirect = rp->value;

	// ReSTIR GI
	float max_radius = min(
		parameters.resolution.x,
		parameters.resolution.y
	)/10.0f;

	// TODO: temporal reporjection at some point
	if (parameters.samples == 0) {
		auto *r_temporal = &parameters.advanced.r_temporal[rp->index];
		// auto *r_spatial = &parameters.advanced.r_spatial[rp->index];
		auto *s_radius = &parameters.advanced.sampling_radii[rp->index];

		// Reset for motion
		// r_temporal->reset();
		r_temporal->count = 0;
		r_temporal->weight = 0.0f;
		r_temporal->mis = 0.0f;

		// r_spatial->reset();
		*s_radius = max_radius;
	}

	ReSTIR_Reservoir *r_temporal = &parameters.advanced.r_temporal[rp->index];

#if 0

	// Generate sample and weight
	PathSample sample {
		.value = f * rp->value * abs(dot(wi, n)),
		.dir = wi,
		.p_pos = x,
		.p_normal = n,
		.s_pos = rp->position,
		.s_normal = rp->normal,
		.target = length(f * rp->value * abs(dot(wi, n))),
		.missed = rp->missed,
	};

	// r_temporal->update(sample, weight);

	// NOTE: why is it incorrect to multiply pdf by path pdf?
	// pdf = rp->pdf;
	r_temporal->mis += pdf + 1e-4f;

	float mis = pdf/r_temporal->mis;
	float weight = mis * (sample.target/pdf);

	r_temporal->weight += weight;
	r_temporal->count = min(r_temporal->count + 1, 20);
	// r_temporal->count++;

	float q = weight/r_temporal->weight;
	float e = fract(random3(rp->seed)).x;
	if (e < q)
		r_temporal->sample = sample;

	sample = r_temporal->sample;

	int count = r_temporal->count;
	// float W = (r_temporal->weight/count)/(sample.target + 1e-6f);
	float W = r_temporal->weight/(sample.target + 1e-4f);

	float3 brdf_value = brdf(material, n, sample.dir, wo, entering, material.type);
	rp->value = direct + sample.value * W;
	
	/* reset
	r_temporal->count = 0;
	r_temporal->weight = 0.0f;
	r_temporal->mis = 0.0f; */

#else

	float3 value = direct + T * indirect;

	PathSample sample {
		.value = value,
		.dir = wi,
		.target = length(value)
	};

	// Update reservoir
	r_temporal->mis += pdf + 1e-4f;

	float mis = pdf/r_temporal->mis;
	float weight = mis * (sample.target/pdf);

	r_temporal->weight += weight;
	r_temporal->count = min(r_temporal->count + 1, 20);
	// r_temporal->count++;

	float q = weight/r_temporal->weight;
	float e = fract(random3(rp->seed)).x;

	if (e < q)
		r_temporal->sample = sample;

	sample = r_temporal->sample;

	float W = r_temporal->weight/(sample.target + 1e-4f);

	rp->value = sample.value * W;

#endif

}

// rp->value = make_float3(1/(1 + W));

// TODO: spatial sampling if samples > 0

/* First actually update the temporal reservoir
temporal_reuse(rp, sample, weight);

auto &r_temporal = parameters.advanced.r_temporal[rp->index];
sample = r_temporal.sample;

float3 brdf = kobra::cuda::brdf(material, n,
	sample.dir, wo,
	entering, material.type
);

rp->value = direct + brdf * sample.value * r_temporal.W *
abs(dot(sample.dir, n)); */

/* if (parameters.samples > 0) {
	// Then use spatiotemporal resampling
	indirect = spatiotemporal_reuse(rp, x, n);

	auto &r_spatial = parameters.advanced.r_temporal[rp->index];

	// TODO: recalculate value of f, using brdf...
	float3 brdf = kobra::cuda::brdf(material, n,
		r_spatial.sample.dir, wo,
		entering, material.type
	);

	float pdf = kobra::cuda::pdf(material, n,
		r_spatial.sample.dir, wo,
		entering, material.type
	);

	rp->value = direct + brdf * r_spatial.sample.value *
		r_spatial.W * abs(dot(r_spatial.sample.dir, n));
} */
