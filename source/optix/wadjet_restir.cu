#include "hip/hip_runtime.h"
#include "wadjet_common.cuh"

// Sample from discrete distribution
KCUDA_INLINE static __device__
int sample_discrete(float *pdfs, int num_pdfs, float eta)
{
	float sum = 0.0f;
	for (int i = 0; i < num_pdfs; ++i)
		sum += pdfs[i];
	
	float cdf = 0.0f;
	for (int i = 0; i < num_pdfs; ++i) {
		cdf += pdfs[i] / sum;
		if (eta < cdf)
			return i;
	}

	return num_pdfs - 1;
}

// Reservoir structure
template <class T>
struct Reservoir {
	int M;
	float weight;
	T sample;
};

// Closest hit program for ReSTIR
extern "C" __global__ void __closesthit__restir()
{
	LOAD_RAYPACKET();
	LOAD_INTERSECTION_DATA();

	// Check if primary ray
	bool primary = (rp->depth == 0);
	
	// TODO: check for light, not just emissive material
	if (hit->material.type == Shading::eEmissive) {
		rp->value = material.emission;
		rp->normal = n;
		rp->albedo = material.diffuse;
		return;
	}
	
	// Offset by normal
	// TODO: use more complex shadow bias functions
	// TODO: an easier check for transmissive objects
	x += (material.type == Shading::eTransmission ? -1 : 1) * n * eps;

	float3 direct = Ld(x, wo, n, material, entering, rp->seed);

	// Update ior
	rp->ior = material.refraction;
	rp->depth++;

	// Resampling Importance Sampling
	constexpr int M = 5;

#if 0

	float3 samples[M];
	float weights[M];
	float wsum = 0;

	for (int i = 0; i < M; i++) {
		// Generate new ray
		Shading out;
		float3 wi;
		float pdf;

		float3 f = eval(material, n, wo, entering, wi, pdf, out, rp->seed);
		if (length(f) < 1e-6f)
			continue;

		// Get threshold value for current ray
		trace <eRegular> (x, wi, i0, i1);

		float3 value = f * rp->value * abs(dot(wi, n));

		// RIS computations
		samples[i] = value;
		weights[i] = length(value)/(M * pdf);
		wsum += weights[i];
	}

	// Sample from the distribution
	float eta = fract(random3(rp->seed)).x;
	int index = sample_discrete(&weights[0], M, eta);

	float3 sample = samples[index];
	float W = wsum/length(sample);
	rp->value = direct + W * samples[index];

#elif 0

	// Reservoir sampling
	::Reservoir <float3> reservoir {
		.M = 0,
		.weight = 0.0f,
		.sample = make_float3(0.0f)
	};

	for (int i = 0; i < M; i++) {
		// Generate new ray
		Shading out;
		float3 wi;
		float pdf;

		float3 f = eval(material, n, wo, entering, wi, pdf, out, rp->seed);
		if (length(f) < 1e-6f)
			continue;

		// Get threshold value for current ray
		trace <eRegular> (x, wi, i0, i1);

		float3 value = f * rp->value * abs(dot(wi, n));

		// RIS computations
		float w = length(value)/(M * pdf);

		reservoir.weight += w;

		float p = w/reservoir.weight;
		float eta = fract(random3(rp->seed)).x;

		if (eta < p || i == 0)
			reservoir.sample = value;

		reservoir.M++;
	}

	float W = reservoir.weight/length(reservoir.sample);
	rp->value = direct + W * reservoir.sample;

#else

	ReSTIR_Reservoir *reservoir = &parameters.advanced.r_temporal[rp->index];
	if (parameters.samples == 0) {
		reservoir->sample = PathSample {};
		reservoir->weight = 0.0f;
		reservoir->count = 0;
	}

	// Temporal RIS
	Shading out;
	float3 wi;
	float pdf;

	float3 f = eval(material, n, wo, entering, wi, pdf, out, rp->seed);

	trace <eRegular> (x, wi, i0, i1);

	float3 value = f * rp->value * abs(dot(wi, n));

	PathSample sample {
		.value = value,
		.position = rp->position,
		.source = x,
		.normal = n,
		.direction = wi,
		.missed = (rp->miss_depth == 1)
	};

	// TODO: figure out how to do M-capping properly
	// reservoir->count = min(reservoir->count + 1, 20);
	reservoir->count++;

	float target = length(value);

	float w = target/(pdf + 1e-6f);
	reservoir->weight += w;

	float p = w/reservoir->weight;
	float eta = fract(random3(rp->seed)).x;

	if (eta < p || reservoir->count == 1)
		reservoir->sample = sample;

	// float W = reservoir->weight/length(reservoir->sample.value);
	// W /= reservoir->count;

	// Copy temporal to previous temporal
	parameters.advanced.r_temporal_prev[rp->index] = *reservoir;

	// Spatial RIS
	// TODO: persistent reservoirs
	ReSTIR_Reservoir spatial {
		.sample = PathSample {},
		.count = 0,
		.weight = 0.0f,
		.mis = 0.0f
	};

	// TODO: insert current sample at least...?

	const int SPATIAL_SAMPLES = 10;

	int width = parameters.resolution.x;
	int height = parameters.resolution.y;
	
	// TODO: adaptive radius
	const float SAMPLE_RADIUS = min(width, height)/10.0f;

	int ix = rp->index % width;
	int iy = rp->index / width;

	for (int i = 0; i < SPATIAL_SAMPLES; i++) {
		// Sample pixel in a radius
		float3 eta = fract(random3(rp->seed));

		int offx = floorf(eta.x * SAMPLE_RADIUS);
		int offy = floorf(eta.y * SAMPLE_RADIUS);

		int sx = ix + offx;
		int sy = iy + offy;

		/* TODO: is wraparound a good idea?
		if (sx < 0 || sx >= width)
			sx = ix - offx;

		if (sy < 0 || sy >= height)
			sy = iy - offy; */

		if (sx < 0 || sx >= width || sy < 0 || sy >= height)
			continue;

		int index = sx + sy * width;

		// Get the temporal resevoir at that pixel
		ReSTIR_Reservoir *temporal = &parameters.advanced.r_temporal_prev[index];
		if (temporal->count == 0)
			continue;
		
		// Get the sample
		const PathSample &sample = temporal->sample;
		if (sample.missed)
			continue;

		// Geometry constraints
		float depth_x = length(parameters.camera - x);
		float depth_y = length(parameters.camera - sample.position);

		float angle = acos(dot(n, sample.normal)) * 180.0f/M_PI;
		float depth = abs(depth_x - depth_y)/max(depth_x, depth_y);

		if (angle > 25.0f || depth > 0.2f)
			continue;

		// Check for occlusion
		float3 L = sample.position - x;
		float d = length(L);
		L /= d;

		// TODO: case when ray missed...
		bool occluded = is_occluded(x + L * 1e-4f, L, d);
		if (occluded)
			continue;

		// Compute weight for the sample for that reservoir
		float W = temporal->weight/length(temporal->sample.value);
		W /= temporal->count;

		// Compute GRIS weight
		float w = W * length(temporal->sample.value);

		// Compute jacobian of reconnection shift map
		// TODO: what if the ray misses? use only cosine term?
		float cos_theta_x = abs(dot(sample.direction, sample.normal));
		float cos_theta_y = abs(dot(L, sample.normal));

		float dist_x = length(sample.position - sample.source);
		float dist_y = d;

		float jacobian = (cos_theta_y/cos_theta_x);
		jacobian *= (dist_x * dist_x)/(dist_y * dist_y);

		w *= jacobian;

		// TODO: when does this happen?
		// assert(!isnan(w));
		if (isnan(w))
			continue;

		// Insert into spatial reservoir
		spatial.weight += w;
		spatial.count++;

		float p = w/spatial.weight;
		if (eta.z < p || spatial.count == 1)
			spatial.sample = temporal->sample;
	}

	// Compute final GRIS weight
	float W = spatial.weight/length(spatial.sample.value);
	W /= spatial.count;

	// assert(!isnan(W));
	if (isnan(W)) {
		// rp->value = make_float3(1, 0, 1);
		// return;
		W = 0.0f;
	}

	rp->value = direct + W * spatial.sample.value;

#endif

	// Pass through features
	rp->normal = n;
	rp->albedo = material.diffuse;
}
