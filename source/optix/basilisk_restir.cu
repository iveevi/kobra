#include "hip/hip_runtime.h"
#include "basilisk_common.cuh"

// Target function
KCUDA_INLINE KCUDA_DEVICE
float target_function(float3 Li)
{
	return Li.x + Li.y + Li.z;
}

// Get direct lighting using RIS
__device__
float3 direct_lighting_ris(const SurfaceHit &sh, Seed seed)
{
	const int M = 10;

	LightReservoir reservoir {
		.sample = LightSample {},
		.count = 0,
		.weight = 0.0f,
		.mis = 0.0f,
	};

	for (int k = 0; k < M; k++) {
		// Get direct lighting sample
		FullLightSample fls = sample_direct(sh, seed);
		if (fls.type != 2)
			continue;

		// Compute lighting
		float3 D = fls.point - sh.x;
		float d = length(D);
		D /= d;

		float3 Li = direct_occluded(sh, fls.Le, fls.normal, fls.type, D, d);

		// Resampling
		// TODO: common target function...
		float target = target_function(Li);
		float pdf = fls.pdf;

		float w = (pdf > 0.0f) ? target/pdf : 0.0f;

		reservoir_update(&reservoir, LightSample {
			.value = Li,
			.target = target,
			.type = fls.type,
			.index = fls.index
		}, w, seed);
	}

	// Get final sample and contribution
	LightSample sample = reservoir.sample;
	float W = (sample.target > 0) ? reservoir.weight/(M * sample.target) : 0.0f;

	return W * sample.value;
}

// Get direct lighting using Temporal RIS
__device__
float3 direct_lighting_temporal_ris(const SurfaceHit &sh, RayPacket *rp)
{
	// Get the reservoir
	LightReservoir *reservoir = &parameters.advanced.r_lights[rp->index];
	if (parameters.samples == 0) {
		reservoir->sample = LightSample {};
		reservoir->count = 0;
		reservoir->weight = 0.0f;
		reservoir->mis = 0.0f;
	}

	// TODO: temporal reprojection?

	// Get direct lighting sample
	FullLightSample fls = sample_direct(sh, rp->seed);

	// Compute lighting
	float3 D = fls.point - sh.x;
	float d = length(D);
	D /= d;

	float3 Li = direct_occluded(sh, fls.Le, fls.normal, fls.type, D, d);

	// Resampling
	float target = target_function(Li);
	float pdf = fls.pdf;

	float w = (pdf > 0.0f) ? target/pdf : 0.0f;

	reservoir_update(reservoir, LightSample {
		.value = Li,
		.target = target,
		.type = fls.type,
		.index = fls.index
	}, w, rp->seed);

	// Get final sample and contribution
	LightSample sample = reservoir->sample;
	float denominator = reservoir->count * sample.target;
	float W = (sample.target > 0) ? reservoir->weight/denominator : 0.0f;

	return W * sample.value;
}

// Get direct lighting using Spatio-Temporal RIS (ReSTIR)
__device__
float3 direct_lighting_restir(const SurfaceHit &sh, int index, Seed seed, int spatial_samples)
{
	// Get the reservoir
	// TODO: option to copy resrvoir and update locally rather than
	//       updating the global reservoir
	// TODO: do we actually need to worry about empty reservoirs?
	LightReservoir *temporal = &parameters.advanced.r_lights[index];
	if (parameters.samples == 0) {
		temporal->sample = LightSample {};
		temporal->count = 0;
		temporal->weight = 0.0f;
		temporal->mis = 0.0f;
	}

	// Get direct lighting sample
	FullLightSample fls = sample_direct(sh, seed);

	// Compute target function (unocculted lighting)
	float3 D = fls.point - sh.x;
	float d = length(D);
	D /= d;

	float3 Li = direct_unoccluded(sh, fls.Le, fls.normal, fls.type, D, d);

	// Temporal Resampling
	float target = target_function(Li);
	float pdf = fls.pdf;

	float w = (pdf > 0.0f) ? target/pdf : 0.0f;

	reservoir_update(temporal, LightSample {
		.value = fls.Le,
		.point = fls.point,
		.normal = fls.normal,
		.target = target,
		.type = fls.type,
		.index = fls.index
	}, w, seed);

	// Spatial Resampling
	LightReservoir spatial {
		.sample = LightSample {},
		.count = 0,
		.weight = 0.0f,
		.mis = 0.0f,
	};

	// Add current sample
	int Z = 0;

	{
		// Compute unbiased weight
		LightSample sample = temporal->sample;
		float denominator = temporal->count * sample.target;
		float W = (sample.target > 0) ? temporal->weight/denominator : 0.0f;

		// Compute value and target
		D = sample.point - sh.x;
		d = length(D);
		D /= d;

		float3 Li = direct_occluded(sh, sample.value, sample.normal, sample.type, D, d);

		// Add to the reservoir
		float target = target_function(Li);

		float w = target * W * temporal->count;

		spatial.weight += w;

		float p = w/spatial.weight;
		float eta = rand_uniform(seed);

		if (eta < p || spatial.count == 0) {
			spatial.sample = LightSample {
				.value = Li,
				.target = target,
				.type = sample.type,
				.index = sample.index
			};
		}

		spatial.count += temporal->count;
		if (target > 0.0f)
			Z += temporal->count;
	}

	// Sample various neighboring reservoirs
	const int WIDTH = parameters.resolution.x;
	const int HEIGHT = parameters.resolution.y;

	const float SAMPLING_RADIUS = min(WIDTH, HEIGHT) * 0.1f;

	int ix = index % WIDTH;
	int iy = index / WIDTH;

	for (int i = 0; i < spatial_samples; i++) {
		// Get offset
		float3 eta = rand_uniform_3f(seed);

		float radius = SAMPLING_RADIUS * sqrt(eta.x);
		float theta = 2.0f * M_PI * eta.y;

		int offx = (int) floorf(radius * cosf(theta));
		int offy = (int) floorf(radius * sinf(theta));

		int nix = ix + offx;
		int niy = iy + offy;

		if (niy < 0 || niy >= HEIGHT || nix < 0 || nix >= WIDTH)
			continue;

		int ni = niy * WIDTH + nix;

		// Get the reservoir
		LightReservoir *reservoir = &parameters.advanced.r_lights[ni];

		// Get sample and resample
		LightSample sample = reservoir->sample;
		float denominator = reservoir->count * sample.target;
		float W = (sample.target > 0) ? reservoir->weight/denominator : 0.0f;

		// Compute value and target
		D = sample.point - sh.x;
		d = length(D);
		D /= d;

		float3 Li = direct_occluded(sh, sample.value, sample.normal, sample.type, D, d);

		// Add to the reservoir
		// TODO: luminance as target?
		float target = target_function(Li);

		float w = target * W * reservoir->count;

		spatial.weight += w;

		float p = w/spatial.weight;
		if (eta.z < p || spatial.count == 0) {
			spatial.sample = LightSample {
				.value = Li,
				.target = target,
				.type = sample.type,
				.index = sample.index
			};
		}

		spatial.count += reservoir->count;
		if (target > 0.0f)
			Z += reservoir->count;
	}

	// Get final sample's contribution	
	LightSample sample = spatial.sample;
	float denominator = spatial.count * sample.target;
	float W = (denominator > 0) ? spatial.weight/denominator : 0.0f;

	// Evaluate the integrand
	return W * sample.value;
}

// Direct lighting for indirect rays, possible reuse
__device__
float3 direct_indirect(const SurfaceHit &surface_hit, Seed seed)
{
	if (!parameters.options.reprojected_reuse)
		return Ld(surface_hit, seed);

	// TODO: method
	const float3 U = parameters.cam_u;
	const float3 V = parameters.cam_v;
	const float3 W = parameters.cam_w;

	float3 D = surface_hit.x - parameters.camera;
	float d = length(D);
	D /= d;

	float D_W = dot(D, W);
	float u = dot(D, U)/(dot(D, W) * dot(U, U));
	float v = dot(D, V)/(dot(D, W) * dot(V, V));

	bool in_u_bounds = (u >= -1.0f && u <= 1.0f);
	bool in_v_bounds = (v >= -1.0f && v <= 1.0f);

	// TODO: how much does checking for occlusion matter?
	if (in_u_bounds && in_v_bounds) {
		u = (u + 1.0f) * 0.5f;
		v = (v + 1.0f) * 0.5f;

		int ix = (int) floorf(u * parameters.resolution.x);
		int iy = (int) floorf(v * parameters.resolution.y);

		int index = iy * parameters.resolution.x + ix;

		return direct_lighting_restir(surface_hit, index, seed, 3);
	}
	
	return Ld(surface_hit, seed);
}

// Closest hit program for ReSTIR
extern "C" __global__ void __closesthit__restir()
{
	LOAD_RAYPACKET();
	LOAD_INTERSECTION_DATA();

	// Check if primary ray
	bool primary = (rp->depth == 0);

	// Offset by normal
	x += (material.type == Shading::eTransmission ? -1 : 1) * n * eps;

	// Construct SurfaceHit instance for lighting calculations
	SurfaceHit surface_hit {
		.mat = material,
		.entering = entering,
		.n = n,
		.wo = wo,
		.x = x,
	};

	// Compute direct ligting
	float3 direct = make_float3(0.0f);

	if (primary) {
		// direct = direct_lighting_ris(surface_hit, rp->seed);
		// direct = direct_lighting_temporal_ris(surface_hit, rp);

		int spatial_samples = 10;
		if (parameters.options.reprojected_reuse)
			spatial_samples = 3;

		direct = direct_lighting_restir(
			surface_hit,
			rp->index, rp->seed,
			spatial_samples
		);
	} else {
		direct = direct_indirect(surface_hit, rp->seed);
	}

	if (material.type == Shading::eEmissive)
		direct += material.emission;
	
	// Generate new ray
	Shading out;
	float3 wi;
	float pdf;

	float3 f = eval(surface_hit, wi, pdf, out, rp->seed);

	// Get threshold value for current ray
	float3 T = f * abs(dot(wi, n))/pdf;

	// Update for next ray
	rp->ior = material.refraction;
	rp->pdf *= pdf;
	rp->depth++;
	
	// Trace the next ray
	float3 indirect = make_float3(0.0f);
	if (pdf > 0) {
		// trace <eRegular> (x, wi, i0, i1);
		trace <eReSTIR> (x, wi, i0, i1);
		indirect = rp->value;
	}

	// Update the value
	bool skip_direct = (primary && parameters.options.indirect_only);
	if (!skip_direct)
		rp->value = direct;

	if (pdf > 0)
		rp->value += T * indirect;

	rp->position = make_float4(x, 1);
	rp->normal = n;
	rp->albedo = material.diffuse;
	rp->wi = wi;
}
