#include "hip/hip_runtime.h"
// Standard headers
#include <cstdint>

// Engine headers
#include "../../include/cuda/math.cuh"
#include "../../include/cuda/color.cuh"
#include "../../include/layers/optix_tracer_common.cuh"

using kobra::HitGroupData;

extern "C"
{
	__constant__ kobra::Params params;
}

// Helper functionss
template <class T>
static __forceinline__ __device__ T *unpack_point(uint32_t i0, uint32_t i1)
{
	const uint64_t uptr = static_cast <uint64_t> (i0) << 32 | i1;
	T *ptr = reinterpret_cast <T *> (uptr);
	return ptr;
}

template <class T>
static __forceinline__ __device__ void pack_pointer(T * ptr, uint32_t &i0, uint32_t &i1)
{
	const uint64_t uptr = reinterpret_cast <uint64_t> (ptr);
	i0 = uptr >> 32;
	i1 = uptr & 0x00000000ffffffff;
}

static __forceinline__ __device__ void make_ray(uint3 idx, uint3 dim, float3 &origin, float3 &direction)
{
	const float3 U = params.cam_u;
	const float3 V = params.cam_v;
	const float3 W = params.cam_w;
	const float2 d = 2.0f * make_float2(float(idx.x)/dim.x, float(idx.y)/dim.y) - 1.0f;

	origin = params.cam_eye;
	direction = normalize(d.x * U + d.y * V + W);
}

// Ray packet data
struct RayPacket
{
	float3 value;
};

extern "C" __global__ void __raygen__rg()
{
	// Lookup our location within the launch grid
	const uint3 idx = optixGetLaunchIndex();
	const uint3 dim = optixGetLaunchDimensions();

	// Generate ray
	float3 ray_origin;
	float3 ray_direction;

	make_ray(idx, dim, ray_origin, ray_direction);

	// Pack payload
	RayPacket ray_packet;

	unsigned int i0, i1;
	pack_pointer(&ray_packet, i0, i1);
	
	// Launch
	optixTrace(params.handle,
		ray_origin, ray_direction,
		0.0f, 1e16f, 0.0f,
		OptixVisibilityMask(255), OPTIX_RAY_FLAG_NONE,
		0, 0, 0,
		i0, i1
	);

	// Unpack payload
	ray_packet = *unpack_point <RayPacket> (i0, i1);

	// Record results in our output raster
	params.image[idx.y * params.image_width + idx.x]
		= kobra::cuda::make_color(ray_packet.value);
}

extern "C" __global__ void __miss__ms()
{
	// Background color based on ray direction
	// TODO: implement background
	kobra::MissData *miss_data = reinterpret_cast
		<kobra::MissData *> (optixGetSbtDataPointer());

	const float3 ray_direction = optixGetWorldRayDirection();

	float u = atan2(ray_direction.x, ray_direction.z) / (2.0f * M_PI) + 0.5f;
	float v = asin(ray_direction.y) / M_PI + 0.5f;

	float4 c = tex2D <float4> (miss_data->bg_tex, u, v);

	// Transfer to payload
	RayPacket *rp;
	unsigned int i0 = optixGetPayload_0();
	unsigned int i1 = optixGetPayload_1();
	rp = unpack_point <RayPacket> (i0, i1);
	rp->value = make_float3(c.x, c.y, c.z);
}

static __forceinline__ __device__ float4 sample_texture
		(HitGroupData *hit_data, hipTextureObject_t tex, int prim, float2 bary)
{
	uint3 triangle = hit_data->triangles[prim];

	float2 uv1 = hit_data->texcoords[triangle.x];
	float2 uv2 = hit_data->texcoords[triangle.y];
	float2 uv3 = hit_data->texcoords[triangle.z];

	float2 uv = (1 - bary.x - bary.y) * uv1 + bary.x * uv2 + bary.y * uv3;

	return tex2D <float4> (tex, uv.x, uv.y);
}

struct mat3 {
	// Column major
	float m[9];

	__device__ __forceinline__ mat3() {}

	__device__ __forceinline__ mat3(float3 c1, float3 c2, float3 c3) {
		// Store in column major order
		m[0] = c1.x; m[3] = c2.x; m[6] = c3.x;
		m[1] = c1.y; m[4] = c2.y; m[7] = c3.y;
		m[2] = c1.z; m[5] = c2.z; m[8] = c3.z;
	}
};

__device__ __forceinline__ float3 operator*(mat3 m, float3 v)
{
	return make_float3(
		m.m[0] * v.x + m.m[3] * v.y + m.m[6] * v.z,
		m.m[1] * v.x + m.m[4] * v.y + m.m[7] * v.z,
		m.m[2] * v.x + m.m[5] * v.y + m.m[8] * v.z
	);
}

extern "C" __global__ void __closesthit__ch()
{
	// Get data from the SBT
	HitGroupData *hit_data = reinterpret_cast <HitGroupData *> (optixGetSbtDataPointer());

	float2 bary = optixGetTriangleBarycentrics();

	int primitive_index = optixGetPrimitiveIndex();

	uint3 triangle = hit_data->triangles[primitive_index];

	float3 n1 = hit_data->normals[triangle.x];
	float3 n2 = hit_data->normals[triangle.y];
	float3 n3 = hit_data->normals[triangle.z];

	float3 normal = (1 - bary.x - bary.y) * n1 + bary.x * n2 + bary.y * n3;
	if (dot(normal, optixGetWorldRayDirection()) > 0)
		normal = -normal;
	normal = normalize(normal);

	if (hit_data->textures.has_normal) {
		float4 n4 = sample_texture(hit_data,
			hit_data->textures.normal,
			primitive_index, bary
		);

		float3 n = 2 * make_float3(n4.x, n4.y, n4.z) - 1;

		// Tangent and bitangent
		float3 t1 = hit_data->tangents[triangle.x];
		float3 t2 = hit_data->tangents[triangle.y];
		float3 t3 = hit_data->tangents[triangle.z];

		float3 b1 = hit_data->bitangents[triangle.x];
		float3 b2 = hit_data->bitangents[triangle.y];
		float3 b3 = hit_data->bitangents[triangle.z];

		float3 tangent = (1 - bary.x - bary.y) * t1 + bary.x * t2 + bary.y * t3;
		float3 bitangent = (1 - bary.x - bary.y) * b1 + bary.x * b2 + bary.y * b3;

		mat3 tbn = mat3(
			normalize(tangent),
			normalize(bitangent),
			normalize(normal)
		);

		normal = normalize(tbn * n);
	} else {
	}

	// float3 color = get_diffuse(hit_data);
	float3 color = 0.5f * normal + 0.5f;

	// Transfer to payload
	RayPacket *rp;
	unsigned int i0 = optixGetPayload_0();
	unsigned int i1 = optixGetPayload_1();
	rp = unpack_point <RayPacket> (i0, i1);
	rp->value = color;
}
