#include "hip/hip_runtime.h"
#include "../../include/cuda/alloc.cuh"
#include "../../include/layers/denoiser.cuh"
#include "../../include/optix/core.cuh"

namespace kobra {

namespace layers {

// Create the denoiser layer
Denoiser Denoiser::make(const vk::Extent2D &extent, uint8_t guides)
{
	// Layer to return
	Denoiser layer;

	// Initialize dimensions
	layer.width = extent.width;
	layer.height = extent.height;
	layer.guides = (Denoiser::Guides) guides;

	// Create an OptiX context
	layer.context = optix::make_context();

	// Create the denoiser
	OptixDenoiserOptions denoiser_options = {};
	denoiser_options.guideNormal = (guides & eNormal);
	denoiser_options.guideAlbedo = (guides & eAlbedo);

	OPTIX_CHECK(optixDenoiserCreate(layer.context,
		OPTIX_DENOISER_MODEL_KIND_AOV,
		&denoiser_options,
		&layer.denoiser
	));

	// Optix denoiser sizes
	OptixDenoiserSizes denoiser_sizes;
	OPTIX_CHECK(
		optixDenoiserComputeMemoryResources(
			layer.denoiser,
			layer.width,
			layer.height,
			&denoiser_sizes
		)
	);

	int scratch_size = std::max(
		denoiser_sizes.withOverlapScratchSizeInBytes,
		denoiser_sizes.withoutOverlapScratchSizeInBytes
	);

	// Create the scratch buffers
	layer.state = cuda::BufferData(denoiser_sizes.stateSizeInBytes);
	layer.scratch = cuda::BufferData(scratch_size);

	// Set up denoiser
	OPTIX_CHECK(
		optixDenoiserSetup(layer.denoiser,
			0, layer.width, layer.height,
			layer.state.dev(), layer.state.size(),
			layer.scratch.dev(), layer.scratch.size()
		)
	);

	// Allocate result buffer
	layer.result = cuda::alloc(layer.width * layer.height * sizeof(float4));

	// Return the layer
	return layer;
}

// Perform the denoising
void denoise(Denoiser &layer, const Denoiser::Input &input)
{
	unsigned int row_stride = layer.width * sizeof(float4);

	// All the inputs
	OptixImage2D color_input {
		.data = input.color,
		.width = layer.width,
		.height = layer.height,
		.rowStrideInBytes = row_stride,
		.pixelStrideInBytes = sizeof(float4),
		.format = OPTIX_PIXEL_FORMAT_FLOAT4
	};

	OptixImage2D normal_image;
	OptixImage2D albedo_image;

	if (layer.guides & Denoiser::eNormal) {
		if (input.normal == 0) {
			KOBRA_LOG_FUNC(Log::ERROR)
				<< "Denoiser layer [" << &layer << "]"
				<< " requires a normal guide, but none was"
				<< " provided.\n";
			return;
		}

		normal_image = {
			.data = input.normal,
			.width = layer.width,
			.height = layer.height,
			.rowStrideInBytes = row_stride,
			.pixelStrideInBytes = sizeof(float4),
			.format = OPTIX_PIXEL_FORMAT_FLOAT4
		};
	}

	if (layer.guides & Denoiser::eAlbedo) {
		if (input.albedo == 0) {
			KOBRA_LOG_FUNC(Log::ERROR)
				<< "Denoiser layer [" << &layer << "]"
				<< " requires an albedo guide, but none was"
				<< " provided.\n";
			return;
		}

		albedo_image = {
			.data = input.albedo,
			.width = layer.width,
			.height = layer.height,
			.rowStrideInBytes = row_stride,
			.pixelStrideInBytes = sizeof(float4),
			.format = OPTIX_PIXEL_FORMAT_FLOAT4
		};
	}

	// Output
	OptixImage2D output {
		.data = layer.result,
		.width = layer.width,
		.height = layer.height,
		.rowStrideInBytes = row_stride,
		.pixelStrideInBytes = sizeof(float4),
		.format = OPTIX_PIXEL_FORMAT_FLOAT4
	};

	// Invoke the denoiser
	OptixDenoiserParams denoiser_params = {};
	
	OptixDenoiserGuideLayer denoiser_guide_layer;
	denoiser_guide_layer.normal = normal_image;
	denoiser_guide_layer.albedo = albedo_image;
	
	OptixDenoiserLayer denoiser_layer;
	denoiser_layer.input = color_input;
	denoiser_layer.output = output;

	// TODO: local CUDA stream?
	OPTIX_CHECK(
		optixDenoiserInvoke(layer.denoiser, 0,
			&denoiser_params,
			layer.state.dev(), layer.state.size(),
			&denoiser_guide_layer, &denoiser_layer,
			1, 0, 0,
			layer.scratch.dev(), layer.scratch.size()
		)
	);
}

}

}
