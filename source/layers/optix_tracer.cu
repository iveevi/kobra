#include "hip/hip_runtime.h"
// OptiX headers
#include <optix_device.h>
#include <optix_host.h>
#include <optix_stack_size.h>
#include <optix_function_table_definition.h>

// Engine headers
#include "../../include/cuda/error.cuh"
#include "../../include/cuda/alloc.cuh"
#include "../../include/layers/optix_tracer.cuh"
#include "../../include/layers/optix_tracer_common.cuh"
#include "../../include/camera.hpp"
#include "../../include/texture_manager.hpp"
#include "../../include/formats.hpp"
#include "../../include/cuda/color.cuh"

#include <stb_image_write.h>

namespace kobra {

namespace layers {

static void generate_mesh_data
		(const kobra::Raytracer *raytracer,
		const Transform &transform,
		optix_rt::HitGroupData &data)
{
	const Mesh &mesh = raytracer->get_mesh();

	std::vector <float3> vertices(mesh.vertices());
	std::vector <float2> uvs(mesh.vertices());
	std::vector <uint3> triangles(mesh.triangles());
	
	std::vector <float3> normals(mesh.vertices());
	std::vector <float3> tangents(mesh.vertices());
	std::vector <float3> bitangents(mesh.vertices());

	int vertex_index = 0;
	int uv_index = 0;
	int triangle_index = 0;
	
	int normal_index = 0;
	int tangent_index = 0;
	int bitangent_index = 0;

	for (const auto &submesh : mesh.submeshes) {
		for (int j = 0; j < submesh.vertices.size(); j++) {
			glm::vec3 n = submesh.vertices[j].normal;
			glm::vec3 t = submesh.vertices[j].tangent;
			glm::vec3 b = submesh.vertices[j].bitangent;
			
			glm::vec3 v = submesh.vertices[j].position;
			glm::vec2 uv = submesh.vertices[j].tex_coords;

			v = transform.apply(v);
			n = transform.apply_vector(n);
			t = transform.apply_vector(t);
			b = transform.apply_vector(b);
			
			normals[normal_index++] = {n.x, n.y, n.z};
			tangents[tangent_index++] = {t.x, t.y, t.z};
			bitangents[bitangent_index++] = {b.x, b.y, b.z};

			vertices[vertex_index++] = {v.x, v.y, v.z};
			uvs[uv_index++] = {uv.x, uv.y};
		}

		for (int j = 0; j < submesh.triangles(); j++) {
			triangles[triangle_index++] = {
				submesh.indices[j * 3 + 0],
				submesh.indices[j * 3 + 1],
				submesh.indices[j * 3 + 2]
			};
		}
	}

	data.vertices = cuda::make_buffer(vertices);
	data.texcoords = cuda::make_buffer(uvs);

	data.normals = cuda::make_buffer(normals);
	data.tangents = cuda::make_buffer(tangents);
	data.bitangents = cuda::make_buffer(bitangents);

	data.triangles = cuda::make_buffer(triangles);
}

static void generate_submesh_data
		(const Submesh &submesh,
		const Transform &transform,
		optix_rt::HitGroupData &data)
{
	std::vector <float3> vertices(submesh.vertices.size());
	std::vector <float2> uvs(submesh.vertices.size());
	std::vector <uint3> triangles(submesh.triangles());
	
	std::vector <float3> normals(submesh.vertices.size());
	std::vector <float3> tangents(submesh.vertices.size());
	std::vector <float3> bitangents(submesh.vertices.size());

	int vertex_index = 0;
	int uv_index = 0;
	int triangle_index = 0;
	
	int normal_index = 0;
	int tangent_index = 0;
	int bitangent_index = 0;

	for (int j = 0; j < submesh.vertices.size(); j++) {
		glm::vec3 n = submesh.vertices[j].normal;
		glm::vec3 t = submesh.vertices[j].tangent;
		glm::vec3 b = submesh.vertices[j].bitangent;
		
		glm::vec3 v = submesh.vertices[j].position;
		glm::vec2 uv = submesh.vertices[j].tex_coords;

		v = transform.apply(v);
		n = transform.apply_vector(n);
		t = transform.apply_vector(t);
		b = transform.apply_vector(b);
		
		normals[normal_index++] = {n.x, n.y, n.z};
		tangents[tangent_index++] = {t.x, t.y, t.z};
		bitangents[bitangent_index++] = {b.x, b.y, b.z};

		vertices[vertex_index++] = {v.x, v.y, v.z};
		uvs[uv_index++] = {uv.x, uv.y};
	}

	for (int j = 0; j < submesh.indices.size(); j += 3) {
		triangles[triangle_index++] = {
			submesh.indices[j],
			submesh.indices[j + 1],
			submesh.indices[j + 2]
		};
	}

	data.vertices = cuda::make_buffer(vertices);
	data.texcoords = cuda::make_buffer(uvs);

	data.normals = cuda::make_buffer(normals);
	data.tangents = cuda::make_buffer(tangents);
	data.bitangents = cuda::make_buffer(bitangents);

	data.triangles = cuda::make_buffer(triangles);
}

const std::vector <DSLB> OptixTracer::_dslb_render = {
	DSLB {
		0, vk::DescriptorType::eCombinedImageSampler,
		1, vk::ShaderStageFlagBits::eFragment
	}
};

template <class T>
struct Record {
	__align__ (OPTIX_SBT_RECORD_ALIGNMENT)
	char header[OPTIX_SBT_RECORD_HEADER_SIZE];

	T data;
};

typedef Record <optix_rt::RayGenData>     RayGenSbtRecord;
typedef Record <optix_rt::MissData>       MissSbtRecord;
typedef Record <optix_rt::HitGroupData>   HitGroupSbtRecord;

inline float3 to_f3(const glm::vec3 &v)
{
	return make_float3(v.x, v.y, v.z);
}

__forceinline__
__host__ __device__
uint32_t to_ui32(uchar4 v)
{
	// Reversed
	return (v.w << 24) | (v.z << 16) | (v.y << 8) | v.x;
}

static void context_log_cb( unsigned int level, const char* tag, const char* message, void* /*cbdata */)
{
	std::stringstream ss;
	ss << level << std::setw(20) << tag;
	logger(ss.str(), Log::AUTO, "OPTIX") << message << std::endl;
}

static hipTextureObject_t import_vulkan_texture(const vk::raii::Device &device, const ImageData &img)
{
	// Create a CUDA texture out of the Vulkan image
	hipExternalMemoryHandleDesc ext_mem_desc {};
	ext_mem_desc.type = hipExternalMemoryHandleTypeOpaqueFd;
	ext_mem_desc.handle.fd = img.get_memory_handle(device);
	ext_mem_desc.size = img.get_size();

	// Import the external memory
	hipExternalMemory_t tex_mem;
	CUDA_CHECK(hipSetDevice(0));
	CUDA_CHECK(hipImportExternalMemory(&tex_mem, &ext_mem_desc));

	// Create a mipmapped array for the texture
	cudaExternalMemoryMipmappedArrayDesc mip_desc {};
	mip_desc.flags = 0;
	mip_desc.formatDesc = hipCreateChannelDesc(8, 8, 8, 8, hipChannelFormatKindUnsigned);
	mip_desc.numLevels = 1;
	mip_desc.offset = 0;
	mip_desc.extent = make_hipExtent(
		img.extent.width,
		img.extent.height, 0
	);

	hipMipmappedArray_t mip_array;
	CUDA_CHECK(cudaExternalMemoryGetMappedMipmappedArray(&mip_array, tex_mem, &mip_desc));

	// Create the final texture object
	hipResourceDesc res_desc {};
	res_desc.resType = hipResourceTypeMipmappedArray;
	res_desc.res.mipmap.mipmap = mip_array;

	hipTextureDesc tex_desc {};
	tex_desc.readMode = hipReadModeNormalizedFloat;
	tex_desc.normalizedCoords = true;
	tex_desc.filterMode = hipFilterModeLinear;

	hipTextureObject_t tex_obj;
	CUDA_CHECK(hipCreateTextureObject(&tex_obj, &res_desc, &tex_desc, nullptr));

	return tex_obj;
}

// Set environment map
void OptixTracer::environment_map(const std::string &path)
{
	// First load the environment map
	_v_environment_map = &TextureManager::load_texture(
		*_ctx.phdev,
		*_ctx.device,
		path, true
	);

	// Update miss group record
	MissSbtRecord miss_record;
	miss_record.data.bg_color = float3 {0.0f, 0.0f, 0.0f};
	miss_record.data.bg_tex = import_vulkan_texture(*_ctx.device, *_v_environment_map);

	OPTIX_CHECK(optixSbtRecordPackHeader(_programs.miss_radiance, &miss_record));
	cuda::copy(_optix_miss_sbt, &miss_record, 1);
}

////////////
// Render //
////////////

void OptixTracer::render(const vk::raii::CommandBuffer &cmd,
		const vk::raii::Framebuffer &framebuffer,
		const ECS &ecs, const RenderArea &ra)
{
	// Get camera and camera transform
	Camera camera;
	Transform camera_transform;
	bool found_camera = false;

	std::vector <const kobra::Raytracer *> raytracers;
	std::vector <Transform> raytracer_transforms;
	std::vector <const Transform *> raytracer_transforms_ptr;

	std::vector <const Light *> lights;
	std::vector <const Transform *> light_transforms;

	bool dirty_raytracers = false;
	int raytracers_index = 0;

	// Iterate over all entities
	for (int i = 0; i < ecs.size(); i++) {
		if (ecs.exists <Camera> (i)) {
			camera = ecs.get <Camera> (i);
			camera_transform = ecs.get <Transform> (i);
			found_camera = true;
		}

		if (ecs.exists <kobra::Raytracer> (i)) {
			// TODO: account for changing transforms
			const kobra::Raytracer *raytracer = &ecs.get <kobra::Raytracer> (i);

			if (raytracers_index >= _c_raytracers.size())
				dirty_raytracers = true;
			else if (_c_raytracers[raytracers_index] != raytracer)
				dirty_raytracers = true;
			// TODO: also check for content changes in the component
			raytracer_transforms.push_back(ecs.get <Transform> (i));
			raytracer_transforms_ptr.push_back(&ecs.get <Transform> (i));
			raytracers.push_back(raytracer);
			raytracers_index++;
		}

		// TODO: check dirty lights
		if (ecs.exists <Light> (i)) {
			const Light *light = &ecs.get <Light> (i);

			if (light->type == Light::eArea) {
				lights.push_back(&ecs.get <Light> (i));
				light_transforms.push_back(&ecs.get <Transform> (i));
			}
		}
	}

	// Dirty means reset samples
	bool dirty = (_cached.camera_transform != camera_transform);
	if (dirty) {
		_cached.camera_transform = camera_transform;
		_accumulated = 0;
	}

	if (dirty_raytracers) {
		KOBRA_LOG_FILE(Log::INFO) << "Need to rebuild AS\n";
		_c_raytracers = raytracers;
		_c_transforms = raytracer_transforms;
		_cached.lights = lights;
		_cached.light_transforms = light_transforms;

		_cached.submeshes.clear();
		_cached.submesh_transforms.clear();

		for (int i = 0; i < _c_raytracers.size(); i++) {
			const kobra::Raytracer *r = _c_raytracers[i];
			for (const auto &s : r->get_mesh().submeshes) {
				_cached.submeshes.push_back(&s);
				_cached.submesh_transforms.push_back(raytracer_transforms_ptr[i]);
			}
		}

		_optix_build();
	}

	// Launch OptiX with the given camera
	_optix_update_materials();
	_optix_trace(camera, camera_transform);

	// Apply render area
	ra.apply(cmd, _ctx.extent);

	// Clear colors
	std::array <vk::ClearValue, 2> clear_values {
		vk::ClearValue {
			vk::ClearColorValue {
				std::array <float, 4> {0.0f, 0.0f, 0.0f, 1.0f}
			}
		},
		vk::ClearValue {
			vk::ClearDepthStencilValue {
				1.0f, 0
			}
		}
	};

	// Copy output to staging buffer
	_staging.upload(_output);

	// Copy staging buffer to image
	_result.transition_layout(cmd, vk::ImageLayout::eTransferDstOptimal);

	copy_data_to_image(cmd,
		_staging.buffer,
		_result.image,
		_result.format,
		width, height
	);

	// Transition image back to shader read
	_result.transition_layout(cmd, vk::ImageLayout::eShaderReadOnlyOptimal);

	// Start the render pass
	cmd.beginRenderPass(
		vk::RenderPassBeginInfo {
			*_render_pass,
			*framebuffer,
			vk::Rect2D {
				vk::Offset2D {0, 0},
				_ctx.extent
			},
			static_cast <uint32_t> (clear_values.size()),
			clear_values.data()
		},
		vk::SubpassContents::eInline
	);

	// Post process pipeline
	cmd.bindPipeline(
		vk::PipelineBindPoint::eGraphics,
		*_pipeline
	);

	// Bind descriptor set
	cmd.bindDescriptorSets(
		vk::PipelineBindPoint::eGraphics,
		*_ppl, 0, {*_ds_render}, {}
	);

	// Draw and end
	cmd.draw(6, 1, 0, 0);
	cmd.endRenderPass();
}

/////////////////////
// Private methods //
/////////////////////

// #define KOPTIX_DEBUG

void OptixTracer::_initialize_optix()
{
	// Storage for logs
	static char log[1024];
	static size_t sizeof_log = sizeof(log);

	// Initialize CUDA
	CUDA_CHECK( hipFree( 0 ) );

	// Initialize the OptiX API, loading all API entry points
	OPTIX_CHECK( optixInit() );

	// Specify context options
	OptixDeviceContextOptions options = {};
	options.logCallbackFunction       = &context_log_cb;
	options.logCallbackLevel          = 4;

	// Associate a CUDA context (and therefore a specific GPU) with this
	// device context
	hipCtx_t cuCtx = 0;  // zero means take the current context

	OPTIX_CHECK(optixDeviceContextCreate( cuCtx, &options, &_optix_ctx));

	// Create the OptiX module
	OptixPipelineCompileOptions pipeline_compile_options = {};

	{
		OptixModuleCompileOptions module_compile_options = {};

#ifdef KOPTIX_DEBUG

#pragma message "CUDA debug enabled"

		module_compile_options.optLevel   = OPTIX_COMPILE_OPTIMIZATION_LEVEL_0;
		module_compile_options.debugLevel = OPTIX_COMPILE_DEBUG_LEVEL_FULL;

#endif

		pipeline_compile_options.usesMotionBlur        = false;
		pipeline_compile_options.traversableGraphFlags = OPTIX_TRAVERSABLE_GRAPH_FLAG_ALLOW_SINGLE_GAS;
		pipeline_compile_options.traversableGraphFlags = OPTIX_TRAVERSABLE_GRAPH_FLAG_ALLOW_SINGLE_LEVEL_INSTANCING;
		pipeline_compile_options.numPayloadValues      = 3;
		pipeline_compile_options.numAttributeValues    = 3;

#ifdef KOPTIX_DEBUG

#pragma message"OptiX debug enabled"

		pipeline_compile_options.exceptionFlags = OPTIX_EXCEPTION_FLAG_DEBUG
			| OPTIX_EXCEPTION_FLAG_TRACE_DEPTH
			| OPTIX_EXCEPTION_FLAG_STACK_OVERFLOW;

#else

		pipeline_compile_options.exceptionFlags = OPTIX_EXCEPTION_FLAG_NONE;

#endif

		pipeline_compile_options.pipelineLaunchParamsVariableName = "params";
		pipeline_compile_options.usesPrimitiveTypeFlags = OPTIX_PRIMITIVE_TYPE_FLAGS_TRIANGLE;

		size_t      inputSize  = 0;
		std::string input = kobra::common::read_file("./bin/ptx/optix_rt.ptx");
		inputSize = input.size();

		size_t sizeof_log = sizeof( log );

		OPTIX_CHECK_LOG(optixModuleCreateFromPTX(
					_optix_ctx,
					&module_compile_options,
					&pipeline_compile_options,
					input.c_str(),
					inputSize,
					log,
					&sizeof_log,
					&_optix_module
					));
	}

	// Create program groups
	{
		// Default program group option
		OptixProgramGroupOptions program_group_options = {};

		// TODO: reate all at once
		OptixProgramGroupDesc raygen_program_desc = {
			.kind = OPTIX_PROGRAM_GROUP_KIND_RAYGEN,
			.raygen = {
				.module = _optix_module,
				.entryFunctionName = "__raygen__rg"
			}
		};

		OPTIX_CHECK_LOG(optixProgramGroupCreate(
			_optix_ctx,
			&raygen_program_desc, 1,
			&program_group_options,
			log, &sizeof_log,
			&_programs.raygen
		));

		// Miss programs
		OptixProgramGroupDesc miss_program_desc = {
			.kind = OPTIX_PROGRAM_GROUP_KIND_MISS,
			.miss = {
				.module = _optix_module,
			}
		};
	
		// Radiance miss program
		miss_program_desc.miss.entryFunctionName = "__miss__radiance";
		OPTIX_CHECK_LOG(optixProgramGroupCreate(
			_optix_ctx,
			&miss_program_desc, 1,
			&program_group_options,
			log, &sizeof_log,
			&_programs.miss_radiance
		));

		// Shadow miss program
		miss_program_desc.miss.entryFunctionName = "__miss__shadow";
		OPTIX_CHECK_LOG(optixProgramGroupCreate(
			_optix_ctx,
			&miss_program_desc, 1,
			&program_group_options,
			log, &sizeof_log,
			&_programs.miss_shadow
		));

		// Hit programs
		OptixProgramGroupDesc hitgroup_program_desc = {
			.kind = OPTIX_PROGRAM_GROUP_KIND_HITGROUP,
			.hitgroup = {
				.moduleCH = _optix_module,
			}
		};

		// Radiance hit program
		hitgroup_program_desc.hitgroup.entryFunctionNameCH = "__closesthit__radiance";
		OPTIX_CHECK_LOG(optixProgramGroupCreate(
			_optix_ctx,
			&hitgroup_program_desc, 1,
			&program_group_options,
			log, &sizeof_log,
			&_programs.hit_radiance
		));

		// Shadow hit program
		hitgroup_program_desc.hitgroup.entryFunctionNameCH = "__closesthit__shadow";
		OPTIX_CHECK_LOG(optixProgramGroupCreate(
			_optix_ctx,
			&hitgroup_program_desc, 1,
			&program_group_options,
			log, &sizeof_log,
			&_programs.hit_shadow
		));
	}

	//
	// Link pipeline
	//
	_optix_pipeline = nullptr;
	{
		const int max_trace_depth = 15;

		OptixProgramGroup program_groups[] = {
			_programs.raygen,
			_programs.hit_radiance,
			_programs.hit_shadow,
			_programs.miss_radiance,
			_programs.miss_shadow
		};

		OptixPipelineLinkOptions pipeline_link_options = {};
		
		pipeline_link_options.maxTraceDepth          = max_trace_depth;
		pipeline_link_options.debugLevel             = OPTIX_COMPILE_DEBUG_LEVEL_FULL;

		OPTIX_CHECK_LOG( optixPipelineCreate(
					_optix_ctx,
					&pipeline_compile_options,
					&pipeline_link_options,
					program_groups,
					sizeof( program_groups ) / sizeof( program_groups[0] ),
					log,
					&sizeof_log,
					&_optix_pipeline
					) );

		OptixStackSizes stack_sizes = {};
		for( auto& prog_group : program_groups )
		{
			OPTIX_CHECK( optixUtilAccumulateStackSizes( prog_group, &stack_sizes ) );
		}

		uint32_t direct_callable_stack_size_from_traversal;
		uint32_t direct_callable_stack_size_from_state;
		uint32_t continuation_stack_size;
		OPTIX_CHECK( optixUtilComputeStackSizes( &stack_sizes, max_trace_depth,
					0,  // maxCCDepth
					0,  // maxDCDEpth
					&direct_callable_stack_size_from_traversal,
					&direct_callable_stack_size_from_state, &continuation_stack_size ) );
		OPTIX_CHECK( optixPipelineSetStackSize( _optix_pipeline, direct_callable_stack_size_from_traversal,
					direct_callable_stack_size_from_state, continuation_stack_size,
					2  // maxTraversableDepth
					) );
	}

	/////////////////////////////////
	// Set up shader binding table //
	/////////////////////////////////

	// Ray generation
	hipDeviceptr_t  raygen_record;
	const size_t raygen_record_size = sizeof( RayGenSbtRecord );
	CUDA_CHECK( hipMalloc( reinterpret_cast<void**>( &raygen_record ), raygen_record_size ) );
	RayGenSbtRecord rg_sbt;
	OPTIX_CHECK( optixSbtRecordPackHeader(_programs.raygen, &rg_sbt ) );
	CUDA_CHECK( hipMemcpy(
				reinterpret_cast<void*>( raygen_record ),
				&rg_sbt,
				raygen_record_size,
				hipMemcpyHostToDevice
			      ) );

	// Ray miss records
	std::vector <MissSbtRecord> miss_sbt_records {
		MissSbtRecord {.data = {0.6f, 0.6f, 0.6f}}, {}
	};
		
	OPTIX_CHECK(optixSbtRecordPackHeader(
		_programs.miss_radiance,
		&miss_sbt_records[0]
	));

	OPTIX_CHECK(optixSbtRecordPackHeader(
		_programs.miss_shadow,
		&miss_sbt_records[1]
	));

	_optix_miss_sbt = cuda::make_buffer_ptr(miss_sbt_records);

	// Ray closest hit program
	_optix_hg_sbt = cuda::alloc(sizeof(HitGroupSbtRecord));
	
	HitGroupSbtRecord hg_sbt;

	OPTIX_CHECK(optixSbtRecordPackHeader(_programs.hit_radiance, &hg_sbt));
	cuda::copy(_optix_hg_sbt, &hg_sbt, 1);

	_optix_sbt = OptixShaderBindingTable {};
	_optix_sbt.raygenRecord                = raygen_record;
	_optix_sbt.missRecordBase              = _optix_miss_sbt;
	_optix_sbt.missRecordStrideInBytes     = sizeof(MissSbtRecord);
	_optix_sbt.missRecordCount             = 2;
	_optix_sbt.hitgroupRecordBase          = _optix_hg_sbt;
	_optix_sbt.hitgroupRecordStrideInBytes = sizeof(HitGroupSbtRecord);
	_optix_sbt.hitgroupRecordCount         = 1;

	// Optix denoiser
	OptixDenoiserOptions denoiser_options = {};
	denoiser_options.guideAlbedo = 1;
	denoiser_options.guideNormal = 1;

	OPTIX_CHECK(optixDenoiserCreate(_optix_ctx,
		OPTIX_DENOISER_MODEL_KIND_AOV,
		&denoiser_options,
		&_optix_denoiser
	));

	// Optix denoiser size
	OptixDenoiserSizes denoiser_sizes;
	OPTIX_CHECK(optixDenoiserComputeMemoryResources(
		_optix_denoiser,
		width, height,
		&denoiser_sizes
	));

	int scratch_size = std::max(
		denoiser_sizes.withOverlapScratchSizeInBytes,
		denoiser_sizes.withoutOverlapScratchSizeInBytes
	);

	_buffers.denoiser_state = std::move(cuda::BufferData(denoiser_sizes.stateSizeInBytes));
	_buffers.denoiser_scratch = std::move(cuda::BufferData(scratch_size));

	// Create stream for OptiX
	CUDA_CHECK(hipStreamCreate(&_optix_stream));

	// Set up denoiser
	OPTIX_CHECK(optixDenoiserSetup(_optix_denoiser, _optix_stream,
		width, height,
		_buffers.denoiser_state.dev(),
		_buffers.denoiser_state.size(),
		_buffers.denoiser_scratch.dev(),
		_buffers.denoiser_scratch.size()
	));

	KOBRA_LOG_FUNC(Log::OK) << "Initialized OptiX and relevant structures" << std::endl;
}

// TODO: also add an optix_update method
void OptixTracer::_optix_build()
{
	// Use default options for simplicity.  In a real use case we would want to
	// enable compaction, etc
	OptixAccelBuildOptions gas_accel_options = {};
	gas_accel_options.buildFlags = OPTIX_BUILD_FLAG_ALLOW_COMPACTION;
	gas_accel_options.operation  = OPTIX_BUILD_OPERATION_BUILD;

	std::vector <OptixTraversableHandle> instance_gas(_cached.submeshes.size());
	std::vector <OptixTraversableHandle> light_gas(_cached.lights.size());
	
	// Flags
	const uint32_t triangle_input_flags[1] = {OPTIX_GEOMETRY_FLAG_NONE};

	KOBRA_LOG_FUNC(Log::INFO) << "Building GAS for instances (# = "
		<< _cached.submeshes.size() << ")" << std::endl;

	// TODO: CACHE the vertices for the sbts

	for (int i = 0; i < _cached.submeshes.size(); i++) {
		const Submesh *s = _cached.submeshes[i];

		// Prepare submesh vertices and triangles
		std::vector <float3> vertices;
		std::vector <uint3> triangles;
		
		// TODO: method to generate accel handle from cuda buffers
		for (int j = 0; j < s->indices.size(); j += 3) {
			triangles.push_back({
				s->indices[j],
				s->indices[j + 1],
				s->indices[j + 2]
			});
		}

		for (int j = 0; j < s->vertices.size(); j++) {
			auto p = s->vertices[j].position;
			vertices.push_back(to_f3(p));
		}

		// Create the build input
		OptixBuildInput build_input {};

		build_input.type = OPTIX_BUILD_INPUT_TYPE_TRIANGLES;

		hipDeviceptr_t d_vertices = cuda::make_buffer_ptr(vertices);
		hipDeviceptr_t d_triangles = cuda::make_buffer_ptr(triangles);

		OptixBuildInputTriangleArray &triangle_array = build_input.triangleArray;
		triangle_array.vertexFormat	= OPTIX_VERTEX_FORMAT_FLOAT3;
		triangle_array.numVertices	= vertices.size();
		triangle_array.vertexBuffers	= &d_vertices;

		triangle_array.indexFormat	= OPTIX_INDICES_FORMAT_UNSIGNED_INT3;
		triangle_array.numIndexTriplets	= triangles.size();
		triangle_array.indexBuffer	= d_triangles;

		triangle_array.flags		= triangle_input_flags;

		// SBT record properties
		triangle_array.numSbtRecords	= 1;
		triangle_array.sbtIndexOffsetBuffer = 0;
		triangle_array.sbtIndexOffsetStrideInBytes = 0;
		triangle_array.sbtIndexOffsetSizeInBytes = 0;

		// Build GAS
		hipDeviceptr_t d_gas_output;
		hipDeviceptr_t d_gas_tmp;

		OptixAccelBufferSizes gas_buffer_sizes;
		OPTIX_CHECK(optixAccelComputeMemoryUsage(
			_optix_ctx, &gas_accel_options,
			&build_input, 1,
			&gas_buffer_sizes
		));
		
		KOBRA_LOG_FUNC(Log::INFO) << "GAS buffer sizes: " << gas_buffer_sizes.tempSizeInBytes
			<< " " << gas_buffer_sizes.outputSizeInBytes << std::endl;

		d_gas_output = cuda::alloc(gas_buffer_sizes.outputSizeInBytes);
		d_gas_tmp = cuda::alloc(gas_buffer_sizes.tempSizeInBytes);

		OptixTraversableHandle handle;
		OPTIX_CHECK(optixAccelBuild(_optix_ctx,
			0, &gas_accel_options,
			&build_input, 1,
			d_gas_tmp, gas_buffer_sizes.tempSizeInBytes,
			d_gas_output, gas_buffer_sizes.outputSizeInBytes,
			&handle, nullptr, 0
		));

		instance_gas[i] = handle;

		// Free data at the end
		cuda::free(d_vertices);
		cuda::free(d_triangles);

		cuda::free(d_gas_tmp);
	}

	// Lights (a cube for now)
	Mesh box = Mesh::box({0, 0, 0}, {0.5, 0.01, 0.5});

	std::vector <float3> vertices;
	std::vector <uint3> triangles;

	for (auto s : box.submeshes) {
		for (int j = 0; j < s.indices.size(); j += 3) {
			triangles.push_back({
				s.indices[j],
				s.indices[j + 1],
				s.indices[j + 2]
			});
		}

		for (int j = 0; j < s.vertices.size(); j++) {
			auto p = s.vertices[j].position;
			vertices.push_back(to_f3(p));
		}
	}

	hipDeviceptr_t d_vertices = cuda::make_buffer_ptr(vertices);
	hipDeviceptr_t d_triangles = cuda::make_buffer_ptr(triangles);

	// Prepare the instance transform
	for (int i = 0; i < _cached.lights.size(); i++) {
		// Create the build input
		OptixBuildInput build_input {};

		build_input.type = OPTIX_BUILD_INPUT_TYPE_TRIANGLES;

		OptixBuildInputTriangleArray &triangle_array = build_input.triangleArray;
		triangle_array.vertexFormat	= OPTIX_VERTEX_FORMAT_FLOAT3;
		triangle_array.numVertices	= vertices.size();
		triangle_array.vertexBuffers	= &d_vertices;

		triangle_array.indexFormat	= OPTIX_INDICES_FORMAT_UNSIGNED_INT3;
		triangle_array.numIndexTriplets	= triangles.size();
		triangle_array.indexBuffer	= d_triangles;

		triangle_array.flags		= triangle_input_flags;

		// SBT record properties
		triangle_array.numSbtRecords	= 1;
		triangle_array.sbtIndexOffsetBuffer = 0;
		triangle_array.sbtIndexOffsetStrideInBytes = 0;
		triangle_array.sbtIndexOffsetSizeInBytes = 0;

		// Build GAS
		hipDeviceptr_t d_gas_output;
		hipDeviceptr_t d_gas_tmp;

		OptixAccelBufferSizes gas_buffer_sizes;
		OPTIX_CHECK(optixAccelComputeMemoryUsage(
			_optix_ctx, &gas_accel_options,
			&build_input, 1,
			&gas_buffer_sizes
		));
		
		KOBRA_LOG_FUNC(Log::INFO) << "Light GAS buffer sizes: " << gas_buffer_sizes.tempSizeInBytes
			<< " " << gas_buffer_sizes.outputSizeInBytes << std::endl;

		d_gas_output = cuda::alloc(gas_buffer_sizes.outputSizeInBytes);
		d_gas_tmp = cuda::alloc(gas_buffer_sizes.tempSizeInBytes);

		OptixTraversableHandle handle;
		OPTIX_CHECK(optixAccelBuild(_optix_ctx,
			0, &gas_accel_options,
			&build_input, 1,
			d_gas_tmp, gas_buffer_sizes.tempSizeInBytes,
			d_gas_output, gas_buffer_sizes.outputSizeInBytes,
			&handle, nullptr, 0
		));

		light_gas[i] = handle;

		// Free data at the end
		cuda::free(d_gas_tmp);
	}

	// Build instances and top level acceleration structure
	std::vector <OptixInstance> instances;

	for (int i = 0; i < _cached.submeshes.size(); i++) {
		glm::mat4 mat = _cached.submesh_transforms[i]->matrix();

		float transform[12] = {
			mat[0][0], mat[1][0], mat[2][0], mat[3][0],
			mat[0][1], mat[1][1], mat[2][1], mat[3][1],
			mat[0][2], mat[1][2], mat[2][2], mat[3][2]
		};

		OptixInstance instance {};
		memcpy(instance.transform, transform, sizeof(float) * 12);

		// Set the instance handle
		instance.traversableHandle = instance_gas[i];
		instance.visibilityMask = 0b1;
		instance.sbtOffset = i;

		instances.push_back(instance);
	}

	for (int i = 0; i < _cached.lights.size(); i++) {
		// Prepare the instance transform
		glm::mat4 mat = _cached.light_transforms[i]->matrix();

		float transform[12] = {
			mat[0][0], mat[1][0], mat[2][0], mat[3][0],
			mat[0][1], mat[1][1], mat[2][1], mat[3][1],
			mat[0][2], mat[1][2], mat[2][2], mat[3][2]
		};

		OptixInstance instance {};
		memcpy(instance.transform, transform, sizeof(float) * 12);

		// Set the instance handle
		instance.traversableHandle = light_gas[i];
		instance.visibilityMask = 0b10;
		instance.sbtOffset = i + _cached.submeshes.size();

		instances.push_back(instance);
	}

	// Create top level acceleration structure
	hipDeviceptr_t d_instances = cuda::make_buffer_ptr(instances);

	// TLAS for objects and lights
	{
		OptixBuildInput ias_build_input {};
		ias_build_input.type = OPTIX_BUILD_INPUT_TYPE_INSTANCES;
		ias_build_input.instanceArray.instances = d_instances;
		ias_build_input.instanceArray.numInstances = instances.size();

		// IAS options
		OptixAccelBuildOptions ias_accel_options {};
		ias_accel_options.buildFlags = OPTIX_BUILD_FLAG_ALLOW_COMPACTION;
		ias_accel_options.operation = OPTIX_BUILD_OPERATION_BUILD;

		// IAS buffer sizes
		OptixAccelBufferSizes ias_buffer_sizes;
		OPTIX_CHECK(optixAccelComputeMemoryUsage(
			_optix_ctx, &ias_accel_options,
			&ias_build_input, 1,
			&ias_buffer_sizes
		));

		KOBRA_LOG_FUNC(Log::INFO) << "IAS buffer sizes: " << ias_buffer_sizes.tempSizeInBytes << " " << ias_buffer_sizes.outputSizeInBytes << std::endl;

		// Allocate the IAS
		hipDeviceptr_t d_ias_output = cuda::alloc(ias_buffer_sizes.outputSizeInBytes);
		hipDeviceptr_t d_ias_tmp = cuda::alloc(ias_buffer_sizes.tempSizeInBytes);

		// Build the IAS
		OPTIX_CHECK(optixAccelBuild(_optix_ctx,
			0, &ias_accel_options,
			&ias_build_input, 1,
			d_ias_tmp, ias_buffer_sizes.tempSizeInBytes,
			d_ias_output, ias_buffer_sizes.outputSizeInBytes,
			&_optix_traversable, nullptr, 0
		));

		cuda::free(d_ias_tmp);
		cuda::free(d_instances);
	}
}

// Update hit group data with materials
// TODO: also update if transforms change
// TODO: refactor to sbts
void OptixTracer::_optix_update_materials()
{
	static std::vector <HitGroupSbtRecord> hg_sbts;
	static std::vector <optix_rt::QuadLight> quad_lights;
	static std::vector <optix_rt::TriangleLight> triangle_lights;

	// Update quad lights
	if (quad_lights.size() != _cached.lights.size()) {
		quad_lights.resize(_cached.lights.size());

		for (int i = 0; i < quad_lights.size(); i++) {
			const Light *light = _cached.lights[i];
			const Transform *transform = _cached.light_transforms[i];
			
			glm::vec3 a {-0.5f, 0, -0.5f};
			glm::vec3 b {0.5f, 0, -0.5f};
			glm::vec3 c {-0.5f, 0, 0.5f};

			a = transform->apply(a);
			b = transform->apply(b);
			c = transform->apply(c);

			quad_lights[i].a = to_f3(a);
			quad_lights[i].ab = to_f3(b - a);
			quad_lights[i].ac = to_f3(c - a);
			quad_lights[i].intensity
				= to_f3(light->power * light->color);
		}

		KOBRA_LOG_FUNC(Log::INFO) << "Number of area lights: " << quad_lights.size() << std::endl;

		_buffers.quad_lights = (hipDeviceptr_t) cuda::make_buffer(quad_lights);
	}

	// Update triangle lights
	{
		int count = 0;

		std::vector <const Submesh *> emissive_submeshes;
		for (const Submesh *s : _cached.submeshes) {
			if (s->material.type == eEmissive) {
				emissive_submeshes.push_back(s);
				count += s->triangles();
			}
		}

		if (count != triangle_lights.size()) {
			triangle_lights.clear();

			int i = 0;
			for (const Submesh *s : emissive_submeshes) {
				const Transform *transform =
					_cached.submesh_transforms[i];

				for (int j = 0; j < s->triangles(); j++) {
					uint32_t i0 = s->indices[j * 3 + 0];
					uint32_t i1 = s->indices[j * 3 + 1];
					uint32_t i2 = s->indices[j * 3 + 2];

					glm::vec3 a = transform->apply(s->vertices[i0].position);
					glm::vec3 b = transform->apply(s->vertices[i1].position);
					glm::vec3 c = transform->apply(s->vertices[i2].position);

					optix_rt::TriangleLight light;
					light.a = to_f3(a);
					light.ab = to_f3(b - a);
					light.ac = to_f3(c - a);
					light.intensity = to_f3(s->material.emission);

					triangle_lights.push_back(light);
				}
			}

			// Upload to GPU
			_buffers.tri_lights = (hipDeviceptr_t) cuda::make_buffer(triangle_lights);
		}
	}

	// Update hit records if necessary
	int required_size = 2 * (_cached.submeshes.size() + _cached.lights.size());
	if (hg_sbts.size() != required_size) {
		hg_sbts.clear();

		// Regular raytracers (submeshes)
		for (int i = 0; i < _cached.submeshes.size(); i++) {
			const Submesh *submesh = _cached.submeshes[i];
			Material mat = submesh->material;

			// Material
			cuda::Material material;
			material.diffuse = to_f3(mat.diffuse);
			material.specular = to_f3(mat.specular);
			material.emission = to_f3(mat.emission);
			material.ambient = to_f3(mat.ambient);
			material.shininess = mat.shininess;
			material.roughness = mat.roughness;
			material.refraction = mat.refraction;
			material.type = mat.type;

			HitGroupSbtRecord hg_sbt {};
			hg_sbt.data.material = material;

			generate_submesh_data(*submesh,
					*_cached.submesh_transforms[i], hg_sbt.data);

			// Import textures if necessary
			// TODO: method?
			if (mat.has_albedo()) {
				const ImageData &diffuse = TextureManager::load_texture(
					_ctx.dev(), mat.albedo_texture
				);

				hg_sbt.data.textures.diffuse = import_vulkan_texture(*_ctx.device, diffuse);
				hg_sbt.data.textures.has_diffuse = true;
			}

			if (mat.has_normal()) {
				const ImageData &normal = TextureManager::load_texture(
					_ctx.dev(), mat.normal_texture
				);

				hg_sbt.data.textures.normal = import_vulkan_texture(*_ctx.device, normal);
				hg_sbt.data.textures.has_normal = true;
			}

			if (mat.has_roughness()) {
				const ImageData &roughness = TextureManager::load_texture(
					_ctx.dev(), mat.roughness_texture
				);

				hg_sbt.data.textures.roughness = import_vulkan_texture(*_ctx.device, roughness);
				hg_sbt.data.textures.has_roughness = true;
			}

			// Lights
			hg_sbt.data.quad_lights = (optix_rt::QuadLight *) _buffers.quad_lights;
			hg_sbt.data.n_quad_lights = quad_lights.size();

			hg_sbt.data.tri_lights = (optix_rt::TriangleLight *) _buffers.tri_lights;
			hg_sbt.data.n_tri_lights = triangle_lights.size();

			OPTIX_CHECK(optixSbtRecordPackHeader(_programs.hit_radiance, &hg_sbt));
			hg_sbts.push_back(hg_sbt);
		}

		// Area lights
		for (int i = 0; i < _cached.lights.size(); i++) {
			HitGroupSbtRecord hg_sbt {};
			hg_sbt.data.quad_lights = (optix_rt::QuadLight *) _buffers.quad_lights;
			hg_sbt.data.n_quad_lights = 1;

			hg_sbt.data.tri_lights = (optix_rt::TriangleLight *) _buffers.tri_lights;
			hg_sbt.data.n_tri_lights = 1;

			hg_sbt.data.material.emission = to_f3(_cached.lights[i]->color);
			hg_sbt.data.material.type = Shading::eEmissive;

			OPTIX_CHECK(optixSbtRecordPackHeader(_programs.hit_radiance, &hg_sbt));
			hg_sbts.push_back(hg_sbt);
		}

		// Duplicate the SBTs for the shadow program
		// TODO: delete hit shadow
		int size = hg_sbts.size();
		for (int i = 0; i < size; i++) {
			HitGroupSbtRecord hg_sbt = hg_sbts[i];
			OPTIX_CHECK(optixSbtRecordPackHeader(_programs.hit_shadow, &hg_sbt));
			hg_sbts.push_back(hg_sbt);
		}

		_optix_hg_sbt = (hipDeviceptr_t) cuda::make_buffer(hg_sbts);

		// Update SBT
		_optix_sbt.hitgroupRecordBase = _optix_hg_sbt;
		_optix_sbt.hitgroupRecordCount = hg_sbts.size();
		_optix_sbt.hitgroupRecordStrideInBytes = sizeof(HitGroupSbtRecord);

		KOBRA_LOG_FILE(Log::INFO) << "OptiX: SBT updated\n";
	}
}

// Taken from nvidia's book
static void generate_halton_sequence(int N, int b, std::vector <float> &dst)
{
	int n = 0;
	int d = 1;

	for (int i = 0; i < N; i++) {
		int x = d - n;
		if (x == 1) {
			n = 1;
			d *= b;
		} else {
			int y = d/b;
			while (x <= y)
				y /= b;
			n = (b + 1) * y - x;
		}

		dst[i] = (float) n / (float) d;
	}
}

static void generate_pixel_offsets(int N, std::vector <float> &x, std::vector <float> &y)
{
	static std::vector <int> bases {2, 3, 5, 7, 11, 13};

	srand(time(0));
	int r1 = rand() % bases.size();
	int r2 = rand() % bases.size();

	if (r1 == r2)
		r2 = (r2 + 1) % bases.size();

	int b1 = bases[r1];
	int b2 = bases[r2];

	generate_halton_sequence(N, b1, x);
	generate_halton_sequence(N, b2, y);

	for (int i = 0; i < N; i++) {
		x[i] -= 0.5f;
		y[i] -= 0.5f;
	}
}

// Tone maps:
//	0 for sRGB
//	1 for ACES
__global__ void compute_pixel_values(float4 *pixels, uint32_t *target,
		int width, int height, int tonemapping = 0)
{
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;

	if (x >= width || y >= height)
		return;

	int in_idx = y * width + x;
	int out_idx = (height - y - 1) * width + x;

	float4 pixel = pixels[in_idx];
	uchar4 color = cuda::make_color(pixel, tonemapping);
	target[out_idx] = to_ui32(color);
}

void OptixTracer::_optix_trace(const Camera &camera, const Transform &transform)
{
	// TODO: refresh every x samples
	static bool first = true;

	// TODO: generate offsets in GPU
	if (first) {
		first = false;
		generate_pixel_offsets(width * height, _buffers.h_xoffsets, _buffers.h_yoffsets);
		cuda::copy(_buffers.xoffset, _buffers.h_xoffsets);
		cuda::copy(_buffers.yoffset, _buffers.h_yoffsets);
	}

	// TODO: keep a persistent param object (and only update what's needed)

	// Settings parameters
	optix_rt::Params params;

	params.spp = samples_per_pixel;
	
	params.pbuffer = (float4 *) _buffers.pbuffer;
	params.nbuffer = (float4 *) _buffers.nbuffer;
	params.abuffer = (float4 *) _buffers.abuffer;
	
	params.xoffset = (float *) _buffers.xoffset;
	params.yoffset = (float *) _buffers.yoffset;

	params.image_width  = width;
	params.image_height = height;

	params.accumulated = _accumulated++;
	params.instances = _cached.submeshes.size() + _cached.lights.size();

	params.handle = _optix_traversable;

	auto uvw = kobra::uvw_frame(camera, transform);

	params.cam_eye      = to_f3(transform.position);

	params.cam_u = to_f3(uvw.u);
	params.cam_v = to_f3(uvw.v);
	params.cam_w = to_f3(uvw.w);

	float ms = timer.elapsed_start();

	params.time = sin(ms * 12.3243f) * cos(1 - ms * 0.123f);

	/// Allocate on the GPU
	hipDeviceptr_t d_param;
	CUDA_CHECK( hipMalloc( reinterpret_cast<void**> (&d_param),
				sizeof(optix_rt::Params)));
	CUDA_CHECK( hipMemcpy(
				reinterpret_cast<void*>( d_param ),
				&params, sizeof( params ),
				hipMemcpyHostToDevice
			      ) );

	OPTIX_CHECK(optixLaunch(_optix_pipeline, _optix_stream, d_param,
				sizeof(optix_rt::Params), &_optix_sbt,
				width, height, 1));
	CUDA_SYNC_CHECK();

	CUDA_CHECK( hipFree( reinterpret_cast<void*>( d_param ) ) );

	hipDeviceptr_t d_result = 0;

	if (denoiser_enabled) {
		// Denoise
		OptixImage2D color_input;
		color_input.data = _buffers.pbuffer;
		color_input.width = width;
		color_input.height = height;
		color_input.rowStrideInBytes = width * sizeof(float4);
		color_input.pixelStrideInBytes = sizeof(float4);
		color_input.format = OPTIX_PIXEL_FORMAT_FLOAT4;

		OptixImage2D normal_input;
		normal_input.data = _buffers.nbuffer;
		normal_input.width = width;
		normal_input.height = height;
		normal_input.rowStrideInBytes = width * sizeof(float4);
		normal_input.pixelStrideInBytes = sizeof(float4);
		normal_input.format = OPTIX_PIXEL_FORMAT_FLOAT4;

		OptixImage2D albedo_input;
		albedo_input.data = _buffers.abuffer;
		albedo_input.width = width;
		albedo_input.height = height;
		albedo_input.rowStrideInBytes = width * sizeof(float4);
		albedo_input.pixelStrideInBytes = sizeof(float4);
		albedo_input.format = OPTIX_PIXEL_FORMAT_FLOAT4;

		OptixImage2D output;
		output.data = _buffers.fbuffer;
		output.width = width;
		output.height = height;
		output.rowStrideInBytes = width * sizeof(float4);
		output.pixelStrideInBytes = sizeof(float4);
		output.format = OPTIX_PIXEL_FORMAT_FLOAT4;

		// Invoke the denoiser
		OptixDenoiserParams denoiser_params = {};

		OptixDenoiserGuideLayer guide_layer;
		guide_layer.normal = normal_input;
		guide_layer.albedo = albedo_input;

		OptixDenoiserLayer layer;
		layer.input = color_input;
		layer.output = output;

		OPTIX_CHECK(optixDenoiserInvoke(_optix_denoiser, _optix_stream,
			&denoiser_params,
			_buffers.denoiser_state.dev(),
			_buffers.denoiser_state.size(),
			&guide_layer,
			&layer, 1,
			0, 0,
			_buffers.denoiser_scratch.dev(),
			_buffers.denoiser_scratch.size()
		));

		d_result = _buffers.fbuffer;
	} else {
		d_result = _buffers.pbuffer;
	}
	
	// Conversion kernel
	dim3 block(16, 16);
	dim3 grid((width + block.x - 1)/block.x, (height + block.y - 1)/block.y);
	compute_pixel_values <<<grid, block>>>
		((float4 *) d_result, (uint32_t *) _buffers.truncated,
		 width, height, tonemapping);

	// TODO: multithread
	if (_output.size() != width * height)
		_output.resize(width * height);

	cuda::copy(_output, _buffers.truncated, width * height);
}

}

}
