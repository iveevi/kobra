#include "hip/hip_runtime.h"
// Engine headers
#include "include/amadeus/path_tracer.cuh"
#include "include/cuda/core.cuh"
#include "include/cuda/random.cuh"
#include "include/optix/core.cuh"

// Editor headers
#include "path_tracer.cuh"

extern "C" {

__constant__ PathTracerParameters parameters;

}

// Aliasing
using namespace kobra;
using namespace kobra::cuda;
using namespace kobra::optix;

struct Packet {
        float3 x;
        float3 n;
        float2 uv;
        bool miss;
        bool entering;
        int64_t id;
};

struct LightInfo {
        float3 position;
        float3 normal;
        float3 emission;
        float area;
};

__device__
void make_ray(uint3 idx, float3 &direction, float3 &seed)
{
	// Jittered halton
	seed = make_float3(idx.x, idx.y, 0);

        float xoff = rand_uniform(seed) - 0.5f;
        float yoff = rand_uniform(seed) - 0.5f;

	// Compute ray origin and direction
        idx.y = parameters.resolution.y - (idx.y + 1);

        float2 d = 2.0f * make_float2(
		float(idx.x + xoff)/parameters.resolution.x,
		float(idx.y + yoff)/parameters.resolution.y
	) - 1.0f;

	direction = normalize(d.x * parameters.U - d.y * parameters.V + parameters.W);
}

__device__
float sample_light(LightInfo &light_info, float3 &seed)
{
        // Choose light
        uint light_index = cuda::rand_uniform(parameters.area.count, seed);

        // Choose triangle
        AreaLight light = parameters.area.lights[light_index];
        light_info.emission = light.emission;

        uint triangle_index = cuda::rand_uniform(light.triangles, seed);
        uint3 triangle = light.indices[triangle_index];

        glm::vec3 v0 = light.vertices[triangle.x].position;
        glm::vec3 v1 = light.vertices[triangle.y].position;
        glm::vec3 v2 = light.vertices[triangle.z].position;

        v0 = light.model * glm::vec4(v0, 1.0f);
        v1 = light.model * glm::vec4(v1, 1.0f);
        v2 = light.model * glm::vec4(v2, 1.0f);

        glm::vec3 gnormal = glm::normalize(glm::cross(v1 - v0, v2 - v0));

        light_info.normal = make_float3(gnormal.x, gnormal.y, gnormal.z);
        light_info.area = glm::length(glm::cross(v1 - v0, v2 - v0));

        // Sample point on triangle
        float3 bary = cuda::pcg3f(seed);

        float u = bary.x;
        float v = bary.y;
        if (u + v > 1.0f) {
                u = 1.0f - u;
                v = 1.0f - v;
        }

        glm::vec3 gpoint = v0 * (1.0f - u - v) + v1 * u + v2 * v;
        light_info.position = make_float3(gpoint.x, gpoint.y, gpoint.z);

        // Return the pdf of sampling this point light
        return 1.0f / (light_info.area * parameters.area.count);
}

__device__
void convert_material(const cuda::_material &src, cuda::Material &dst, float2 uv)
{
        dst.diffuse = src.diffuse;
        dst.specular = src.specular;
        dst.emission = src.emission;
        dst.roughness = src.roughness;
        dst.refraction = src.refraction;
        dst.type = src.type;

        if (src.textures.has_diffuse) {
                float4 diffuse = tex2D <float4> (src.textures.diffuse, uv.x, uv.y);
                dst.diffuse = make_float3(diffuse);
        }
}

__device__
float3 radiance(const SurfaceHit &sh, float3 &seed, int depth)
{
        LightInfo light_info;
        float light_pdf = sample_light(light_info, seed);

        float3 direction = normalize(light_info.position - sh.x);
        float3 origin = sh.x;

        float sign = (sh.mat.type == eTransmission) ? -1.0f : 1.0f;
        if (isnan(sh.n.x) || isnan(sh.n.y) || isnan(sh.n.z))
                origin += sign * direction * 1e-3f;
        else
                origin += sign * sh.n * 1e-3f;
        
        uint i0 = 0;
        uint i1 = 0;

        Packet packet;
        packet.miss = false;

        pack_pointer(&packet, i0, i1);

        optixTrace(
                parameters.handle,
                origin, direction,
                0.0f, length(light_info.position - origin) - 1e-3f, 0.0f,
                OptixVisibilityMask(0xFF),
                OPTIX_RAY_FLAG_DISABLE_ANYHIT
                | OPTIX_RAY_FLAG_DISABLE_CLOSESTHIT
                | OPTIX_RAY_FLAG_TERMINATE_ON_FIRST_HIT,
                // TODO: use miss instead of hit and disable closest hit, etc
                0, 1, 0, i0, i1
        );

        float3 out_radiance = sh.mat.emission;
        if (packet.miss && length(out_radiance) < 1e-3f) {
                float3 wi = normalize(light_info.position - sh.x);
                float R = length(light_info.position - sh.x);
	
                float3 brdf = cuda::brdf(sh, wi, sh.mat.type);
                out_radiance = brdf * light_info.emission * abs(dot(light_info.normal, wi)) * abs(dot(sh.n, wi))/(light_pdf * R * R);
        }

        return out_radiance;
}

extern "C" __global__ void __raygen__()
{
	// Get the launch index
	const uint3 idx = optixGetLaunchIndex();
       
        float4 raw_position;
        surf2Dread(&raw_position, parameters.position_surface, idx.x * sizeof(float4),
                parameters.resolution.y - (idx.y + 1));
        
        float4 raw_normal;
        surf2Dread(&raw_normal, parameters.normal_surface, idx.x * sizeof(float4),
                parameters.resolution.y - (idx.y + 1));

        float4 raw_uv;
        surf2Dread(&raw_uv, parameters.uv_surface, idx.x * sizeof(float4),
                parameters.resolution.y - (idx.y + 1));

        int32_t raw_index;
        surf2Dread(&raw_index, parameters.index_surface, idx.x * sizeof(int32_t),
                parameters.resolution.y - (idx.y + 1));

        int32_t triangle_id = raw_index >> 16;
        int32_t material_id = raw_index & 0xFFFF;
        
        int index = idx.x + idx.y * parameters.resolution.x;
        if (raw_index == -1) {
                parameters.color[index] = make_float4(0.5f, 0.5f, 0.5f, 1.0f);
                return;
        }

        float3 position = { raw_position.x, raw_position.y, raw_position.z };
        float3 normal = { raw_normal.x, raw_normal.y, raw_normal.z };
        float2 uv = { raw_uv.x, raw_uv.y };

        // Correct the normal
        float3 ray = position - parameters.origin;
        if (dot(ray, normal) > 0.0f)
                normal = -normal;

        float3 seed = make_float3(idx.x, idx.y, parameters.time);
        
        cuda::_material m = parameters.materials[material_id];
        
        SurfaceHit sh;
        sh.x = position; // TODO: offset...
        sh.wo = normalize(parameters.origin - position);
        sh.n = normalize(normal);
        sh.entering = false;

        convert_material(m, sh.mat, uv);

        // float3 color = radiance(sh, seed, 0);
        float3 color = make_float3(0.0f);
        float3 beta = make_float3(1.0f);

        static constexpr int MAX_DEPTH = 8;
        for (int depth = 0; depth < MAX_DEPTH; depth++) {
                color += beta * radiance(sh, seed, depth);
                
                float3 wi;
                float pdf;
                Shading out;

                float3 brdf = eval(sh, wi, pdf, out, seed);
                if (pdf > 0.0 && depth < MAX_DEPTH - 1) {
                        Packet packet;
                        packet.miss = false;
                        packet.entering = false;

                        float sign = (sh.mat.type == eTransmission) ? -1.0f : 1.0f;
                        float3 origin = sh.x + sign * sh.n * 1e-3f;

                        uint i0;
                        uint i1;

                        pack_pointer(&packet, i0, i1);

                        optixTrace(
                                parameters.handle,
                                origin, wi,
                                0.0f, 1e16f, 0.0f,
                                OptixVisibilityMask(0xFF),
                                OPTIX_RAY_FLAG_DISABLE_ANYHIT,
                                // TODO: use miss instead of hit and disable closest hit, etc
                                0, 1, 0, i0, i1
                        );

                        if (!packet.miss) {
                                cuda::_material m = parameters.materials[packet.id];
                                
                                sh.x = packet.x;
                                sh.wo = -wi;
                                sh.n = packet.n;
                                sh.entering = false;

                                convert_material(m, sh.mat, packet.uv);

                                beta *= brdf * abs(dot(sh.n, wi)) / pdf;
                        } else {
                                break;
                        }
                } else {
                        break;
                }
        }

        // Store color
        parameters.color[index] = make_float4(color);

        // TODO: if shadow ray fails (e.g. hits a surface), cache that
        // information and use it to accumulate radiance (or skip new ray
        // altogether...)
}

extern "C" __global__ void __closesthit__()
{
        uint i0 = optixGetPayload_0();
        uint i1 = optixGetPayload_1();

        Packet *packet = unpack_pointer <Packet> (i0, i1);

        // packet->miss = false;
        ::Hit *hit = (::Hit *) optixGetSbtDataPointer();

        // Indices
        int32_t mat_id = hit->index;
        int32_t tri_id = optixGetPrimitiveIndex();
        packet->id = mat_id;

        // Compute position and normal
        float2 bary = optixGetTriangleBarycentrics();

        float bu = bary.x;
        float bv = bary.y;
        float bw = 1.0f - bu - bv;

        uint3 triangle = hit->triangles[tri_id];
        Vertex v0 = hit->vertices[triangle.x];
        Vertex v1 = hit->vertices[triangle.y];
        Vertex v2 = hit->vertices[triangle.z];

        float2 uv0 = { v0.tex_coords.x, v0.tex_coords.y };
        float2 uv1 = { v1.tex_coords.x, v1.tex_coords.y };
        float2 uv2 = { v2.tex_coords.x, v2.tex_coords.y };
        float2 uv = bw * uv0 + bu * uv1 + bv * uv2;

        packet->uv = { uv.x, uv.y };

        glm::vec3 glm_pos = bw * v0.position + bu * v1.position + bv * v2.position;
        glm_pos = hit->model * glm::vec4(glm_pos, 1.0f);

        packet->x = { glm_pos.x, glm_pos.y, glm_pos.z };

        // Compute normal
        glm::vec3 e1 = v1.position - v0.position;
        glm::vec3 e2 = v2.position - v0.position;

        e1 = hit->model * glm::vec4(e1, 0.0f);
        e2 = hit->model * glm::vec4(e2, 0.0f);

        glm::vec3 glm_normal = glm::normalize(glm::cross(e1, e2));

        // Shading normal
        glm::vec3 glm_shading_normal = bw * v0.normal + bu * v1.normal + bv * v2.normal;
        glm_shading_normal = hit->model * glm::vec4(glm_shading_normal, 0.0f);

        if (glm::dot(glm_normal, glm_shading_normal) < 0.0f)
                glm_shading_normal = -glm_shading_normal;

        float3 normal = { glm_shading_normal.x, glm_shading_normal.y, glm_shading_normal.z };
        normal = normalize(normal);

        float3 wo = optixGetWorldRayDirection();
        if (dot(wo, normal) > 0.0f) {
                normal = -normal;
                packet->entering = true;
        }

        // Transfer to packet
        packet->n = { normal.x, normal.y, normal.z };
}

extern "C" __global__ void __miss__()
{
        uint i0 = optixGetPayload_0();
        uint i1 = optixGetPayload_1();

        Packet *packet = unpack_pointer <Packet> (i0, i1);

        packet->miss = true;
}
