#include "hip/hip_runtime.h"
#include "editor_viewport.cuh"
#include "include/cuda/cast.cuh"
#include "include/cuda/error.cuh"
#include "include/daemons/material.hpp"
#include "optix/sparse_gi_shader.cuh"

// Gaussian blur of irradiance per pixel
// __constant__ float gauss_filter_kernel_5x5[25] = {
//         0.00296902,    0.0133062,    0.0219382,    0.0133062,    0.00296902,
//         0.0133062,    0.0596343,    0.0983203,    0.0596343,    0.0133062,
//         0.0219382,    0.0983203,    0.162103,    0.0983203,    0.0219382,
//         0.0133062,    0.0596343,    0.0983203,    0.0596343,    0.0133062,
//         0.00296902,    0.0133062,    0.0219382,    0.0133062,    0.00296902
// };
//
// __global__
// void box_filter(float4 *irradiance, float4 *out, vk::Extent2D extent, hipSurfaceObject_t normal_surface, int N)
// {
//         int index = threadIdx.x + blockIdx.x * blockDim.x;
//         int base_x = index % extent.width;
//         int base_y = index / extent.width;
//
//         if (base_x < 0 || base_x >= extent.width ||
//             base_y < 0 || base_y >= extent.height)
//                 return;
//
//         float3 sum = make_float3(0.0f);
//
//         float4 raw_central_normal = make_float4(0.0f);
//         surf2Dread(&raw_central_normal, normal_surface, base_x * sizeof(float4), extent.height - (base_y + 1));
//         float3 central_normal = make_float3(raw_central_normal);
//
//         int samples = 0;
//         for (int i = -N; i <= N; i++) {
//                 for (int j = -N; j <= N; j++) {
//                         int x = base_x + i;
//                         int y = base_y + j;
//
//                         if (x < 0 || x >= extent.width ||
//                             y < 0 || y >= extent.height)
//                                 continue;
//
//                         int new_index = x + y * extent.width;
//                         float4 raw_normal = make_float4(0.0f);
//                         surf2Dread(&raw_normal, normal_surface, x * sizeof(float4), extent.height - (y + 1));
//                         float3 normal = make_float3(raw_normal);
//
//                         if (dot(normal, central_normal) < 0.4f)
//                                 continue;
//
//                         sum += make_float3(irradiance[new_index]);
//                         samples++;
//                 }
//         }
//
//         out[index] = samples > 0 ? make_float4(sum/float(samples), 1.0f) : irradiance[index];
//
//         // Copy back to irradiance (except last channel which may be important)
//         irradiance[index].x = out[index].x;
//         irradiance[index].y = out[index].y;
//         irradiance[index].z = out[index].z;
//
//         // TODO: better filter that accounts for normals (bilateral filter...)
// }
//
// __global__
// void gauss_filter(float4 *irradiance, float4 *out,
//                   hipSurfaceObject_t normals,
//                   hipSurfaceObject_t positions,
//                   vk::Extent2D extent)
// {
//         int x = threadIdx.x + blockIdx.x * blockDim.x;
//
//         if (x < 0 || x >= extent.width * extent.height)
//                 return;
//
//         int2 pos = make_int2(x % extent.width, x / extent.width);
//         float4 raw_normal = make_float4(0.0f);
//         float4 raw_pos = make_float4(0.0f);
//         surf2Dread(&raw_normal, normals, pos.x * sizeof(float4), extent.height - (pos.y + 1));
//         surf2Dread(&raw_pos, positions, pos.x * sizeof(float4), extent.height - (pos.y + 1));
//         float3 normal = make_float3(raw_normal);
//         float3 position = make_float3(raw_pos);
//
//         float3 sum = make_float3(0.0f, 0.0f, 0.0f);
//         float sum_weights = 0.0f;
//
//         for (int i = -2; i <= 2; i++) {
//                 for (int j = -2; j <= 2; j++) {
//                         int2 offset = make_int2(i, j);
//                         int2 pos = make_int2(x % extent.width, x / extent.width);
//                         int2 new_pos = pos + offset;
//
//                         if (new_pos.x < 0 || new_pos.x >= extent.width ||
//                                 new_pos.y < 0 || new_pos.y >= extent.height)
//                                 continue;
//
//                         float4 raw_new_normal = make_float4(0.0f);
//                         float4 raw_new_pos = make_float4(0.0f);
//                         surf2Dread(&raw_new_normal, normals, new_pos.x * sizeof(float4), extent.height - (new_pos.y + 1));
//                         surf2Dread(&raw_new_pos, positions, new_pos.x * sizeof(float4), extent.height - (new_pos.y + 1));
//                         float3 new_normal = make_float3(raw_new_normal);
//                         float3 new_position = make_float3(raw_new_pos);
//
//                         if (dot(normal, new_normal) < 0.4f ||
//                                 length(new_position - position) > 0.5f)
//                                 continue;
//
//                         int new_index = new_pos.x + new_pos.y * extent.width;
//                         float weight = gauss_filter_kernel_5x5[(i + 2) + (j + 2) * 5];
//                         sum += make_float3(irradiance[new_index] * weight);
//                         sum_weights += weight;
//                 }
//         }
//
//         out[x] = sum_weights > 0 ? make_float4(sum/sum_weights, 1.0f) : irradiance[x];
//         
//         // Copy back to irradiance (except last channel which may be important)
//         irradiance[x].x = out[x].x;
//         irradiance[x].y = out[x].y;
//         irradiance[x].z = out[x].z;
//
//         // TODO: better filter that accounts for normals (bilateral filter...)
// }

__forceinline__ __device__
float3 cleanse(float3 in)
{
        if (isnan(in.x) || isnan(in.y) || isnan(in.z))
                return make_float3(0.0f);
        return in;
}

struct IrradianceFilterInfo {
        float4 *irradiance;
        float4 *dst;
        float4 *mean_directions;
        
        hipSurfaceObject_t positions;
        hipSurfaceObject_t normals;
        hipSurfaceObject_t uvs;
        hipSurfaceObject_t indices;

        vk::Extent2D extent;
        int radius;
};

__global__
void irradiance_filter(IrradianceFilterInfo info)
{
        int width = info.extent.width;
        int height = info.extent.height;

        int x = threadIdx.x + blockIdx.x * blockDim.x;
        if (x < 0 || x >= width * height)
                return;

        int2 pos = make_int2(x % width, x/width);
        
        // Retrieve surface data
        float4 raw_position;
        float4 raw_normal;
        float4 raw_uv;
        int32_t raw_index;

        surf2Dread(&raw_position, info.positions, pos.x * sizeof(float4),
                   info.extent.height - (pos.y + 1));
        surf2Dread(&raw_normal, info.normals, pos.x * sizeof(float4),
                   info.extent.height - (pos.y + 1));
        surf2Dread(&raw_uv, info.uvs, pos.x * sizeof(float4), info.extent.height -
                   (pos.y + 1));
        surf2Dread(&raw_index, info.indices, pos.x * sizeof(int32_t),
                   info.extent.height - (pos.y + 1));

        float3 normal = make_float3(raw_normal);
        float3 position = make_float3(raw_position);
        float2 uv = make_float2(raw_uv);
        float3 mean_direction = make_float3(info.mean_directions[x]);
        int32_t material_id = raw_index & 0xFFFF;
        
        float3 sum = make_float3(0.0f, 0.0f, 0.0f);
        float wsum = 0;

        int N = info.radius;
        for (int i = -N; i <= N; i++) {
                for (int j = -N; j <= N; j++) {
                        int2 offset = make_int2(i, j);
                        int2 pos = make_int2(x % width, x/width);
                        int2 new_pos = pos + offset;

                        if (new_pos.x < 0 || new_pos.x >= width ||
                                new_pos.y < 0 || new_pos.y >= height)
                                continue;

                        float4 raw_new_position;
                        float4 raw_new_normal;
                        float4 raw_new_uv;
                        int32_t raw_new_index;

                        surf2Dread(&raw_new_position, info.positions, new_pos.x * sizeof(float4),
                                   info.extent.height - (new_pos.y + 1));
                        surf2Dread(&raw_new_normal, info.normals, new_pos.x * sizeof(float4),
                                  info.extent.height - (new_pos.y + 1));
                        surf2Dread(&raw_new_uv, info.uvs, new_pos.x * sizeof(float4),
                                  info.extent.height - (new_pos.y + 1));
                        surf2Dread(&raw_new_index, info.indices, new_pos.x * sizeof(int32_t),
                                   info.extent.height - (new_pos.y + 1));

                        float3 new_position = make_float3(raw_new_position);
                        float3 new_normal = make_float3(raw_new_normal);
                        float3 new_mean_direction = make_float3(info.mean_directions[new_pos.x + new_pos.y * width]);

                        if (raw_new_index == -1
                                || dot(normal, new_normal) < 0.4f
                                || length(new_position - position) > 0.5f)
                                continue;

                        float w = abs(dot(new_mean_direction, normal)/dot(mean_direction, normal));
                        int new_index = new_pos.x + new_pos.y * width;
                        sum += w * make_float3(info.irradiance[new_index]);
                        wsum += w;
                }
        }

        float3 final = cleanse(wsum > 0 ? sum/wsum : make_float3(0.0f, 0.0f, 0.0f));
        info.dst[x] = make_float4(final, 1.0f);
}

__global__
void irradiance_filter_restart(IrradianceFilterInfo info)
{
        int width = info.extent.width;
        int height = info.extent.height;
        int x = threadIdx.x + blockIdx.x * blockDim.x;
        if (x < 0 || x >= width * height)
                return;

        info.irradiance[x].x = info.dst[x].x;
        info.irradiance[x].y = info.dst[x].y;
        info.irradiance[x].z = info.dst[x].z;
}

struct FinalGatherInfo {
        CameraAxis camera;
        cuda::_material *materials;
        float time;
        float4 *color;
        float4 *directions;

        Reservoir <DirectLightingSample> *direct_lighting;
        float4 *indirect_irradiance;
                
        hipSurfaceObject_t position_surface;
        hipSurfaceObject_t normal_surface;
        hipSurfaceObject_t uv_surface;
        hipSurfaceObject_t index_surface;

        Sky sky;
        
        vk::Extent2D extent;

        bool direct;
        bool indirect;
        bool irradiance;
        bool mean_direction;
};

__device__
float3 ray_at(CameraAxis camera, int x, int y)
{
        y = camera.resolution.y - (y + 1);
        float u = 2.0f * float(x) / float(camera.resolution.x) - 1.0f;
        float v = 2.0f * float(y) / float(camera.resolution.y) - 1.0f;
	return normalize(u * camera.U - v * camera.V + camera.W);
}

__global__
void final_gather(FinalGatherInfo info)
{
        int index = threadIdx.x + blockIdx.x * blockDim.x;

        int x = index % info.extent.width;
        int y = index / info.extent.width;

        if (x >= info.extent.width || y >= info.extent.height)
                return;

        // Retrieve surface data
        float4 raw_position;
        float4 raw_normal;
        float4 raw_uv;
        int32_t raw_index;

        surf2Dread(&raw_position, info.position_surface, x * sizeof(float4), info.extent.height - (y + 1));
        surf2Dread(&raw_normal, info.normal_surface, x * sizeof(float4), info.extent.height - (y + 1));
        surf2Dread(&raw_uv, info.uv_surface, x * sizeof(float4), info.extent.height - (y + 1));
        surf2Dread(&raw_index, info.index_surface, x * sizeof(int32_t), info.extent.height - (y + 1));

        // If there is a miss, then exit...
        if (raw_index == -1) {
                float3 ray = ray_at(info.camera, x, y);
                info.color[index] = sky_at(info.sky, ray);
                return;
        }

        int32_t triangle_id = raw_index >> 16;
        int32_t material_id = raw_index & 0xFFFF;

        // Reconstruct the surface hit
        float3 position = { raw_position.x, raw_position.y, raw_position.z };
        float3 normal = { raw_normal.x, raw_normal.y, raw_normal.z };
        float2 uv = { raw_uv.x, raw_uv.y };

        // Correct the normal
        float3 ray = position - info.camera.origin;
        if (dot(ray, normal) > 0.0f)
                normal = -normal;

        float3 seed = make_float3(x, y, info.time);

        cuda::_material m = info.materials[material_id];

        cuda::SurfaceHit sh;
        sh.x = position;
        sh.wo = normalize(info.camera.origin - position);
        sh.n = normalize(normal);
        sh.entering = (raw_normal.w > 0.0f);

        convert_material(m, sh.mat, uv);
        float sign = (sh.mat.type == eTransmission) ? -1.0f : 1.0f;
        sh.x += sign * sh.n * 1e-3f;

        // Get brdf value
        float3 wi = make_float3(info.directions[index]);
        float3 brdf = cuda::brdf(sh, wi, eDiffuse);
        float pdf = cuda::pdf(sh, wi, eDiffuse);

        float3 indirect = pdf > 0.0 ? brdf * make_float3(info.indirect_irradiance[index])/pdf : make_float3(0.0f);
        if (info.irradiance)
                indirect = make_float3(info.indirect_irradiance[index]);

        float3 direct = info.direct_lighting[index].data.Le;
        float3 color = cleanse(info.direct * direct + info.indirect * indirect);
        if (info.mean_direction)
                color = wi * 0.5 + 0.5;

        info.color[index] = make_float4(color, 1.0f);
}

void SparseGI::render(EditorViewport *ev,
                const RenderInfo &render_info,
                const std::vector <Entity> &entities,
                const MaterialDaemon *md)
{
        const Camera &camera = render_info.camera;
        const Transform &camera_transform = render_info.camera_transform;
        
        // Handle resizing
        if (resize_queue.size() > 0) {
                vk::Extent2D new_extent = resize_queue.back();
                resize_queue = {};

                if (launch_params.previous_position != 0)
                        CUDA_CHECK(hipFree((void *) launch_params.previous_position));

                if (launch_params.indirect.screen_irradiance != 0)
                        CUDA_CHECK(hipFree((void *) launch_params.indirect.screen_irradiance));

                if (launch_params.indirect.irradiance_directions != 0)
                        CUDA_CHECK(hipFree((void *) launch_params.indirect.irradiance_directions));
                
                if (launch_params.indirect.direction_samples != 0)
                        CUDA_CHECK(hipFree((void *) launch_params.indirect.direction_samples));

                if (launch_params.indirect.final_irradiance != 0)
                        CUDA_CHECK(hipFree((void *) launch_params.indirect.final_irradiance));
                
                if (launch_params.direct_lighting != 0)
                        CUDA_CHECK(hipFree((void *) launch_params.direct_lighting));
                
                if (launch_params.indirect.block_offsets != 0)
                        CUDA_CHECK(hipFree((void *) launch_params.indirect.block_offsets));

                int size = new_extent.width * new_extent.height;
                launch_params.previous_position = cuda::alloc <float4> (size);
                // CUDA_CHECK(hipMalloc((void **) &launch_params.previous_position, size * sizeof(float4)));
                CUDA_CHECK(hipMalloc((void **) &launch_params.indirect.screen_irradiance, size * sizeof(float4)));
                CUDA_CHECK(hipMalloc((void **) &launch_params.indirect.final_irradiance, size * sizeof(float4)));
                CUDA_CHECK(hipMalloc((void **) &launch_params.indirect.irradiance_directions, size * sizeof(float4)));
                CUDA_CHECK(hipMalloc((void **) &launch_params.indirect.direction_samples, size * sizeof(float)));
                CUDA_CHECK(hipMalloc((void **) &launch_params.direct_lighting, size * sizeof(Reservoir <DirectLightingSample>)));

                // Generate block offsets
                uint N2 = launch_params.indirect.N * launch_params.indirect.N;
                uint2 nblocks;
                nblocks.x = 1 + (new_extent.width / launch_params.indirect.N);
                nblocks.y = 1 + (new_extent.height / launch_params.indirect.N);

                std::vector <uint> block_offsets(nblocks.x * nblocks.y);
                std::mt19937 rng;
                std::uniform_int_distribution <uint> dist(0, N2 - 1);
                for (uint i = 0; i < block_offsets.size(); i++) {
                        uint offset = dist(rng);
                        block_offsets[i] = offset;
                }

                launch_params.indirect.block_offsets = cuda::make_buffer(block_offsets);
        }

        // Configure launch parameters
        launch_params.time = ev->common_rtx.timer.elapsed_start();
        launch_params.dirty = render_info.camera_transform_dirty;
        launch_params.reset = ev->render_state.sparse_gi_reset
                        | ev->common_rtx.material_reset
                        | manual_reset;
        launch_params.samples++;

        uint N = launch_params.indirect.N;
        launch_params.counter = (launch_params.counter + 1) % (N * N);
                
        if (launch_params.reset)
                manual_reset = false;

        ev->render_state.sparse_gi_reset = false;
        if (ev->render_state.sparse_gi_reset) {
                launch_params.previous_view = camera.view_matrix(camera_transform);
                launch_params.previous_projection = camera.perspective_matrix();
                launch_params.samples = 0;
        }

        // Configure camera axis
        auto uvw = uvw_frame(camera, camera_transform);

        launch_params.camera.U = cuda::to_f3(uvw.u);
        launch_params.camera.V = cuda::to_f3(uvw.v);
        launch_params.camera.W = cuda::to_f3(uvw.w);
        launch_params.camera.origin = cuda::to_f3(render_info.camera_transform.position);
        launch_params.camera.resolution = { ev->extent.width, ev->extent.height };

        // Configure surfaces
        launch_params.position_surface = ev->framebuffer_images.cu_position_surface;
        launch_params.normal_surface = ev->framebuffer_images.cu_normal_surface;
        launch_params.uv_surface = ev->framebuffer_images.cu_uv_surface;
        launch_params.index_surface = ev->framebuffer_images.cu_material_index_surface;

        launch_params.materials = (cuda::_material *) ev->common_rtx.dev_materials;

        launch_params.sky.texture = ev->environment_map.texture;
        launch_params.sky.enabled = ev->environment_map.valid;
        
        SparseGIParameters *dev_params = (SparseGIParameters *) dev_launch_params;
        CUDA_CHECK(hipMemcpy(dev_params, &launch_params, sizeof(SparseGIParameters), hipMemcpyHostToDevice));
        
        OPTIX_CHECK(
                optixLaunch(pipeline, 0,
                        dev_launch_params,
                        sizeof(SparseGIParameters),
                        &sbt, ev->extent.width, ev->extent.height, 1
                )
        );

        // TODO: afterward perform a brdf convolution
        // TODO: separate visibility for direct lighting?

        CUDA_SYNC_CHECK();

        // Final gather
        if (filter) {
                IrradianceFilterInfo info;
                info.irradiance = launch_params.indirect.screen_irradiance;
                info.dst = launch_params.indirect.final_irradiance;
                info.normals = launch_params.normal_surface;
                info.mean_directions = launch_params.indirect.irradiance_directions;
                info.positions = launch_params.position_surface;
                info.uvs = launch_params.uv_surface;
                info.indices = launch_params.index_surface;
                info.extent = ev->extent;
                info.radius = 2;

                uint blocks = (ev->extent.width * ev->extent.height + 255) / 256;
                // irradiance_filter <<< blocks, 256 >>> (info);

                for (int i = 0; i < 3; i++) {
                        // TODO: a trous style without needing to block and copy
                        irradiance_filter <<< blocks, 256 >>> (info);
                        CUDA_SYNC_CHECK();
                        irradiance_filter_restart <<< blocks, 256 >>> (info);
                        CUDA_SYNC_CHECK();
                }

                // CUDA_SYNC_CHECK();
        } else {
                CUDA_CHECK(hipMemcpy(
                        (void *) launch_params.indirect.final_irradiance,
                        (void *) launch_params.indirect.screen_irradiance,
                        ev->extent.width * ev->extent.height * sizeof(float4),
                        hipMemcpyDeviceToDevice
                ));
        }

        FinalGatherInfo info;
        info.directions = launch_params.indirect.irradiance_directions;
        info.camera = launch_params.camera;
        info.color = ev->common_rtx.dev_color;
        info.direct_lighting = launch_params.direct_lighting;
        info.index_surface = launch_params.index_surface;
        info.indirect_irradiance = launch_params.indirect.final_irradiance;
        info.materials = (cuda::_material *) ev->common_rtx.dev_materials;
        info.normal_surface = launch_params.normal_surface;
        info.position_surface = launch_params.position_surface;
        info.time = launch_params.time;
        info.uv_surface = launch_params.uv_surface;
        info.sky.texture = ev->environment_map.texture;
        info.sky.enabled = ev->environment_map.valid;
        info.extent = ev->extent;
        info.direct = direct;
        info.indirect = indirect;
        info.irradiance = irradiance;
        info.mean_direction = mean_direction;

        uint blocks = (ev->extent.width * ev->extent.height + 255) / 256;
        final_gather <<< blocks, 256 >>> (info);
        // TODO: push to a stream and sync only when next frame is ready
        CUDA_SYNC_CHECK();
        
        // Update previous view and projection matrices
        launch_params.previous_view = camera.view_matrix(camera_transform);
        launch_params.previous_projection = camera.perspective_matrix();
        launch_params.previous_origin = cuda::to_f3(render_info.camera_transform.position);

        // Report any IO exchanges
        // std::string io = optix_io_read(&launch_params.io);
        // if (io.size() > 0) {
        //         std::cout << "Sparse GI output: \"" << io << "\"" << std::endl;
        //         optix_io_clear(&launch_params.io);
        // }
}
